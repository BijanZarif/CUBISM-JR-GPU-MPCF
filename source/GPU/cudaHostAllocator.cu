/* *
 * cudaHostAllocator.cu
 *
 * Created by Fabian Wermelinger on 06/06/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */

#include <hip/hip_runtime.h>
#include <cstdlib>

void *_cudaAllocHost(const std::size_t bytes)
{
    void *palloc;
    hipHostAlloc(&palloc, bytes, hipHostMallocDefault);
    return palloc;
}

void _cudaFreeHost(void *ptr)
{
    hipHostFree(ptr);
}
