#include "hip/hip_runtime.h"
/* *
 * GPUhousehold.cu
 *
 * Created by Fabian Wermelinger on 6/24/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */
#include <stdio.h>
#include <vector>
#include <algorithm>
using namespace std;

#include "GPU.h" // includes Types.h

#ifdef _CUDA_TIMER_
#include "CUDA_Timer.cuh"
#endif


enum { VSIZE = NodeBlock::NVAR };

///////////////////////////////////////////////////////////////////////////////
// GLOBAL VARIABLES
///////////////////////////////////////////////////////////////////////////////
RealPtrVec_t d_tmp(VSIZE, NULL);
RealPtrVec_t d_rhs(VSIZE, NULL);
RealPtrVec_t d_xgl(VSIZE, NULL);
RealPtrVec_t d_xgr(VSIZE, NULL);
RealPtrVec_t d_ygl(VSIZE, NULL);
RealPtrVec_t d_ygr(VSIZE, NULL);

/* RealPtrVec_t d_flux(VSIZE, NULL); */
RealPtrVec_t d_xflux(VSIZE, NULL);
RealPtrVec_t d_yflux(VSIZE, NULL);
RealPtrVec_t d_zflux(VSIZE, NULL);

// extraterms for advection equations
Real *d_Gm, *d_Gp;
Real *d_Pm, *d_Pp;
Real *d_hllc_vel;
Real *d_sumG, *d_sumP, *d_divU;

// 3D arrays (GPU input)
vector<hipArray_t> d_GPUin(VSIZE, NULL);

// Max SOS
int *h_maxSOS; // host, mapped
int *d_maxSOS; // device, mapped (different address)

// use non-null stream (async)
hipStream_t stream1;
hipStream_t stream2;
hipStream_t stream3;

// events
hipEvent_t divergence_completed;
hipEvent_t update_completed;
hipEvent_t h2d_3Darray_completed;
hipEvent_t h2d_tmp_completed;
hipEvent_t d2h_rhs_completed;
hipEvent_t d2h_tmp_completed;


///////////////////////////////////////////////////////////////////////////////
// IMPLEMENTATION
///////////////////////////////////////////////////////////////////////////////
static void _h2d_3DArray(hipArray_t dst, const Real * const src, const int nslices)
{
    hipMemcpy3DParms copyParams = {0};
    copyParams.extent            = make_hipExtent(NodeBlock::sizeX, NodeBlock::sizeY, nslices);
    copyParams.kind              = hipMemcpyHostToDevice;
    copyParams.srcPtr            = make_hipPitchedPtr((void *)src, NodeBlock::sizeX * sizeof(Real), NodeBlock::sizeX, NodeBlock::sizeY);
    copyParams.dstArray          = dst;

    hipMemcpy3DAsync(&copyParams, stream1);
}


extern "C"
{
    ///////////////////////////////////////////////////////////////////////////
    // GPU Memory alloc / dealloc
    ///////////////////////////////////////////////////////////////////////////
    void GPU::alloc(void** sos, const uint_t nslices, const bool isroot)
    {
#ifndef _MUTE_GPU_
        /* hipDeviceReset(); */
        /* hipSetDeviceFlags(hipDeviceMapHost); */

        // processing slice size (normal to z-direction)
        const uint_t SLICE_GPU = NodeBlock::sizeX * NodeBlock::sizeY;

        // GPU output size
        const uint_t outputSize = SLICE_GPU * nslices;

        // fluxes
        const uint_t xflxSize = (NodeBlock::sizeX+1)*NodeBlock::sizeY*nslices;
        const uint_t yflxSize = NodeBlock::sizeX*(NodeBlock::sizeY+1)*nslices;
        const uint_t zflxSize = NodeBlock::sizeX*NodeBlock::sizeY*(nslices+1);
        const uint_t maxflxSize = max(xflxSize, max(yflxSize, zflxSize));

        // x-/yghosts
        const uint_t xgSize = 3*NodeBlock::sizeY*nslices;
        const uint_t ygSize = NodeBlock::sizeX*3*nslices;

        // GPU allocation
        hipChannelFormatDesc fmt = hipCreateChannelDesc<Real>();
        for (int var = 0; var < VSIZE; ++var)
        {
            //tmp
            hipMalloc(&d_tmp[var], outputSize*sizeof(Real));
            hipMemset(d_tmp[var], 0, outputSize*sizeof(Real));

            // rhs
            hipMalloc(&d_rhs[var], outputSize*sizeof(Real));
            hipMemset(d_rhs[var], 0, outputSize*sizeof(Real));

            // fluxes
            /* hipMalloc(&d_flux[var], maxflxSize*sizeof(Real)); */
            /* hipMemset(d_flux[var], 0, maxflxSize*sizeof(Real)); */
            hipMalloc(&d_xflux[var], xflxSize*sizeof(Real));
            hipMalloc(&d_yflux[var], yflxSize*sizeof(Real));
            hipMalloc(&d_zflux[var], zflxSize*sizeof(Real));
            hipMemset(d_xflux[var], 0, xflxSize*sizeof(Real));
            hipMemset(d_yflux[var], 0, yflxSize*sizeof(Real));
            hipMemset(d_zflux[var], 0, zflxSize*sizeof(Real));

            // x-/yghosts
            hipMalloc(&d_xgl[var], xgSize*sizeof(Real));
            hipMalloc(&d_xgr[var], xgSize*sizeof(Real));

            hipMalloc(&d_ygl[var], ygSize*sizeof(Real));
            hipMalloc(&d_ygr[var], ygSize*sizeof(Real));

            // GPU input (+6 slices for zghosts)
            hipMalloc3DArray(&d_GPUin[var], &fmt, make_hipExtent(NodeBlock::sizeX, NodeBlock::sizeY, nslices+6));
        }

        // extraterm for advection
        hipMalloc(&d_Gm, maxflxSize * sizeof(Real));
        hipMalloc(&d_Gp, maxflxSize * sizeof(Real));
        hipMalloc(&d_Pm, maxflxSize * sizeof(Real));
        hipMalloc(&d_Pp, maxflxSize * sizeof(Real));
        hipMalloc(&d_hllc_vel, maxflxSize * sizeof(Real));
        hipMalloc(&d_sumG, outputSize * sizeof(Real));
        hipMalloc(&d_sumP, outputSize * sizeof(Real));
        hipMalloc(&d_divU, outputSize * sizeof(Real));

        // zero-copy maxSOS (TODO: should this be unsigned int?)
        hipHostAlloc((void**)&h_maxSOS, sizeof(int), hipHostMallocMapped);
        hipHostGetDevicePointer(&d_maxSOS, h_maxSOS, 0);
        *(int**)sos = h_maxSOS; // return a reference to the caller

        // create streams
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);

        // create events
        hipEventCreate(&divergence_completed);
        hipEventCreate(&update_completed);
        hipEventCreate(&h2d_3Darray_completed);
        hipEventCreate(&h2d_tmp_completed);
        hipEventCreate(&d2h_rhs_completed);
        hipEventCreate(&d2h_tmp_completed);

        // Stats
        if (isroot)
        {
            int dev;
            hipDeviceProp_t prop;
            hipGetDevice(&dev);
            hipGetDeviceProperties(&prop, dev);

            printf("=====================================================================\n");
            printf("[GPU ALLOCATION FOR %s]\n", prop.name);
            printf("[%5.1f MB (input GPU)]\n", VSIZE*(SLICE_GPU*(nslices+6))*sizeof(Real) / 1024. / 1024);
            printf("[%5.1f MB (tmp)]\n", VSIZE*outputSize*sizeof(Real) / 1024. / 1024);
            printf("[%5.1f MB (rhs)]\n", VSIZE*outputSize*sizeof(Real) / 1024. / 1024);
            printf("[%5.1f MB (flux storage)]\n", VSIZE*(xflxSize + yflxSize + zflxSize)*sizeof(Real) / 1024. / 1024);
            printf("[%5.1f MB (x/yghosts)]\n", VSIZE*(xgSize + ygSize)*2*sizeof(Real) / 1024. / 1024);
            printf("[%5.1f MB (extraterm)]\n", (5*maxflxSize + 3*outputSize)*sizeof(Real) / 1024. / 1024);
            GPU::tell_memUsage_GPU();
            printf("=====================================================================\n");
        }
#endif
    }


    void GPU::dealloc(const bool isroot)
    {
#ifndef _MUTE_GPU_
        for (int var = 0; var < VSIZE; ++var)
        {
            // tmp
            hipFree(d_tmp[var]);

            // rhs
            hipFree(d_rhs[var]);

            // fluxes
            /* hipFree(d_flux[var]); */
            hipFree(d_xflux[var]);
            hipFree(d_yflux[var]);
            hipFree(d_zflux[var]);

            // x-/yghosts
            hipFree(d_xgl[var]);
            hipFree(d_xgr[var]);
            hipFree(d_ygl[var]);
            hipFree(d_ygr[var]);

            // input GPU
            hipFreeArray(d_GPUin[var]);
        }

        // extraterms
        hipFree(d_Gm);
        hipFree(d_Gp);
        hipFree(d_Pm);
        hipFree(d_Pp);
        hipFree(d_hllc_vel);
        hipFree(d_sumG);
        hipFree(d_sumP);
        hipFree(d_divU);

        // Max SOS
        hipHostFree(h_maxSOS);

        // destroy streams
        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);
        hipStreamDestroy(stream3);

        // destroy events
        hipEventDestroy(divergence_completed);
        hipEventDestroy(update_completed);
        hipEventDestroy(h2d_3Darray_completed);
        hipEventDestroy(h2d_tmp_completed);
        hipEventDestroy(d2h_rhs_completed);
        hipEventDestroy(d2h_tmp_completed);

        // Stats
        if (isroot)
        {
            int dev;
            hipDeviceProp_t prop;
            hipGetDevice(&dev);
            hipGetDeviceProperties(&prop, dev);

            printf("=====================================================================\n");
            printf("[FREE GPU %s]\n", prop.name);
            GPU::tell_memUsage_GPU();
            printf("=====================================================================\n");
        }
#endif
    }


    ///////////////////////////////////////////////////////////////////////////
    // H2D / D2H
    ///////////////////////////////////////////////////////////////////////////
    void GPU::upload_xy_ghosts(const uint_t Nxghost, const RealPtrVec_t& xghost_l, const RealPtrVec_t& xghost_r,
            const uint_t Nyghost, const RealPtrVec_t& yghost_l, const RealPtrVec_t& yghost_r)
    {
#ifndef _MUTE_GPU_
        // TODO: use larger arrays for ghosts to minimize API overhead +
        // increase BW performance
        tCUDA_START(stream1)
        for (int i = 0; i < VSIZE; ++i)
        {
            // x
            hipMemcpyAsync(d_xgl[i], xghost_l[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
            hipMemcpyAsync(d_xgr[i], xghost_r[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
            // y
            hipMemcpyAsync(d_ygl[i], yghost_l[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
            hipMemcpyAsync(d_ygr[i], yghost_r[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
        }
        tCUDA_STOP(stream1, "[GPU UPLOAD X/YGHOSTS]: ")
#endif
    }


    void GPU::h2d_3DArray(const RealPtrVec_t& src, const uint_t nslices)
    {
#ifndef _MUTE_GPU_
        tCUDA_START(stream1)
        for (int i = 0; i < VSIZE; ++i)
            _h2d_3DArray(d_GPUin[i], src[i], nslices);
        tCUDA_STOP(stream1, "[GPU UPLOAD 3DArray]: ")
        hipEventRecord(h2d_3Darray_completed, stream1);
#endif
    }


    void GPU::h2d_tmp(const RealPtrVec_t& src, const uint_t N)
    {
#ifndef _MUTE_GPU_
        hipStreamWaitEvent(stream3, h2d_3Darray_completed, 0);

        tCUDA_START(stream3)
        for (int i = 0; i < VSIZE; ++i)
            hipMemcpyAsync(d_tmp[i], src[i], N*sizeof(Real), hipMemcpyHostToDevice, stream3);
        tCUDA_STOP(stream3, "[GPU UPLOAD TMP]: ")
        hipEventRecord(h2d_tmp_completed, stream3);
#endif
    }


    void GPU::d2h_rhs(RealPtrVec_t& dst, const uint_t N)
    {
#ifndef _MUTE_GPU_
        hipStreamWaitEvent(stream2, divergence_completed, 0);

        // copy content of d_rhs to host, using the stream2 (after divergence)
        tCUDA_START(stream2)
        for (int i = 0; i < VSIZE; ++i)
            hipMemcpyAsync(dst[i], d_rhs[i], N*sizeof(Real), hipMemcpyDeviceToHost, stream2);
        tCUDA_STOP(stream2, "[GPU DOWNLOAD RHS]: ")
        hipEventRecord(d2h_rhs_completed, stream2);
#endif
    }


    void GPU::d2h_tmp(RealPtrVec_t& dst, const uint_t N)
    {
#ifndef _MUTE_GPU_
        hipStreamWaitEvent(stream2, update_completed, 0);

        // copy content of d_tmp to host, using the stream1
        tCUDA_START(stream1)
        for (int i = 0; i < VSIZE; ++i)
            hipMemcpyAsync(dst[i], d_tmp[i], N*sizeof(Real), hipMemcpyDeviceToHost, stream2);
        tCUDA_STOP(stream1, "[GPU DOWNLOAD TMP]: ")
        hipEventRecord(d2h_tmp_completed, stream2);
#endif
    }


    ///////////////////////////////////////////////////////////////////////////
    // Sync
    ///////////////////////////////////////////////////////////////////////////
    void GPU::h2d_3DArray_wait()
    {
#ifndef _MUTE_GPU_
        // wait until h2d_3DArray has finished
        hipEventSynchronize(h2d_3Darray_completed);
#endif
    }


    void GPU::d2h_rhs_wait()
    {
#ifndef _MUTE_GPU_
        // wait until d2h_rhs has finished
        hipEventSynchronize(d2h_rhs_completed);
#endif
    }


    void GPU::d2h_tmp_wait()
    {
#ifndef _MUTE_GPU_
        // wait until d2h_tmp has finished
        hipEventSynchronize(d2h_tmp_completed);
#endif
    }


    void GPU::syncGPU()
    {
#ifndef _MUTE_GPU_
        hipDeviceSynchronize();
#endif
    }


    void GPU::syncStream(streamID s)
    {
#ifndef _MUTE_GPU_
        switch (s)
        {
            case S1: hipStreamSynchronize(stream1); break;
            case S2: hipStreamSynchronize(stream2); break;
        }
#endif
    }


    ///////////////////////////////////////////////////////////////////////////
    // Stats
    ///////////////////////////////////////////////////////////////////////////
    void GPU::tell_memUsage_GPU()
    {
#ifndef _MUTE_GPU_
        size_t free_byte, total_byte;
        const int status = hipMemGetInfo(&free_byte, &total_byte);
        if (hipSuccess != status)
        {
            printf("Hoppla! Can not get memory stats from GPU...\n");
            return;
        }
        const size_t used = total_byte - free_byte;
        printf("GPU memory usage: free = %5.1f MB, total = %5.1f MB (%5.1f MB used)\n",
                (double)free_byte / 1024 / 1024,
                (double)total_byte / 1024 / 1024,
                (double)used / 1024 / 1024);
#endif
    }


    void GPU::tell_GPU()
    {
#ifndef _MUTE_GPU_
        int dev;
        hipDeviceProp_t prop;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);
        printf("Using device %d (%s)\n", dev, prop.name);
#endif
    }
}
