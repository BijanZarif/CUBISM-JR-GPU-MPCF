#include "hip/hip_runtime.h"
/* *
 * GPUhousehold.cu
 *
 * Created by Fabian Wermelinger on 6/24/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */
#include "GPU.cuh"

#include <cstdio>
#include <cstdlib>
#include <algorithm>

///////////////////////////////////////////////////////////////////////////////
// GLOBAL VARIABLES
///////////////////////////////////////////////////////////////////////////////
// reconstruction
real_vector_t d_recon_p(VSIZE, NULL);
real_vector_t d_recon_m(VSIZE, NULL);

// extraterms for advection equations
Real *d_sumG, *d_sumP, *d_divU;

// Max SOS
int *h_maxSOS; // host, mapped
int *d_maxSOS; // device, mapped (different address)

struct GPU_COMM gpu_comm[_NUM_GPU_BUF_];

// use non-null stream (async)
hipStream_t *stream;

// events
hipEvent_t *event_h2d;
hipEvent_t *event_d2h;
hipEvent_t *event_compute;


///////////////////////////////////////////////////////////////////////////////
// IMPLEMENTATION
///////////////////////////////////////////////////////////////////////////////
Profiler GPU::profiler; // combined CPU/GPU profiler

static void _h2d_3DArray(hipArray_t dst, const Real * const src, const int nslices, const int s_id)
{
    hipMemcpy3DParms copyParams = {0};
    copyParams.extent            = make_hipExtent(NX, NY, nslices);
    copyParams.kind              = hipMemcpyHostToDevice;
    copyParams.srcPtr            = make_hipPitchedPtr((void *)src, NX * sizeof(Real), NX, NY);
    copyParams.dstArray          = dst;

    hipMemcpy3DAsync(&copyParams, stream[s_id]);
}


///////////////////////////////////////////////////////////////////////////
// GPU Memory alloc / dealloc
///////////////////////////////////////////////////////////////////////////
void GPU::alloc(void** sos, const uint_t nslices, const bool isroot)
{
    /* hipDeviceReset(); */
    /* hipSetDeviceFlags(hipDeviceMapHost); */

    // processing slice size (normal to z-direction)
    const uint_t SLICE_GPU = NX * NY;

    // GPU output size
    const uint_t outputSize = SLICE_GPU * nslices;

    // fluxes
    const uint_t xflxSize = (NX+1)*NY*nslices;
    const uint_t yflxSize = NX*(NY+1)*nslices;
    const uint_t zflxSize = NX*NY*(nslices+1);
    const uint_t maxflxSize = max(xflxSize, max(yflxSize, zflxSize));

    // x-/yghosts
    const uint_t xgSize = 3*NY*nslices;
    const uint_t ygSize = NX*3*nslices;

    // GPU intermediate data
    size_t computational_bytes = 0;
    for (int var = 0; var < VSIZE; ++var)
    {
        hipMalloc(&d_recon_p[var], maxflxSize*sizeof(Real));
        hipMalloc(&d_recon_m[var], maxflxSize*sizeof(Real));
        computational_bytes += 2*maxflxSize * sizeof(Real);
    }

    // extraterm for advection
    hipMalloc(&d_sumG, outputSize * sizeof(Real));
    hipMalloc(&d_sumP, outputSize * sizeof(Real));
    hipMalloc(&d_divU, outputSize * sizeof(Real));
    computational_bytes += 3 * outputSize * sizeof(Real);

    // Communication buffers
    size_t ghost_bytes  = 0;
    size_t trans_bytes  = 0;
    hipChannelFormatDesc fmt = hipCreateChannelDesc<Real>();
    for (int i = 0; i < _NUM_GPU_BUF_; ++i)
    {
        GPU_COMM * const mybuf = &gpu_comm[i];
        for (int var = 0; var < VSIZE; ++var)
        {
            // x-/yghosts
            hipMalloc(&(mybuf->d_xgl[var]), xgSize*sizeof(Real));
            hipMalloc(&(mybuf->d_xgr[var]), xgSize*sizeof(Real));
            hipMalloc(&(mybuf->d_ygl[var]), ygSize*sizeof(Real));
            hipMalloc(&(mybuf->d_ygr[var]), ygSize*sizeof(Real));
            ghost_bytes += 2 * xgSize * sizeof(Real) + 2 * ygSize * sizeof(Real);

            // GPU transition buffer
            hipMalloc(&(mybuf->d_inout[var]), SLICE_GPU*(nslices+6)*sizeof(Real));
            trans_bytes += SLICE_GPU * (nslices+6) * sizeof(Real);

            // GPU tex buffer (+6 slices for zghosts)
            hipMalloc3DArray(&(mybuf->d_GPU3D[var]), &fmt, make_hipExtent(NX, NY, nslices+6));
            computational_bytes += NX * NY * (nslices+6) * sizeof(Real);
        }
    }

    // zero-copy maxSOS (TODO: should this be unsigned int?)
    hipHostAlloc((void**)&h_maxSOS, sizeof(int), hipHostMallocMapped);
    computational_bytes += sizeof(int);
    hipHostGetDevicePointer(&d_maxSOS, h_maxSOS, 0);
    *(int**)sos = h_maxSOS; // return a reference to the caller

    // create streams
    stream = (hipStream_t *) malloc(_NUM_STREAMS_ * sizeof(hipStream_t));
    assert(stream != NULL);
    for (int i = 0 ; i < _NUM_STREAMS_; ++i)
        hipStreamCreate(&stream[i]);

    // create events
    event_h2d     = (hipEvent_t *) malloc(_NUM_STREAMS_ * sizeof(hipEvent_t));
    event_d2h     = (hipEvent_t *) malloc(_NUM_STREAMS_ * sizeof(hipEvent_t));
    event_compute = (hipEvent_t *) malloc(_NUM_STREAMS_ * sizeof(hipEvent_t));
    assert(event_h2d != NULL);
    assert(event_d2h != NULL);
    assert(event_compute != NULL);
    for (int i = 0; i < _NUM_STREAMS_; ++i)
    {
        hipEventCreate(&event_h2d[i]);
        hipEventCreate(&event_d2h[i]);
        hipEventCreate(&event_compute[i]);
    }

    // Stats
    int dev;
    hipDeviceProp_t prop;
    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop, dev);
    const bool tooMuch = (trans_bytes + ghost_bytes + computational_bytes) > prop.totalGlobalMem;
    if (isroot)
    {
        printf("=====================================================================\n");
        printf("[GPU ALLOCATION FOR %s (GMEM = %6.1f MB)]\n", prop.name, prop.totalGlobalMem / 1024. / 1024.);
        printf("[%6.1f MB (GPU chunk data)]\n",  trans_bytes / 1024. / 1024.);
        printf("[%6.1f MB (GPU ghosts)]\n",      ghost_bytes / 1024. / 1024.);
        printf("[%6.1f MB (Compute storage)]\n", computational_bytes / 1024. / 1024.);
        GPU::tell_memUsage_GPU();
        printf("=====================================================================\n");
        if (tooMuch)
            printf("ERROR: You are trying to eat more than you can chew!\nnslices = %d -> reduce this parameter to make room on the GPU.\n", nslices);
    }
    if (tooMuch) abort();
}


void GPU::dealloc(const bool isroot)
{
    for (int var = 0; var < VSIZE; ++var)
    {
        hipFree(d_recon_p[var]);
        hipFree(d_recon_m[var]);
    }

    // extraterms
    hipFree(d_sumG);
    hipFree(d_sumP);
    hipFree(d_divU);

    for (int i = 0; i < _NUM_GPU_BUF_; ++i)
    {
        GPU_COMM * const mybuf = &gpu_comm[i];
        for (int var = 0; var < VSIZE; ++var)
        {
            // x-/yghosts
            hipFree(mybuf->d_xgl[var]);
            hipFree(mybuf->d_xgr[var]);
            hipFree(mybuf->d_ygl[var]);
            hipFree(mybuf->d_ygr[var]);

            // GPU transition buffer
            hipFree(mybuf->d_inout[var]);

            // GPU tex buffer
            hipFreeArray(mybuf->d_GPU3D[var]);
        }
    }

    // Max SOS
    hipHostFree(h_maxSOS);

    // destroy streams
    for (int i = 0; i < _NUM_STREAMS_; ++i)
        hipStreamDestroy(stream[i]);
    free(stream);

    // destroy events
    for (int i = 0; i < _NUM_STREAMS_; ++i)
    {
        hipEventDestroy(event_h2d[i]);
        hipEventDestroy(event_d2h[i]);
        hipEventDestroy(event_compute[i]);
    }
    free(event_h2d);
    free(event_d2h);
    free(event_compute);

    // Stats
    if (isroot)
    {
        int dev;
        hipDeviceProp_t prop;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);

        printf("=====================================================================\n");
        printf("[FREE GPU %s]\n", prop.name);
        GPU::tell_memUsage_GPU();
        printf("=====================================================================\n");
    }
}


///////////////////////////////////////////////////////////////////////////
// COMMUNICATION H2D / D2H
///////////////////////////////////////////////////////////////////////////
void GPU::h2d_input(
        const uint_t Nxghost, const real_vector_t& xghost_l, const real_vector_t& xghost_r,
        const uint_t Nyghost, const real_vector_t& yghost_l, const real_vector_t& yghost_r,
        const real_vector_t& src, const uint_t nslices,
        const uint_t gbuf_id, const int chunk_id)
{
    assert(gbuf_id < _NUM_GPU_BUF_);

    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    // previous stream has priority, don't interrupt
    const uint_t s_idm1 = ((chunk_id-1) + _NUM_STREAMS_) % _NUM_STREAMS_;
    assert(s_idm1 < _NUM_STREAMS_);
    hipStreamWaitEvent(stream[s_id], event_h2d[s_idm1], 0);

    char prof_item[256];

    // TODO: use larger arrays for ghosts to minimize API overhead +
    // increase BW performance. (LOW PRIORITY)
    sprintf(prof_item, "SEND GHOSTS (%d)", s_id);
    GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
    for (int i = 0; i < VSIZE; ++i)
    {
        // x
        hipMemcpyAsync(mybuf->d_xgl[i], xghost_l[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
        hipMemcpyAsync(mybuf->d_xgr[i], xghost_r[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
        // y
        hipMemcpyAsync(mybuf->d_ygl[i], yghost_l[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
        hipMemcpyAsync(mybuf->d_ygr[i], yghost_r[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
    }
    GPU::profiler.pop_stopCUDA();

    // h2d chunk + zghosts
    sprintf(prof_item, "SEND CHUNK (%d)", s_id);
    GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
    for (int i = 0; i < VSIZE; ++i)
        hipMemcpyAsync(mybuf->d_inout[i], src[i], NX*NY*nslices*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
    GPU::profiler.pop_stopCUDA();

    hipEventRecord(event_h2d[s_id], stream[s_id]);
}


void GPU::h2d_3DArray(const real_vector_t& src, const uint_t nslices,
        const uint_t gbuf_id, const int chunk_id)
{
    assert(gbuf_id < _NUM_GPU_BUF_);

    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    assert(s_id < _NUM_STREAMS_);
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    char prof_item[256];
    sprintf(prof_item, "SEND 3DARRAY (%d)", s_id);

    GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
    for (int i = 0; i < VSIZE; ++i)
        _h2d_3DArray(mybuf->d_GPU3D[i], src[i], nslices, s_id);
    GPU::profiler.pop_stopCUDA();

    hipEventRecord(event_h2d[s_id], stream[s_id]);
}


void GPU::d2h_divF(real_vector_t& dst, const uint_t N,
        const uint_t gbuf_id, const int chunk_id)
{
    assert(gbuf_id < _NUM_GPU_BUF_);

    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    assert(s_id < _NUM_STREAMS_);
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    char prof_item[256];
    sprintf(prof_item, "RECV DIVF (%d)", s_id);

    GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
    for (int i = 0; i < VSIZE; ++i)
        hipMemcpyAsync(dst[i], mybuf->d_inout[i], N*sizeof(Real), hipMemcpyDeviceToHost, stream[s_id]);
    GPU::profiler.pop_stopCUDA();

    hipEventRecord(event_d2h[s_id], stream[s_id]);
}


///////////////////////////////////////////////////////////////////////////
// Sync
///////////////////////////////////////////////////////////////////////////
void GPU::wait_h2d(const int chunk_id)
{
    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    hipEventSynchronize(event_h2d[s_id]);
}


void GPU::wait_d2h(const int chunk_id)
{
    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    hipEventSynchronize(event_d2h[s_id]);
}


void GPU::syncGPU()
{
    hipDeviceSynchronize();
}


void GPU::syncStream(const int chunk_id)
{
    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    hipStreamSynchronize(stream[s_id]);
}


///////////////////////////////////////////////////////////////////////////
// Stats
///////////////////////////////////////////////////////////////////////////
void GPU::tell_memUsage_GPU()
{
    size_t free_byte, total_byte;
    const int status = hipMemGetInfo(&free_byte, &total_byte);
    if (hipSuccess != status)
    {
        printf("Can not get memory stats from GPU...\n");
        return;
    }
    const size_t used = total_byte - free_byte;
    printf("GPU memory usage: free = %5.1f MB, total = %5.1f MB (%5.1f MB used)\n",
            (double)free_byte / 1024 / 1024,
            (double)total_byte / 1024 / 1024,
            (double)used / 1024 / 1024);
}


void GPU::tell_GPU()
{
    int dev;
    hipDeviceProp_t prop;
    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop, dev);
    printf("Using device %d (%s)\n", dev, prop.name);
}
