#include "hip/hip_runtime.h"
/* *
 * GPUhousehold.cu
 *
 * Created by Fabian Wermelinger on 6/24/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */
#include "GPU.h" // includes Types.h

#include <stdio.h>
#include <vector>
#include <algorithm>
using namespace std;

enum { VSIZE = NodeBlock::NVAR };

///////////////////////////////////////////////////////////////////////////////
// GLOBAL VARIABLES
///////////////////////////////////////////////////////////////////////////////
real_vector_t d_flux(VSIZE, NULL);
real_vector_t d_xgl(VSIZE, NULL);
real_vector_t d_xgr(VSIZE, NULL);
real_vector_t d_ygl(VSIZE, NULL);
real_vector_t d_ygr(VSIZE, NULL);

// extraterms for advection equations
Real *d_Gm, *d_Gp;
Real *d_Pm, *d_Pp;
Real *d_hllc_vel;
Real *d_sumG, *d_sumP, *d_divU;

// GPU output
real_vector_t d_divF(VSIZE, NULL);

// 3D arrays (GPU input)
vector<hipArray_t> d_GPUin(VSIZE, NULL);

// Max SOS
int *h_maxSOS; // host, mapped
int *d_maxSOS; // device, mapped (different address)

// use non-null stream (async)
#define _NUM_STREAMS_ 2
hipStream_t *stream;

// events
#define _NUM_EVENTS_ 2
hipEvent_t *event_h2d;
hipEvent_t *event_d2h;


///////////////////////////////////////////////////////////////////////////////
// IMPLEMENTATION
///////////////////////////////////////////////////////////////////////////////
Profiler GPU::profiler; // combined CPU/GPU profiler

static void _h2d_3DArray(hipArray_t dst, const Real * const src, const int nslices, const int s_id)
{
    hipMemcpy3DParms copyParams = {0};
    copyParams.extent            = make_hipExtent(NodeBlock::sizeX, NodeBlock::sizeY, nslices);
    copyParams.kind              = hipMemcpyHostToDevice;
    copyParams.srcPtr            = make_hipPitchedPtr((void *)src, NodeBlock::sizeX * sizeof(Real), NodeBlock::sizeX, NodeBlock::sizeY);
    copyParams.dstArray          = dst;

    hipMemcpy3DAsync(&copyParams, stream[s_id]);
}


///////////////////////////////////////////////////////////////////////////
// GPU Memory alloc / dealloc
///////////////////////////////////////////////////////////////////////////
void GPU::alloc(void** sos, const uint_t nslices, const bool isroot)
{
#ifndef _MUTE_GPU_
    /* hipDeviceReset(); */
    /* hipSetDeviceFlags(hipDeviceMapHost); */

    // processing slice size (normal to z-direction)
    const uint_t SLICE_GPU = NodeBlock::sizeX * NodeBlock::sizeY;

    // GPU output size
    const uint_t outputSize = SLICE_GPU * nslices;

    // fluxes
    const uint_t xflxSize = (NodeBlock::sizeX+1)*NodeBlock::sizeY*nslices;
    const uint_t yflxSize = NodeBlock::sizeX*(NodeBlock::sizeY+1)*nslices;
    const uint_t zflxSize = NodeBlock::sizeX*NodeBlock::sizeY*(nslices+1);
    const uint_t maxflxSize = max(xflxSize, max(yflxSize, zflxSize));

    // x-/yghosts
    const uint_t xgSize = 3*NodeBlock::sizeY*nslices;
    const uint_t ygSize = NodeBlock::sizeX*3*nslices;

    // GPU allocation
    hipChannelFormatDesc fmt = hipCreateChannelDesc<Real>();
    for (int var = 0; var < VSIZE; ++var)
    {
        // fluxes
        hipMalloc(&d_flux[var], maxflxSize*sizeof(Real));

        // x-/yghosts
        hipMalloc(&d_xgl[var], xgSize*sizeof(Real));
        hipMalloc(&d_xgr[var], xgSize*sizeof(Real));

        hipMalloc(&d_ygl[var], ygSize*sizeof(Real));
        hipMalloc(&d_ygr[var], ygSize*sizeof(Real));

        // GPU output
        hipMalloc(&d_divF[var], outputSize*sizeof(Real));

        // GPU input (+6 slices for zghosts)
        hipMalloc3DArray(&d_GPUin[var], &fmt, make_hipExtent(NodeBlock::sizeX, NodeBlock::sizeY, nslices+6));
    }

    // extraterm for advection
    hipMalloc(&d_Gm, maxflxSize * sizeof(Real));
    hipMalloc(&d_Gp, maxflxSize * sizeof(Real));
    hipMalloc(&d_Pm, maxflxSize * sizeof(Real));
    hipMalloc(&d_Pp, maxflxSize * sizeof(Real));
    hipMalloc(&d_hllc_vel, maxflxSize * sizeof(Real));
    hipMalloc(&d_sumG, outputSize * sizeof(Real));
    hipMalloc(&d_sumP, outputSize * sizeof(Real));
    hipMalloc(&d_divU, outputSize * sizeof(Real));

    // zero-copy maxSOS (TODO: should this be unsigned int?)
    hipHostAlloc((void**)&h_maxSOS, sizeof(int), hipHostMallocMapped);
    hipHostGetDevicePointer(&d_maxSOS, h_maxSOS, 0);
    *(int**)sos = h_maxSOS; // return a reference to the caller

    // create streams
    stream = (hipStream_t *) malloc(_NUM_STREAMS_ * sizeof(hipStream_t));
    assert(stream != NULL);
    for (int i = 0 ; i < _NUM_STREAMS_; ++i)
        hipStreamCreate(&stream[i]);

    // create events
    event_h2d = (hipEvent_t *) malloc(_NUM_EVENTS_ * sizeof(hipEvent_t));
    event_d2h = (hipEvent_t *) malloc(_NUM_EVENTS_ * sizeof(hipEvent_t));
    assert(event_h2d != NULL);
    assert(event_d2h != NULL);
    for (int i = 0; i < _NUM_EVENTS_; ++i)
    {
        hipEventCreate(&event_h2d[i]);
        hipEventCreate(&event_d2h[i]);
    }

    // Stats
    if (isroot)
    {
        int dev;
        hipDeviceProp_t prop;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);

        printf("=====================================================================\n");
        printf("[GPU ALLOCATION FOR %s]\n", prop.name);
        printf("[%5.1f MB (input GPU)]\n", VSIZE*(SLICE_GPU*(nslices+6))*sizeof(Real) / 1024. / 1024);
        printf("[%5.1f MB (tmp)]\n", VSIZE*outputSize*sizeof(Real) / 1024. / 1024);
        printf("[%5.1f MB (rhs)]\n", VSIZE*outputSize*sizeof(Real) / 1024. / 1024);
        printf("[%5.1f MB (flux storage)]\n", VSIZE*(xflxSize + yflxSize + zflxSize)*sizeof(Real) / 1024. / 1024);
        printf("[%5.1f MB (x/yghosts)]\n", VSIZE*(xgSize + ygSize)*2*sizeof(Real) / 1024. / 1024);
        printf("[%5.1f MB (extraterm)]\n", (5*maxflxSize + 3*outputSize)*sizeof(Real) / 1024. / 1024);
        GPU::tell_memUsage_GPU();
        printf("=====================================================================\n");
    }
#endif
}


void GPU::dealloc(const bool isroot)
{
#ifndef _MUTE_GPU_
    for (int var = 0; var < VSIZE; ++var)
    {
        // fluxes
        hipFree(d_flux[var]);

        // x-/yghosts
        hipFree(d_xgl[var]);
        hipFree(d_xgr[var]);
        hipFree(d_ygl[var]);
        hipFree(d_ygr[var]);

        // GPU output
        hipFree(d_divF[var]);

        // input GPU
        hipFreeArray(d_GPUin[var]);
    }

    // extraterms
    hipFree(d_Gm);
    hipFree(d_Gp);
    hipFree(d_Pm);
    hipFree(d_Pp);
    hipFree(d_hllc_vel);
    hipFree(d_sumG);
    hipFree(d_sumP);
    hipFree(d_divU);

    // Max SOS
    hipHostFree(h_maxSOS);

    // destroy streams
    for (int i = 0; i < _NUM_STREAMS_; ++i)
        hipStreamDestroy(stream[i]);
    free(stream);

    // destroy events
    for (int i = 0; i < _NUM_EVENTS_; ++i)
    {
        hipEventDestroy(event_h2d[i]);
        hipEventDestroy(event_d2h[i]);
    }
    free(event_h2d);
    free(event_d2h);

    // Stats
    if (isroot)
    {
        int dev;
        hipDeviceProp_t prop;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);

        printf("=====================================================================\n");
        printf("[FREE GPU %s]\n", prop.name);
        GPU::tell_memUsage_GPU();
        printf("=====================================================================\n");
    }
#endif
}


///////////////////////////////////////////////////////////////////////////
// COMMUNICATION H2D / D2H
///////////////////////////////////////////////////////////////////////////
void GPU::upload_xy_ghosts(const uint_t Nxghost, const real_vector_t& xghost_l, const real_vector_t& xghost_r,
        const uint_t Nyghost, const real_vector_t& yghost_l, const real_vector_t& yghost_r, const int s_id)
{
#ifndef _MUTE_GPU_
    assert(0 <= s_id && s_id < _NUM_STREAMS_);

    // TODO: use larger arrays for ghosts to minimize API overhead +
    // increase BW performance
    GPU::profiler.push_startCUDA("SEND GHOSTS", &stream[s_id]);
    for (int i = 0; i < VSIZE; ++i)
    {
        // x
        hipMemcpyAsync(d_xgl[i], xghost_l[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
        hipMemcpyAsync(d_xgr[i], xghost_r[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
        // y
        hipMemcpyAsync(d_ygl[i], yghost_l[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
        hipMemcpyAsync(d_ygr[i], yghost_r[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
    }
    GPU::profiler.pop_stopCUDA();
#endif
}


void GPU::h2d_3DArray(const real_vector_t& src, const uint_t nslices, const int s_id)
{
#ifndef _MUTE_GPU_
    assert(0 <= s_id && s_id < _NUM_STREAMS_);
    GPU::profiler.push_startCUDA("SEND 3DARRAY", &stream[s_id]);
    for (int i = 0; i < VSIZE; ++i)
        _h2d_3DArray(d_GPUin[i], src[i], nslices, s_id);
    GPU::profiler.pop_stopCUDA();
    hipEventRecord(event_h2d[s_id], stream[s_id]);
#endif
}


void GPU::d2h_divF(real_vector_t& dst, const uint_t N, const int s_id)
{
#ifndef _MUTE_GPU_
    // download divF for current chunk
    GPU::profiler.push_startCUDA("RECV DIV(F)", &stream[s_id]);
    for (int i = 0; i < VSIZE; ++i)
        hipMemcpyAsync(dst[i], d_divF[i], N*sizeof(Real), hipMemcpyDeviceToHost, stream[s_id]);
    GPU::profiler.pop_stopCUDA();
    hipEventRecord(event_d2h[s_id], stream[s_id]);
#endif
}


///////////////////////////////////////////////////////////////////////////
// Sync
///////////////////////////////////////////////////////////////////////////
void GPU::wait_h2d(const int e_id)
{
#ifndef _MUTE_GPU_
    hipEventSynchronize(event_h2d[e_id]);
#endif
}


void GPU::wait_d2h(const int e_id)
{
#ifndef _MUTE_GPU_
    hipEventSynchronize(event_d2h[e_id]);
#endif
}


void GPU::syncGPU()
{
#ifndef _MUTE_GPU_
    hipDeviceSynchronize();
#endif
}


void GPU::syncStream(const int s_id)
{
#ifndef _MUTE_GPU_
    hipStreamSynchronize(stream[s_id]);
#endif
}


///////////////////////////////////////////////////////////////////////////
// Stats
///////////////////////////////////////////////////////////////////////////
void GPU::tell_memUsage_GPU()
{
#ifndef _MUTE_GPU_
    size_t free_byte, total_byte;
    const int status = hipMemGetInfo(&free_byte, &total_byte);
    if (hipSuccess != status)
    {
        printf("Whoot! Can not get memory stats from GPU...\n");
        return;
    }
    const size_t used = total_byte - free_byte;
    printf("GPU memory usage: free = %5.1f MB, total = %5.1f MB (%5.1f MB used)\n",
            (double)free_byte / 1024 / 1024,
            (double)total_byte / 1024 / 1024,
            (double)used / 1024 / 1024);
#endif
}


void GPU::tell_GPU()
{
#ifndef _MUTE_GPU_
    int dev;
    hipDeviceProp_t prop;
    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop, dev);
    printf("Using device %d (%s)\n", dev, prop.name);
#endif
}
