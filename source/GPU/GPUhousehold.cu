/* *
 * GPUhousehold.cu
 *
 * Created by Fabian Wermelinger on 6/24/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */
#include <stdio.h>
#include <vector>

#include "CUDA_Timer.cuh"
#include "NodeBlock.h"
#include "GPU.h"

enum { VSIZE = NodeBlock::NVAR };

///////////////////////////////////////////////////////////////////////////////
// GLOBAL VARIABLES
///////////////////////////////////////////////////////////////////////////////
RealPtrVec_t d_tmp(VSIZE, NULL);
RealPtrVec_t d_rhs(VSIZE, NULL);
RealPtrVec_t d_xgl(VSIZE, NULL);
RealPtrVec_t d_xgr(VSIZE, NULL);
RealPtrVec_t d_ygl(VSIZE, NULL);
RealPtrVec_t d_ygr(VSIZE, NULL);

RealPtrVec_t d_xflux(VSIZE, NULL);
RealPtrVec_t d_yflux(VSIZE, NULL);
RealPtrVec_t d_zflux(VSIZE, NULL);

// extraterms for advection equations
Real *d_Gm, *d_Gp;
Real *d_Pm, *d_Pp;
Real *d_hllc_vel;
Real *d_sumG, *d_sumP, *d_divU;

// 3D arrays
std::vector<hipArray_t> d_SOA(VSIZE, NULL);

// Max SOS
int* h_maxSOS; // host, mapped
int* d_maxSOS; // device, mapped (different address)

// use non-null stream (async)
hipStream_t stream1;
hipStream_t stream2;

// events
hipEvent_t divergence_completed;
hipEvent_t h2d_3Darray_completed;
hipEvent_t h2d_tmp_completed;
hipEvent_t d2h_rhs_completed;
hipEvent_t d2h_tmp_completed;


///////////////////////////////////////////////////////////////////////////////
// IMPLEMENTATION
///////////////////////////////////////////////////////////////////////////////
static void _h2d_3DArray(hipArray_t dst, const Real * const src, const int NX, const int NY, const int NZ)
{
    hipMemcpy3DParms copyParams = {0};
    copyParams.extent            = make_hipExtent(NX, NY, NZ);
    copyParams.kind              = hipMemcpyHostToDevice;
    copyParams.srcPtr            = make_hipPitchedPtr((void *)src, NX * sizeof(Real), NX, NY);
    copyParams.dstArray          = dst;

    hipMemcpy3DAsync(&copyParams, stream1);
}


extern "C"
{
    ///////////////////////////////////////////////////////////////////////////
    // GPU Memory alloc / dealloc
    ///////////////////////////////////////////////////////////////////////////
    void GPU::alloc(void** sos, const uint_t BSX_GPU, const uint_t BSY_GPU, const uint_t BSZ_GPU, const uint_t CHUNK_WIDTH)
    {
        // THE FOLLOWING ASSUMES CUBIC DOAMIN
        const uint_t SLICE_GPU = BSX_GPU * BSY_GPU;

        // GPU output size
        const uint_t outputSize = SLICE_GPU * CHUNK_WIDTH;

        // Fluxes (use one array later, process after each flux dimension)
        const uint_t bSflx = (BSX_GPU+1)*BSY_GPU*CHUNK_WIDTH;
        const uint_t bSfly = BSX_GPU*(BSY_GPU+1)*CHUNK_WIDTH;
        const uint_t bSflz = BSX_GPU*BSY_GPU*(CHUNK_WIDTH+1);

        // Ghosts
        /* const uint_t xgSize = 3*BSY_GPU*BSZ_GPU; */
        /* const uint_t ygSize = BSX_GPU*3*BSZ_GPU; */
        const uint_t xgSize = 3 * SLICE_GPU;
        const uint_t ygSize = 3 * SLICE_GPU;

        // Allocate
        hipChannelFormatDesc fmt =  hipCreateChannelDesc<Real>();
        for (int var = 0; var < VSIZE; ++var)
        {
            //tmp
            hipMalloc(&d_tmp[var], outputSize*sizeof(Real));
            hipMemset(d_tmp[var], 0, outputSize*sizeof(Real));

            // rhs
            hipMalloc(&d_rhs[var], outputSize*sizeof(Real));
            hipMemset(d_rhs[var], 0, outputSize*sizeof(Real));

            // fluxes
            hipMalloc(&d_xflux[var], bSflx*sizeof(Real));
            hipMalloc(&d_yflux[var], bSfly*sizeof(Real));
            hipMalloc(&d_zflux[var], bSflz*sizeof(Real));
            hipMemset(d_xflux[var], 0, bSflx*sizeof(Real));
            hipMemset(d_yflux[var], 0, bSfly*sizeof(Real));
            hipMemset(d_zflux[var], 0, bSflz*sizeof(Real));

            // ghosts
            hipMalloc(&d_xgl[var], xgSize*sizeof(Real));
            hipMalloc(&d_xgr[var], xgSize*sizeof(Real));
            hipMalloc(&d_ygl[var], ygSize*sizeof(Real));
            hipMalloc(&d_ygr[var], ygSize*sizeof(Real));

            // GPU input SOA
            hipMalloc3DArray(&d_SOA[var], &fmt, make_hipExtent(BSX_GPU, BSY_GPU, CHUNK_WIDTH+6));
        }

        // extraterm for advection
        hipMalloc(&d_Gm, bSflz * sizeof(Real));
        hipMalloc(&d_Gp, bSflz * sizeof(Real));
        hipMalloc(&d_Pm, bSflz * sizeof(Real));
        hipMalloc(&d_Pp, bSflz * sizeof(Real));
        hipMalloc(&d_hllc_vel, bSflz * sizeof(Real));
        hipMalloc(&d_sumG, outputSize * sizeof(Real));
        hipMalloc(&d_sumP, outputSize * sizeof(Real));
        hipMalloc(&d_divU, outputSize * sizeof(Real));

        // zero-copy maxSOS
        hipSetDeviceFlags(hipDeviceMapHost);
        hipHostAlloc((void**)&h_maxSOS, sizeof(int), hipHostMallocMapped);
        hipHostGetDevicePointer(&d_maxSOS, h_maxSOS, 0);
        *(int**)sos = h_maxSOS; // return a reference to the caller

        // create stream
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);

        // create event
        hipEventCreate(&divergence_completed);
        hipEventCreate(&h2d_3Darray_completed);
        hipEventCreate(&h2d_tmp_completed);
        hipEventCreate(&d2h_rhs_completed);
        hipEventCreate(&d2h_tmp_completed);

        // Stats
        int dev;
        hipDeviceProp_t prop;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);

        printf("=====================================================================\n");
        printf("[GPU ALLOCATION FOR %s]\n", prop.name);
        printf("[%5.1f MB (input SOA)]\n", VSIZE*(SLICE_GPU*(CHUNK_WIDTH+6))*sizeof(Real) / 1024. / 1024);
        printf("[%5.1f MB (tmp)]\n", VSIZE*outputSize*sizeof(Real) / 1024. / 1024);
        printf("[%5.1f MB (rhs)]\n", VSIZE*outputSize*sizeof(Real) / 1024. / 1024);
        printf("[%5.1f MB (flux storage)]\n", VSIZE*(bSflx + bSfly + bSflz)*sizeof(Real) / 1024. / 1024);
        printf("[%5.1f MB (x/yghosts)]\n", VSIZE*(xgSize + ygSize)*2*sizeof(Real) / 1024. / 1024);
        printf("[%5.1f MB (extraterm)]\n", (5*bSflx + 3*outputSize)*sizeof(Real) / 1024. / 1024);
        GPU::tell_memUsage_GPU();
        printf("=====================================================================\n");
    }


    void GPU::dealloc()
    {
        for (int var = 0; var < VSIZE; ++var)
        {
            // tmp
            hipFree(d_tmp[var]);

            // rhs
            hipFree(d_rhs[var]);

            // fluxes
            hipFree(d_xflux[var]);
            hipFree(d_yflux[var]);
            hipFree(d_zflux[var]);

            // ghosts
            hipFree(d_xgl[var]);
            hipFree(d_xgr[var]);
            hipFree(d_ygl[var]);
            hipFree(d_ygr[var]);

            // input SOA
            hipFreeArray(d_SOA[var]);
        }

        // extraterms
        hipFree(d_Gm);
        hipFree(d_Gp);
        hipFree(d_Pm);
        hipFree(d_Pp);
        hipFree(d_hllc_vel);
        hipFree(d_sumG);
        hipFree(d_sumP);
        hipFree(d_divU);

        // Max SOS
        hipHostFree(h_maxSOS);

        // destroy stream
        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);

        // destroy events
        hipEventDestroy(divergence_completed);
        hipEventDestroy(h2d_3Darray_completed);
        hipEventDestroy(h2d_tmp_completed);
        hipEventDestroy(d2h_rhs_completed);
        hipEventDestroy(d2h_tmp_completed);

        // Stats
        int dev;
        hipDeviceProp_t prop;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);

        printf("=====================================================================\n");
        printf("[FREE GPU %s]\n", prop.name);
        GPU::tell_memUsage_GPU();
        printf("=====================================================================\n");
    }


    ///////////////////////////////////////////////////////////////////////////
    // H2D / D2H
    ///////////////////////////////////////////////////////////////////////////
    /* void GPU::upload_ghosts(const uint_t Nghost, */
    /*         const Real* const xghost_L, const Real* const xghost_R, */
    /*         const Real* const yghost_L, const Real* const yghost_R) */
    /* { */
    /*     for (int i = 0; i < VSIZE; ++i) */
    /*     { */
    /*         hipMemcpyAsync(d_xgl[i], &xghost_L[i*Nghost], Nghost*sizeof(Real), hipMemcpyHostToDevice, stream1); */
    /*         hipMemcpyAsync(d_xgr[i], &xghost_R[i*Nghost], Nghost*sizeof(Real), hipMemcpyHostToDevice, stream1); */
    /*         hipMemcpyAsync(d_ygl[i], &yghost_L[i*Nghost], Nghost*sizeof(Real), hipMemcpyHostToDevice, stream1); */
    /*         hipMemcpyAsync(d_ygr[i], &yghost_R[i*Nghost], Nghost*sizeof(Real), hipMemcpyHostToDevice, stream1); */
    /*     } */
    /* } */


    void GPU::upload_xy_ghosts(const uint_t Nxghost, const RealPtrVec_t& xghost_l, const RealPtrVec_t& xghost_r,
            const uint_t Nyghost, const RealPtrVec_t& yghost_l, const RealPtrVec_t& yghost_r)
    {
        for (int i = 0; i < VSIZE; ++i)
        {
            // x
            hipMemcpyAsync(d_xgl[i], xghost_l[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
            hipMemcpyAsync(d_xgr[i], xghost_r[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
            // y
            hipMemcpyAsync(d_ygl[i], yghost_l[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
            hipMemcpyAsync(d_ygr[i], yghost_r[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
        }
    }


    void GPU::h2d_3DArray(const RealPtrVec_t& src, const uint_t NX, const uint_t NY, const uint_t NZ)
    {
        GPUtimer upload;
        upload.start(stream1);
        for (int i = 0; i < VSIZE; ++i)
            _h2d_3DArray(d_SOA[i], src[i], NX, NY, NZ);
        upload.stop(stream1);
        upload.print("[GPU UPLOAD 3DArray]: ");

        hipEventRecord(h2d_3Darray_completed, stream1);
    }


    void GPU::h2d_tmp(const RealPtrVec_t& src, const uint_t N)
    {
        hipStreamWaitEvent(stream2, h2d_3Darray_completed, 0);

        GPUtimer upload;
        upload.start(stream2);
        for (int i = 0; i < VSIZE; ++i)
            hipMemcpyAsync(d_tmp[i], src[i], N*sizeof(Real), hipMemcpyHostToDevice, stream2);
        upload.stop(stream2);
        upload.print("[GPU UPLOAD TMP]: ");

        hipEventRecord(h2d_tmp_completed, stream2);
    }


    void GPU::d2h_rhs(RealPtrVec_t& dst, const uint_t N)
    {
        hipStreamWaitEvent(stream2, divergence_completed, 0);

        // copy content of d_rhs to host, using the stream2 (after divergence)
        GPUtimer download;
        download.start(stream2);
        for (int i = 0; i < VSIZE; ++i)
            hipMemcpyAsync(dst[i], d_rhs[i], N*sizeof(Real), hipMemcpyDeviceToHost, stream2);
        download.stop(stream2);
        download.print("[GPU DOWNLOAD RHS]: ");

        hipEventRecord(d2h_rhs_completed, stream2);
    }


    void GPU::d2h_tmp(RealPtrVec_t& dst, const uint_t N)
    {
        /* // wait until the device to host copy of the rhs has finished. This will */
        /* // hide the SOA to AOS conversion of the RHS data on the host, while the */
        /* // updated solution is copied to the host. */
        /* hipStreamWaitEvent(stream1, d2h_rhs_completed, 0); */

        // copy content of d_tmp to host, using the stream1
        GPUtimer download;
        download.start(stream1);
        for (int i = 0; i < VSIZE; ++i)
            hipMemcpyAsync(dst[i], d_tmp[i], N*sizeof(Real), hipMemcpyDeviceToHost, stream2);
        download.stop(stream1);
        download.print("[GPU DOWNLOAD TMP]: ");

        hipEventRecord(d2h_tmp_completed, stream2);
    }


    ///////////////////////////////////////////////////////////////////////////
    // Sync
    ///////////////////////////////////////////////////////////////////////////
    void GPU::h2d_3DArray_wait()
    {
        // wait until h2d_3DArray has finished
        hipEventSynchronize(h2d_3Darray_completed);
    }


    void GPU::d2h_rhs_wait()
    {
        // wait until d2h_rhs has finished
        hipEventSynchronize(d2h_rhs_completed);
    }


    void GPU::d2h_tmp_wait()
    {
        // wait until d2h_tmp has finished
        hipEventSynchronize(d2h_tmp_completed);
    }


    void GPU::syncGPU()
    {
        hipDeviceSynchronize();
    }


    void GPU::syncStream(streamID s)
    {
        switch (s)
        {
            case S1: hipStreamSynchronize(stream1); break;
            case S2: hipStreamSynchronize(stream2); break;
        }
    }


    ///////////////////////////////////////////////////////////////////////////
    // Stats
    ///////////////////////////////////////////////////////////////////////////
    void GPU::tell_memUsage_GPU()
    {
        size_t free_byte, total_byte;
        const int status = hipMemGetInfo(&free_byte, &total_byte);
        if (hipSuccess != status)
        {
            printf("Hoppla! Can not get memory stats from GPU...\n");
            return;
        }
        const size_t used = total_byte - free_byte;
        printf("GPU memory usage: free = %5.1f MB, total = %5.1f MB (%5.1f MB used)\n",
                (double)free_byte / 1024 / 1024,
                (double)total_byte / 1024 / 1024,
                (double)used / 1024 / 1024);
    }


    void GPU::tell_GPU()
    {
        int dev;
        hipDeviceProp_t prop;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);
        printf("Using device %d (%s)\n", dev, prop.name);
    }
}
