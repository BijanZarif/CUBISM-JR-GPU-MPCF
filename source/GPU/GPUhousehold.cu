/* *
 * GPUhousehold.cu
 *
 * Created by Fabian Wermelinger on 6/24/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */
#include <stdio.h>
#include <vector>

#include "GPU.h" // includes Types.h

#ifdef _CUDA_TIMER_
#include "CUDA_Timer.cuh"
#endif


enum { VSIZE = NodeBlock::NVAR };

///////////////////////////////////////////////////////////////////////////////
// GLOBAL VARIABLES
///////////////////////////////////////////////////////////////////////////////
RealPtrVec_t d_tmp(VSIZE, NULL);
RealPtrVec_t d_rhs(VSIZE, NULL);
RealPtrVec_t d_xgl(VSIZE, NULL);
RealPtrVec_t d_xgr(VSIZE, NULL);
RealPtrVec_t d_ygl(VSIZE, NULL);
RealPtrVec_t d_ygr(VSIZE, NULL);

RealPtrVec_t d_xflux(VSIZE, NULL);
RealPtrVec_t d_yflux(VSIZE, NULL);
RealPtrVec_t d_zflux(VSIZE, NULL);

// extraterms for advection equations
Real *d_Gm, *d_Gp;
Real *d_Pm, *d_Pp;
Real *d_hllc_vel;
Real *d_sumG, *d_sumP, *d_divU;

// 3D arrays (GPU input)
std::vector<hipArray_t> d_SOAin(VSIZE, NULL);

// Max SOS
int *h_maxSOS; // host, mapped
int *d_maxSOS; // device, mapped (different address)

// use non-null stream (async)
hipStream_t stream1;
hipStream_t stream2;
hipStream_t stream3;

// events
hipEvent_t divergence_completed;
hipEvent_t update_completed;
hipEvent_t h2d_3Darray_completed;
hipEvent_t h2d_tmp_completed;
hipEvent_t d2h_rhs_completed;
hipEvent_t d2h_tmp_completed;


///////////////////////////////////////////////////////////////////////////////
// IMPLEMENTATION
///////////////////////////////////////////////////////////////////////////////
static void _h2d_3DArray(hipArray_t dst, const Real * const src, const int NX, const int NY, const int NZ)
{
    hipMemcpy3DParms copyParams = {0};
    copyParams.extent            = make_hipExtent(NX, NY, NZ);
    copyParams.kind              = hipMemcpyHostToDevice;
    copyParams.srcPtr            = make_hipPitchedPtr((void *)src, NX * sizeof(Real), NX, NY);
    copyParams.dstArray          = dst;

    hipMemcpy3DAsync(&copyParams, stream1);
}


extern "C"
{
    ///////////////////////////////////////////////////////////////////////////
    // GPU Memory alloc / dealloc
    ///////////////////////////////////////////////////////////////////////////
    void GPU::alloc(void** sos, const uint_t BSX_GPU, const uint_t BSY_GPU, const uint_t BSZ_GPU, const uint_t CHUNK_WIDTH, const bool isroot)
    {
#ifndef _MUTE_GPU_

        // processing slice size (normal to z-direction)
        const uint_t SLICE_GPU = BSX_GPU * BSY_GPU;

        // GPU output size
        const uint_t outputSize = SLICE_GPU * CHUNK_WIDTH;

        // fluxes (TODO: use one array later, process after each flux computation)
        const uint_t bSflx = (BSX_GPU+1)*BSY_GPU*CHUNK_WIDTH;
        const uint_t bSfly = BSX_GPU*(BSY_GPU+1)*CHUNK_WIDTH;
        const uint_t bSflz = BSX_GPU*BSY_GPU*(CHUNK_WIDTH+1);

        // x-/yghosts
        const uint_t xgSize = 3*BSY_GPU*CHUNK_WIDTH;
        const uint_t ygSize = BSX_GPU*3*CHUNK_WIDTH;

        // GPU allocation
        hipChannelFormatDesc fmt = hipCreateChannelDesc<Real>();
        for (int var = 0; var < VSIZE; ++var)
        {
            //tmp
            hipMalloc(&d_tmp[var], outputSize*sizeof(Real));
            hipMemset(d_tmp[var], 0, outputSize*sizeof(Real));

            // rhs
            hipMalloc(&d_rhs[var], outputSize*sizeof(Real));
            hipMemset(d_rhs[var], 0, outputSize*sizeof(Real));

            // fluxes
            hipMalloc(&d_xflux[var], bSflx*sizeof(Real));
            hipMalloc(&d_yflux[var], bSfly*sizeof(Real));
            hipMalloc(&d_zflux[var], bSflz*sizeof(Real));
            hipMemset(d_xflux[var], 0, bSflx*sizeof(Real));
            hipMemset(d_yflux[var], 0, bSfly*sizeof(Real));
            hipMemset(d_zflux[var], 0, bSflz*sizeof(Real));

            // x-/yghosts
            hipMalloc(&d_xgl[var], xgSize*sizeof(Real));
            hipMalloc(&d_xgr[var], xgSize*sizeof(Real));

            hipMalloc(&d_ygl[var], ygSize*sizeof(Real));
            hipMalloc(&d_ygr[var], ygSize*sizeof(Real));

            // GPU input SOA (+6 slices for zghosts)
            hipMalloc3DArray(&d_SOAin[var], &fmt, make_hipExtent(BSX_GPU, BSY_GPU, CHUNK_WIDTH+6));
        }

        // extraterm for advection (TODO: remove this uglyness!)
        hipMalloc(&d_Gm, bSflz * sizeof(Real));
        hipMalloc(&d_Gp, bSflz * sizeof(Real));
        hipMalloc(&d_Pm, bSflz * sizeof(Real));
        hipMalloc(&d_Pp, bSflz * sizeof(Real));
        hipMalloc(&d_hllc_vel, bSflz * sizeof(Real));
        hipMalloc(&d_sumG, outputSize * sizeof(Real));
        hipMalloc(&d_sumP, outputSize * sizeof(Real));
        hipMalloc(&d_divU, outputSize * sizeof(Real));

        // zero-copy maxSOS (TODO: should this be unsigned int??)
        hipSetDeviceFlags(hipDeviceMapHost);
        hipHostAlloc((void**)&h_maxSOS, sizeof(int), hipHostMallocMapped);
        hipHostGetDevicePointer(&d_maxSOS, h_maxSOS, 0);
        *(int**)sos = h_maxSOS; // return a reference to the caller

        // create streams
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);

        // create events
        hipEventCreate(&divergence_completed);
        hipEventCreate(&update_completed);
        hipEventCreate(&h2d_3Darray_completed);
        hipEventCreate(&h2d_tmp_completed);
        hipEventCreate(&d2h_rhs_completed);
        hipEventCreate(&d2h_tmp_completed);

        // Stats
        if (isroot)
        {
            int dev;
            hipDeviceProp_t prop;
            hipGetDevice(&dev);
            hipGetDeviceProperties(&prop, dev);

            printf("=====================================================================\n");
            printf("[GPU ALLOCATION FOR %s]\n", prop.name);
            printf("[%5.1f MB (input SOA)]\n", VSIZE*(SLICE_GPU*(CHUNK_WIDTH+6))*sizeof(Real) / 1024. / 1024);
            printf("[%5.1f MB (tmp)]\n", VSIZE*outputSize*sizeof(Real) / 1024. / 1024);
            printf("[%5.1f MB (rhs)]\n", VSIZE*outputSize*sizeof(Real) / 1024. / 1024);
            printf("[%5.1f MB (flux storage)]\n", VSIZE*(bSflx + bSfly + bSflz)*sizeof(Real) / 1024. / 1024);
            printf("[%5.1f MB (x/yghosts)]\n", VSIZE*(xgSize + ygSize)*2*sizeof(Real) / 1024. / 1024);
            printf("[%5.1f MB (extraterm)]\n", (5*bSflx + 3*outputSize)*sizeof(Real) / 1024. / 1024);
            GPU::tell_memUsage_GPU();
            printf("=====================================================================\n");
        }
#endif
    }


    void GPU::dealloc(const bool isroot)
    {
#ifndef _MUTE_GPU_
        for (int var = 0; var < VSIZE; ++var)
        {
            // tmp
            hipFree(d_tmp[var]);

            // rhs
            hipFree(d_rhs[var]);

            // fluxes
            hipFree(d_xflux[var]);
            hipFree(d_yflux[var]);
            hipFree(d_zflux[var]);

            // x-/yghosts
            hipFree(d_xgl[var]);
            hipFree(d_xgr[var]);
            hipFree(d_ygl[var]);
            hipFree(d_ygr[var]);

            // input SOA
            hipFreeArray(d_SOAin[var]);
        }

        // extraterms
        hipFree(d_Gm);
        hipFree(d_Gp);
        hipFree(d_Pm);
        hipFree(d_Pp);
        hipFree(d_hllc_vel);
        hipFree(d_sumG);
        hipFree(d_sumP);
        hipFree(d_divU);

        // Max SOS
        hipHostFree(h_maxSOS);

        // destroy streams
        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);
        hipStreamDestroy(stream3);

        // destroy events
        hipEventDestroy(divergence_completed);
        hipEventDestroy(update_completed);
        hipEventDestroy(h2d_3Darray_completed);
        hipEventDestroy(h2d_tmp_completed);
        hipEventDestroy(d2h_rhs_completed);
        hipEventDestroy(d2h_tmp_completed);

        // Stats
        if (isroot)
        {
            int dev;
            hipDeviceProp_t prop;
            hipGetDevice(&dev);
            hipGetDeviceProperties(&prop, dev);

            printf("=====================================================================\n");
            printf("[FREE GPU %s]\n", prop.name);
            GPU::tell_memUsage_GPU();
            printf("=====================================================================\n");
        }
#endif
    }


    ///////////////////////////////////////////////////////////////////////////
    // H2D / D2H
    ///////////////////////////////////////////////////////////////////////////
    void GPU::upload_xy_ghosts(const uint_t Nxghost, const RealPtrVec_t& xghost_l, const RealPtrVec_t& xghost_r,
            const uint_t Nyghost, const RealPtrVec_t& yghost_l, const RealPtrVec_t& yghost_r)
    {
#ifndef _MUTE_GPU_
        // TODO: use larger arrays for ghosts to minimize API overhead +
        // increase BW performance
        tCUDA_START(stream1)
        for (int i = 0; i < VSIZE; ++i)
        {
            // x
            hipMemcpyAsync(d_xgl[i], xghost_l[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
            hipMemcpyAsync(d_xgr[i], xghost_r[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
            // y
            hipMemcpyAsync(d_ygl[i], yghost_l[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
            hipMemcpyAsync(d_ygr[i], yghost_r[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream1);
        }
        tCUDA_STOP(stream1, "[GPU UPLOAD X/YGHOSTS]: ")
#endif
    }


    void GPU::h2d_3DArray(const RealPtrVec_t& src, const uint_t NX, const uint_t NY, const uint_t NZ)
    {
#ifndef _MUTE_GPU_
        tCUDA_START(stream1)
        for (int i = 0; i < VSIZE; ++i)
            _h2d_3DArray(d_SOAin[i], src[i], NX, NY, NZ);
        tCUDA_STOP(stream1, "[GPU UPLOAD 3DArray]: ")
        hipEventRecord(h2d_3Darray_completed, stream1);
#endif
    }


    void GPU::h2d_tmp(const RealPtrVec_t& src, const uint_t N)
    {
#ifndef _MUTE_GPU_
        hipStreamWaitEvent(stream3, h2d_3Darray_completed, 0);

        tCUDA_START(stream3)
        for (int i = 0; i < VSIZE; ++i)
            hipMemcpyAsync(d_tmp[i], src[i], N*sizeof(Real), hipMemcpyHostToDevice, stream3);
        tCUDA_STOP(stream3, "[GPU UPLOAD TMP]: ")
        hipEventRecord(h2d_tmp_completed, stream3);
#endif
    }


    void GPU::d2h_rhs(RealPtrVec_t& dst, const uint_t N)
    {
#ifndef _MUTE_GPU_
        hipStreamWaitEvent(stream2, divergence_completed, 0);

        // copy content of d_rhs to host, using the stream2 (after divergence)
        tCUDA_START(stream2)
        for (int i = 0; i < VSIZE; ++i)
            hipMemcpyAsync(dst[i], d_rhs[i], N*sizeof(Real), hipMemcpyDeviceToHost, stream2);
        tCUDA_STOP(stream2, "[GPU DOWNLOAD RHS]: ")
        hipEventRecord(d2h_rhs_completed, stream2);
#endif
    }


    void GPU::d2h_tmp(RealPtrVec_t& dst, const uint_t N)
    {
#ifndef _MUTE_GPU_
        hipStreamWaitEvent(stream2, update_completed, 0);

        // copy content of d_tmp to host, using the stream1
        tCUDA_START(stream1)
        for (int i = 0; i < VSIZE; ++i)
            hipMemcpyAsync(dst[i], d_tmp[i], N*sizeof(Real), hipMemcpyDeviceToHost, stream2);
        tCUDA_STOP(stream1, "[GPU DOWNLOAD TMP]: ")
        hipEventRecord(d2h_tmp_completed, stream2);
#endif
    }


    ///////////////////////////////////////////////////////////////////////////
    // Sync
    ///////////////////////////////////////////////////////////////////////////
    void GPU::h2d_3DArray_wait()
    {
#ifndef _MUTE_GPU_
        // wait until h2d_3DArray has finished
        hipEventSynchronize(h2d_3Darray_completed);
#endif
    }


    void GPU::d2h_rhs_wait()
    {
#ifndef _MUTE_GPU_
        // wait until d2h_rhs has finished
        hipEventSynchronize(d2h_rhs_completed);
#endif
    }


    void GPU::d2h_tmp_wait()
    {
#ifndef _MUTE_GPU_
        // wait until d2h_tmp has finished
        hipEventSynchronize(d2h_tmp_completed);
#endif
    }


    void GPU::syncGPU()
    {
#ifndef _MUTE_GPU_
        hipDeviceSynchronize();
#endif
    }


    void GPU::syncStream(streamID s)
    {
#ifndef _MUTE_GPU_
        switch (s)
        {
            case S1: hipStreamSynchronize(stream1); break;
            case S2: hipStreamSynchronize(stream2); break;
        }
#endif
    }


    ///////////////////////////////////////////////////////////////////////////
    // Stats
    ///////////////////////////////////////////////////////////////////////////
    void GPU::tell_memUsage_GPU()
    {
#ifndef _MUTE_GPU_
        size_t free_byte, total_byte;
        const int status = hipMemGetInfo(&free_byte, &total_byte);
        if (hipSuccess != status)
        {
            printf("Hoppla! Can not get memory stats from GPU...\n");
            return;
        }
        const size_t used = total_byte - free_byte;
        printf("GPU memory usage: free = %5.1f MB, total = %5.1f MB (%5.1f MB used)\n",
                (double)free_byte / 1024 / 1024,
                (double)total_byte / 1024 / 1024,
                (double)used / 1024 / 1024);
#endif
    }


    void GPU::tell_GPU()
    {
#ifndef _MUTE_GPU_
        int dev;
        hipDeviceProp_t prop;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);
        printf("Using device %d (%s)\n", dev, prop.name);
#endif
    }
}
