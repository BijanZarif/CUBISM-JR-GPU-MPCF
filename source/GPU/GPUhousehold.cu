#include "hip/hip_runtime.h"
/* *
 * GPUhousehold.cu
 *
 * Created by Fabian Wermelinger on 6/24/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */
#include "GPU.cuh"

#include <stdio.h>
#include <algorithm>

///////////////////////////////////////////////////////////////////////////////
// GLOBAL VARIABLES
///////////////////////////////////////////////////////////////////////////////
// flux storage (COMPUTE stream)
real_vector_t d_flux(VSIZE, NULL);

// extraterms for advection equations (COMPUTE stream)
Real *d_Gm, *d_Gp;
Real *d_Pm, *d_Pp;
Real *d_hllc_vel;
Real *d_sumG, *d_sumP, *d_divU;

// Max SOS
int *h_maxSOS; // host, mapped
int *d_maxSOS; // device, mapped (different address)

struct GPU_COMM gpu_comm[_NUM_GPU_BUF_];

// use non-null stream (async)
hipStream_t *stream;

// events
hipEvent_t *event_h2d;
hipEvent_t *event_d2h;
hipEvent_t *event_compute;


///////////////////////////////////////////////////////////////////////////////
// IMPLEMENTATION
///////////////////////////////////////////////////////////////////////////////
Profiler GPU::profiler; // combined CPU/GPU profiler

static void _h2d_3DArray(hipArray_t dst, const Real * const src, const int nslices, const int s_id)
{
    hipMemcpy3DParms copyParams = {0};
    copyParams.extent            = make_hipExtent(NX, NY, nslices);
    copyParams.kind              = hipMemcpyHostToDevice;
    copyParams.srcPtr            = make_hipPitchedPtr((void *)src, NX * sizeof(Real), NX, NY);
    copyParams.dstArray          = dst;

    hipMemcpy3DAsync(&copyParams, stream[s_id]);
}


///////////////////////////////////////////////////////////////////////////
// GPU Memory alloc / dealloc
///////////////////////////////////////////////////////////////////////////
void GPU::alloc(void** sos, const uint_t nslices, const bool isroot)
{
#ifndef _MUTE_GPU_
    /* hipDeviceReset(); */
    /* hipSetDeviceFlags(hipDeviceMapHost); */

    // processing slice size (normal to z-direction)
    const uint_t SLICE_GPU = NodeBlock::sizeX * NodeBlock::sizeY;

    // GPU output size
    const uint_t outputSize = SLICE_GPU * nslices;

    // fluxes
    const uint_t xflxSize = (NodeBlock::sizeX+1)*NodeBlock::sizeY*nslices;
    const uint_t yflxSize = NodeBlock::sizeX*(NodeBlock::sizeY+1)*nslices;
    const uint_t zflxSize = NodeBlock::sizeX*NodeBlock::sizeY*(nslices+1);
    const uint_t maxflxSize = max(xflxSize, max(yflxSize, zflxSize));

    // x-/yghosts
    const uint_t xgSize = 3*NodeBlock::sizeY*nslices;
    const uint_t ygSize = NodeBlock::sizeX*3*nslices;

    // GPU allocation
    // Flux storage
    size_t computational_bytes = 0;
    for (int var = 0; var < VSIZE; ++var)
    {
        hipMalloc(&d_flux[var], maxflxSize*sizeof(Real));
        computational_bytes += maxflxSize * sizeof(Real);
    }

    // extraterm for advection
    hipMalloc(&d_Gm, maxflxSize * sizeof(Real));
    hipMalloc(&d_Gp, maxflxSize * sizeof(Real));
    hipMalloc(&d_Pm, maxflxSize * sizeof(Real));
    hipMalloc(&d_Pp, maxflxSize * sizeof(Real));
    hipMalloc(&d_hllc_vel, maxflxSize * sizeof(Real));
    computational_bytes += 5 * maxflxSize * sizeof(Real);
    hipMalloc(&d_sumG, outputSize * sizeof(Real));
    hipMalloc(&d_sumP, outputSize * sizeof(Real));
    hipMalloc(&d_divU, outputSize * sizeof(Real));
    computational_bytes += 3 * outputSize * sizeof(Real);

    // Communication buffers
    size_t ghost_bytes  = 0;
    size_t input_bytes  = 0;
    size_t output_bytes = 0;
    hipChannelFormatDesc fmt = hipCreateChannelDesc<Real>();
    for (int i = 0; i < _NUM_GPU_BUF_; ++i)
    {
        GPU_COMM * const mybuf = &gpu_comm[i];
        for (int var = 0; var < VSIZE; ++var)
        {
            // x-/yghosts
            hipMalloc(&(mybuf->d_xgl[var]), xgSize*sizeof(Real));
            hipMalloc(&(mybuf->d_xgr[var]), xgSize*sizeof(Real));
            hipMalloc(&(mybuf->d_ygl[var]), ygSize*sizeof(Real));
            hipMalloc(&(mybuf->d_ygr[var]), ygSize*sizeof(Real));
            ghost_bytes += 2 * xgSize * sizeof(Real) + 2 * ygSize * sizeof(Real);

            // GPU output
            hipMalloc(&(mybuf->d_divF[var]), outputSize*sizeof(Real));
            output_bytes += outputSize * sizeof(Real);

            // GPU input (+6 slices for zghosts)
            hipMalloc3DArray(&(mybuf->d_GPUin[var]), &fmt, make_hipExtent(NX, NY, nslices+6));
            input_bytes += NX * NY * (nslices+6) * sizeof(Real);
        }
    }

    // zero-copy maxSOS (TODO: should this be unsigned int?)
    hipHostAlloc((void**)&h_maxSOS, sizeof(int), hipHostMallocMapped);
    computational_bytes += sizeof(int);
    hipHostGetDevicePointer(&d_maxSOS, h_maxSOS, 0);
    *(int**)sos = h_maxSOS; // return a reference to the caller

    // create streams
    stream = (hipStream_t *) malloc(_NUM_STREAMS_ * sizeof(hipStream_t));
    assert(stream != NULL);
    for (int i = 0 ; i < _NUM_STREAMS_; ++i)
        hipStreamCreate(&stream[i]);

    // create events
    event_h2d     = (hipEvent_t *) malloc(_NUM_STREAMS_ * sizeof(hipEvent_t));
    event_d2h     = (hipEvent_t *) malloc(_NUM_STREAMS_ * sizeof(hipEvent_t));
    event_compute = (hipEvent_t *) malloc(_NUM_STREAMS_ * sizeof(hipEvent_t));
    assert(event_h2d != NULL);
    assert(event_d2h != NULL);
    assert(event_compute != NULL);
    for (int i = 0; i < _NUM_STREAMS_; ++i)
    {
        hipEventCreate(&event_h2d[i]);
        hipEventCreate(&event_d2h[i]);
        hipEventCreate(&event_compute[i]);
    }

    // Stats
    if (isroot)
    {
        int dev;
        hipDeviceProp_t prop;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);

        printf("=====================================================================\n");
        printf("[GPU ALLOCATION FOR %s]\n",   prop.name);
        printf("[%5.1f MB (GPU input)]\n",    input_bytes / 1024. / 1024);
        printf("[%5.1f MB (GPU ghosts)]\n",   ghost_bytes / 1024. / 1024);
        printf("[%5.1f MB (GPU output)]\n",   output_bytes / 1024. / 1024);
        printf("[%5.1f MB (Compute storage)]\n", computational_bytes / 1024. / 1024);
        GPU::tell_memUsage_GPU();
        printf("=====================================================================\n");
    }
#endif
}


void GPU::dealloc(const bool isroot)
{
#ifndef _MUTE_GPU_
    for (int var = 0; var < VSIZE; ++var)
        hipFree(d_flux[var]);

    // extraterms
    hipFree(d_Gm);
    hipFree(d_Gp);
    hipFree(d_Pm);
    hipFree(d_Pp);
    hipFree(d_hllc_vel);
    hipFree(d_sumG);
    hipFree(d_sumP);
    hipFree(d_divU);

    for (int i = 0; i < _NUM_GPU_BUF_; ++i)
    {
        GPU_COMM * const mybuf = &gpu_comm[i];
        for (int var = 0; var < VSIZE; ++var)
        {
            // x-/yghosts
            hipFree(mybuf->d_xgl[var]);
            hipFree(mybuf->d_xgr[var]);
            hipFree(mybuf->d_ygl[var]);
            hipFree(mybuf->d_ygr[var]);

            // GPU output
            hipFree(mybuf->d_divF[var]);

            // input GPU
            hipFreeArray(mybuf->d_GPUin[var]);
        }
    }

    // Max SOS
    hipHostFree(h_maxSOS);

    // destroy streams
    for (int i = 0; i < _NUM_STREAMS_; ++i)
        hipStreamDestroy(stream[i]);
    free(stream);

    // destroy events
    for (int i = 0; i < _NUM_STREAMS_; ++i)
    {
        hipEventDestroy(event_h2d[i]);
        hipEventDestroy(event_d2h[i]);
        hipEventDestroy(event_compute[i]);
    }
    free(event_h2d);
    free(event_d2h);
    free(event_compute);

    // Stats
    if (isroot)
    {
        int dev;
        hipDeviceProp_t prop;
        hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);

        printf("=====================================================================\n");
        printf("[FREE GPU %s]\n", prop.name);
        GPU::tell_memUsage_GPU();
        printf("=====================================================================\n");
    }
#endif
}


///////////////////////////////////////////////////////////////////////////
// COMMUNICATION H2D / D2H
///////////////////////////////////////////////////////////////////////////
void GPU::h2d_input(
        const uint_t Nxghost, const real_vector_t& xghost_l, const real_vector_t& xghost_r,
        const uint_t Nyghost, const real_vector_t& yghost_l, const real_vector_t& yghost_r,
        const real_vector_t& src, const uint_t nslices,
        const uint_t gbuf_id, const int chunk_id)
{
#ifndef _MUTE_GPU_
    assert(gbuf_id < _NUM_GPU_BUF_);

    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    // previous stream has priority, don't interrupt
    const uint_t s_idm1 = ((chunk_id-1) + _NUM_STREAMS_) % _NUM_STREAMS_;
    assert(s_idm1 < _NUM_STREAMS_);
    hipStreamWaitEvent(stream[s_id], event_h2d[s_idm1], 0);

    char prof_item[256];

    // TODO: use larger arrays for ghosts to minimize API overhead +
    // increase BW performance. (LOW PRIORITY)
    sprintf(prof_item, "SEND GHOSTS (%d)", s_id);
    GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
    for (int i = 0; i < VSIZE; ++i)
    {
        // x
        hipMemcpyAsync(mybuf->d_xgl[i], xghost_l[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
        hipMemcpyAsync(mybuf->d_xgr[i], xghost_r[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
        // y
        hipMemcpyAsync(mybuf->d_ygl[i], yghost_l[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
        hipMemcpyAsync(mybuf->d_ygr[i], yghost_r[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]);
    }
    GPU::profiler.pop_stopCUDA();

    GPU::h2d_3DArray(src, nslices, gbuf_id, chunk_id);
#endif

}


/* void GPU::upload_xy_ghosts(const uint_t Nxghost, const real_vector_t& xghost_l, const real_vector_t& xghost_r, */
/*         const uint_t Nyghost, const real_vector_t& yghost_l, const real_vector_t& yghost_r, */
/*         const uint_t gbuf_id, const int chunk_id) */
/* { */
/* #ifndef _MUTE_GPU_ */
/*     assert(gbuf_id < _NUM_GPU_BUF_); */

/*     const uint_t s_id = chunk_id % _NUM_STREAMS_; */
/*     GPU_COMM * const mybuf = &gpu_comm[gbuf_id]; */

/*     char prof_item[256]; */
/*     sprintf(prof_item, "SEND GHOSTS (%d)", s_id); */

/*     // TODO: use larger arrays for ghosts to minimize API overhead + */
/*     // increase BW performance. (LOW PRIORITY) */
/*     GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
/*     for (int i = 0; i < VSIZE; ++i) */
/*     { */
/*         // x */
/*         hipMemcpyAsync(mybuf->d_xgl[i], xghost_l[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]); */
/*         hipMemcpyAsync(mybuf->d_xgr[i], xghost_r[i], Nxghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]); */
/*         // y */
/*         hipMemcpyAsync(mybuf->d_ygl[i], yghost_l[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]); */
/*         hipMemcpyAsync(mybuf->d_ygr[i], yghost_r[i], Nyghost*sizeof(Real), hipMemcpyHostToDevice, stream[s_id]); */
/*     } */
/*     GPU::profiler.pop_stopCUDA(); */
/* #endif */
/* } */


void GPU::h2d_3DArray(const real_vector_t& src, const uint_t nslices,
        const uint_t gbuf_id, const int chunk_id)
{
#ifndef _MUTE_GPU_
    assert(gbuf_id < _NUM_GPU_BUF_);

    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    assert(s_id < _NUM_STREAMS_);
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    char prof_item[256];
    sprintf(prof_item, "SEND 3DARRAY (%d)", s_id);

    GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
    for (int i = 0; i < VSIZE; ++i)
        _h2d_3DArray(mybuf->d_GPUin[i], src[i], nslices, s_id);
    GPU::profiler.pop_stopCUDA();

    hipEventRecord(event_h2d[s_id], stream[s_id]);
#endif
}


void GPU::d2h_divF(real_vector_t& dst, const uint_t N,
        const uint_t gbuf_id, const int chunk_id)
{
#ifndef _MUTE_GPU_
    assert(gbuf_id < _NUM_GPU_BUF_);

    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    assert(s_id < _NUM_STREAMS_);
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    char prof_item[256];
    sprintf(prof_item, "RECV DIVF (%d)", s_id);

    /* // previous stream has priority, don't interrupt */
    /* const uint_t s_idm1 = ((chunk_id-1) + _NUM_STREAMS_) % _NUM_STREAMS_; */
    /* assert(s_idm1 < _NUM_STREAMS_); */
    /* hipStreamWaitEvent(stream[s_id], event_d2h[s_idm1]); */

    GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
    for (int i = 0; i < VSIZE; ++i)
        hipMemcpyAsync(dst[i], mybuf->d_divF[i], N*sizeof(Real), hipMemcpyDeviceToHost, stream[s_id]);
    GPU::profiler.pop_stopCUDA();

    hipEventRecord(event_d2h[s_id], stream[s_id]);
#endif
}


///////////////////////////////////////////////////////////////////////////
// Sync
///////////////////////////////////////////////////////////////////////////
void GPU::wait_h2d(const int chunk_id)
{
#ifndef _MUTE_GPU_
    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    hipEventSynchronize(event_h2d[s_id]);
#endif
}


void GPU::wait_d2h(const int chunk_id)
{
#ifndef _MUTE_GPU_
    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    hipEventSynchronize(event_d2h[s_id]);
#endif
}


void GPU::syncGPU()
{
#ifndef _MUTE_GPU_
    hipDeviceSynchronize();
#endif
}


void GPU::syncStream(const int chunk_id)
{
#ifndef _MUTE_GPU_
    const uint_t s_id = chunk_id % _NUM_STREAMS_;
    hipStreamSynchronize(stream[s_id]);
#endif
}


///////////////////////////////////////////////////////////////////////////
// Stats
///////////////////////////////////////////////////////////////////////////
void GPU::tell_memUsage_GPU()
{
#ifndef _MUTE_GPU_
    size_t free_byte, total_byte;
    const int status = hipMemGetInfo(&free_byte, &total_byte);
    if (hipSuccess != status)
    {
        printf("Can not get memory stats from GPU...\n");
        return;
    }
    const size_t used = total_byte - free_byte;
    printf("GPU memory usage: free = %5.1f MB, total = %5.1f MB (%5.1f MB used)\n",
            (double)free_byte / 1024 / 1024,
            (double)total_byte / 1024 / 1024,
            (double)used / 1024 / 1024);
#endif
}


void GPU::tell_GPU()
{
#ifndef _MUTE_GPU_
    int dev;
    hipDeviceProp_t prop;
    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop, dev);
    printf("Using device %d (%s)\n", dev, prop.name);
#endif
}
