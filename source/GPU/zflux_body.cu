#include "hip/hip_runtime.h"
/* File        : zflux_body.cu */
/* Creator     : Fabian Wermelinger <fabianw@student.ethz.ch> */
/* Created     : Thu 14 Aug 2014 03:00:29 PM CEST */
/* Modified    : Thu 14 Aug 2014 03:00:55 PM CEST */
/* Description : Computational body of z-flux */

// 2.)
// convert to primitive variables
#pragma unroll 6
for (uint_t i = 0; i < 6; ++i)
{
    e[i] = (e[i] - 0.5f*(u[i]*u[i] + v[i]*v[i] + w[i]*w[i])/r[i] - P[i]) / G[i];
    u[i] = u[i]/r[i];
    v[i] = v[i]/r[i];
    w[i] = w[i]/r[i];
} // 6 x (8 MUL/ADD/SUB + 5 DIV) = 78 FLOPS

const Real rm = _weno_minus_clipped(r[0], r[1], r[2], r[3], r[4]); // 96 FLOP (6 DIV)
const Real rp = _weno_pluss_clipped(r[1], r[2], r[3], r[4], r[5]); // 96 FLOP (6 DIV)
assert(!isnan(rp)); assert(!isnan(rm));

const Real Gm = _weno_minus_clipped(G[0], G[1], G[2], G[3], G[4]); // 96 FLOP (6 DIV)
const Real Gp = _weno_pluss_clipped(G[1], G[2], G[3], G[4], G[5]); // 96 FLOP (6 DIV)
assert(!isnan(Gp)); assert(!isnan(Gm));

const Real Pm = _weno_minus_clipped(P[0], P[1], P[2], P[3], P[4]); // 96 FLOP (6 DIV)
const Real Pp = _weno_pluss_clipped(P[1], P[2], P[3], P[4], P[5]); // 96 FLOP (6 DIV)
assert(!isnan(Pp)); assert(!isnan(Pm));

const Real pm = _weno_minus_clipped(e[0], e[1], e[2], e[3], e[4]); // 96 FLOP (6 DIV)
const Real pp = _weno_pluss_clipped(e[1], e[2], e[3], e[4], e[5]); // 96 FLOP (6 DIV)
assert(!isnan(pp)); assert(!isnan(pm));

const Real wm = _weno_minus_clipped(w[0], w[1], w[2], w[3], w[4]); // 96 FLOP (6 DIV)
const Real wp = _weno_pluss_clipped(w[1], w[2], w[3], w[4], w[5]); // 96 FLOP (6 DIV)
assert(!isnan(wp)); assert(!isnan(wm));

const Real um = _weno_minus_clipped(u[0], u[1], u[2], u[3], u[4]); // 96 FLOP (6 DIV)
const Real up = _weno_pluss_clipped(u[1], u[2], u[3], u[4], u[5]); // 96 FLOP (6 DIV)
assert(!isnan(up)); assert(!isnan(um));

const Real vm = _weno_minus_clipped(v[0], v[1], v[2], v[3], v[4]); // 96 FLOP (6 DIV)
const Real vp = _weno_pluss_clipped(v[1], v[2], v[3], v[4], v[5]); // 96 FLOP (6 DIV)
assert(!isnan(vp)); assert(!isnan(vm));

// 3.)
Real sm, sp;
_char_vel_einfeldt(rm, rp, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp); // 29 FLOP (6 DIV)
const Real ss = _char_vel_star(rm, rp, wm, wp, pm, pp, sm, sp); // 11 FLOP (1 DIV)
assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

// 4.)
const Real fr = _hllc_rho(rm, rp, wm, wp, sm, sp, ss); // 23 FLOP (2 DIV)
const Real fu = _hllc_vel(rm, rp, um, up, wm, wp, sm, sp, ss); // 25 FLOP (2 DIV)
const Real fv = _hllc_vel(rm, rp, vm, vp, wm, wp, sm, sp, ss); // 25 FLOP (2 DIV)
const Real fw = _hllc_pvel(rm, rp, wm, wp, pm, pp, sm, sp, ss); // 29 FLOP (2 DIV)
const Real fe = _hllc_e(rm, rp, wm, wp, um, up, vm, vp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss); // 59 FLOP (4 DIV)
const Real fG = _hllc_rho(Gm, Gp, wm, wp, sm, sp, ss); // 23 FLOP (2 DIV)
const Real fP = _hllc_rho(Pm, Pp, wm, wp, sm, sp, ss); // 23 FLOP (2 DIV)

const Real hllc_vel = _extraterm_hllc_vel(wm, wp, Gm, Gp, Pm, Pp, sm, sp, ss); // 19 FLOP (2 DIV)
