#include "hip/hip_runtime.h"
/* *
 * GPUkernels.cu
 *
 * Created by Fabian Wermelinger on 6/25/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */
#include <assert.h>
#include <stdio.h>

#include "GPU.cuh"

#if _BLOCKSIZEX_ < 5
#error Minimum _BLOCKSIZEX_ is 5
#elif _BLOCKSIZEY_ < 5
#error Minimum _BLOCKSIZEY_ is 5
#elif _BLOCKSIZEZ_ < 1
#error Minimum _BLOCKSIZEZ_ is 1
#endif

// TODO: this might is no longer needed
#if NX % _TILE_DIM_ != 0
#error _BLOCKSIZEX_ should be an integer multiple of _TILE_DIM_
#endif
#if NY % _TILE_DIM_ != 0
#error _BLOCKSIZEY_ should be an integer multiple of _TILE_DIM_
#endif


// DEBUG / CHECK
#include <fstream>
#include <sstream>
#include <string>
using namespace std;

///////////////////////////////////////////////////////////////////////////////
//                           GLOBAL VARIABLES                                //
///////////////////////////////////////////////////////////////////////////////
// helper storage
extern real_vector_t d_recon_p;
extern real_vector_t d_recon_m;
extern Real *d_sumG, *d_sumP, *d_divU;

// max SOS
extern int *d_maxSOS;

// GPU input/output
extern struct GPU_COMM gpu_comm[_NUM_GPU_BUF_];

// use non-null stream (async)
extern hipStream_t *stream;

// compute events
extern hipEvent_t *event_compute;

// texture references
texture<float, 3, hipReadModeElementType> tex00;
texture<float, 3, hipReadModeElementType> tex01;
texture<float, 3, hipReadModeElementType> tex02;
texture<float, 3, hipReadModeElementType> tex03;
texture<float, 3, hipReadModeElementType> tex04;
texture<float, 3, hipReadModeElementType> tex05;
texture<float, 3, hipReadModeElementType> tex06;

// TODO: REMOVE
#include "Texture.cu"

///////////////////////////////////////////////////////////////////////////////
//                             DEVICE FUNCTIONS                              //
///////////////////////////////////////////////////////////////////////////////
__device__
inline Real _weno_pluss(const Real b, const Real c, const Real d, const Real e, const Real f)
{
    const Real wenoeps_f = (Real)WENOEPS;
#ifndef _WENO3_
    // (90 MUL/ADD/SUB + 6 DIV) = 96 FLOP
    const Real inv6 = 1.0f/6.0f;
    const Real inv3 = 1.0f/3.0f;
    const Real q1 =  10.0f*inv3;
    const Real q2 =  31.0f*inv3;
    const Real q3 =  11.0f*inv3;
    const Real q4 =  25.0f*inv3;
    const Real q5 =  19.0f*inv3;
    const Real q6 =   4.0f*inv3;
    const Real q7 =  13.0f*inv3;
    const Real q8 =   5.0f*inv3;

    const Real sum0 =  inv3*f - 7.0f*inv6*e + 11.0f*inv6*d;
    const Real sum1 = -inv6*e + 5.0f*inv6*d + inv3*c;
    const Real sum2 =  inv3*d + 5.0f*inv6*c - inv6*b;

    const Real is0 = d*(d*q1 - e*q2 + f*q3) + e*(e*q4 - f*q5) + f*f*q6;
    const Real is1 = c*(c*q6 - d*q7 + e*q8) + d*(d*q7 - e*q7) + e*e*q6;
    const Real is2 = b*(b*q6 - c*q5 + d*q3) + c*(c*q4 - d*q2) + d*d*q1;

    const Real is0plus = is0 + wenoeps_f;
    const Real is1plus = is1 + wenoeps_f;
    const Real is2plus = is2 + wenoeps_f;

    const Real alpha0 = 1.0f / (10.0f*is0plus*is0plus);
    const Real alpha1 = 6.0f * (1.0f / (10.0f*is1plus*is1plus));
    const Real alpha2 = 3.0f * (1.0f / (10.0f*is2plus*is2plus));
    const Real alphasumInv = 1.0f / (alpha0+alpha1+alpha2);

    const Real omega0 = alpha0 * alphasumInv;
    const Real omega1 = alpha1 * alphasumInv;
    const Real omega2 = 1.0f - omega0 - omega1;

    return omega0*sum0 + omega1*sum1 + omega2*sum2;

#else
    // 28 FLOP
    const Real sum0 = 1.5f*d - 0.5f*e;
    const Real sum1 = 0.5f*(d + c);

    const Real is0 = (d-e)*(d-e);
    const Real is1 = (d-c)*(d-c);

    const Real alpha0 = 1.0f / (3.0f * (is0+wenoeps_f)*(is0+wenoeps_f));
    const Real alpha1 = 2.0f * (1.0f / (3.0f * (is1+wenoeps_f)*(is1+wenoeps_f)));

    const Real omega0 = alpha0 / (alpha0+alpha1);
    const Real omega1 = 1.0f - omega0;

    return omega0*sum0 + omega1*sum1;

#endif
}


__device__
inline Real _weno_minus(const Real a, const Real b, const Real c, const Real d, const Real e)
{
    const Real wenoeps_f = (Real)WENOEPS;
#ifndef _WENO3_
    // (90 MUL/ADD/SUB + 6 DIV) = 96 FLOP
    const Real inv6 = 1.0f/6.0f;
    const Real inv3 = 1.0f/3.0f;
    const Real q1 =   4.0f*inv3;
    const Real q2 =  19.0f*inv3;
    const Real q3 =  11.0f*inv3;
    const Real q4 =  25.0f*inv3;
    const Real q5 =  31.0f*inv3;
    const Real q6 =  10.0f*inv3;
    const Real q7 =  13.0f*inv3;
    const Real q8 =   5.0f*inv3;

    const Real sum0 =  inv3*a - 7.0f*inv6*b + 11.0f*inv6*c;
    const Real sum1 = -inv6*b + 5.0f*inv6*c + inv3*d;
    const Real sum2 =  inv3*c + 5.0f*inv6*d - inv6*e;

    const Real is0 = a*(a*q1 - b*q2 + c*q3) + b*(b*q4 - c*q5) + c*c*q6;
    const Real is1 = b*(b*q1 - c*q7 + d*q8) + c*(c*q7 - d*q7) + d*d*q1;
    const Real is2 = c*(c*q6 - d*q5 + e*q3) + d*(d*q4 - e*q2) + e*e*q1;

    const Real is0plus = is0 + wenoeps_f;
    const Real is1plus = is1 + wenoeps_f;
    const Real is2plus = is2 + wenoeps_f;

    const Real alpha0 = 1.0f / (10.0f*is0plus*is0plus);
    const Real alpha1 = 6.0f * (1.0f / (10.0f*is1plus*is1plus));
    const Real alpha2 = 3.0f * (1.0f / (10.0f*is2plus*is2plus));
    const Real alphasumInv = 1.0f / (alpha0+alpha1+alpha2);

    const Real omega0 = alpha0 * alphasumInv;
    const Real omega1 = alpha1 * alphasumInv;
    const Real omega2 = 1.0f - omega0 - omega1;

    return omega0*sum0 + omega1*sum1 + omega2*sum2;

#else
    // 28 FLOP
    const Real sum0 = 1.5f*c - 0.5f*b;
    const Real sum1 = 0.5f*(c + d);

    const Real is0 = (c-b)*(c-b);
    const Real is1 = (d-c)*(d-c);

    const Real alpha0 = 1.0f / (3.0f * (is0+wenoeps_f)*(is0+wenoeps_f));
    const Real alpha1 = 2.0f * (1.0f / (3.0f * (is1+wenoeps_f)*(is1+wenoeps_f)));

    const Real omega0 = alpha0 / (alpha0+alpha1);
    const Real omega1 = 1.0f - omega0;

    return omega0*sum0 + omega1*sum1;

#endif
}


__device__
inline Real _weno_pluss_clipped(const Real b, const Real c, const Real d, const Real e, const Real f)
{
    const Real retval = _weno_pluss(b,c,d,e,f);
    const Real min_in = fminf( fminf(c,d), e );
    const Real max_in = fmaxf( fmaxf(c,d), e );
    return fminf(fmaxf(retval, min_in), max_in);
}


__device__
inline Real _weno_minus_clipped(const Real a, const Real b, const Real c, const Real d, const Real e)
{
    const Real retval = _weno_minus(a,b,c,d,e);
    const Real min_in = fminf( fminf(b,c), d );
    const Real max_in = fmaxf( fmaxf(b,c), d );
    return fminf(fmaxf(retval, min_in), max_in);
}


__device__
inline void _char_vel_einfeldt(const Real rm, const Real rp,
        const Real vm, const Real vp,
        const Real pm, const Real pp,
        const Real Gm, const Real Gp,
        const Real Pm, const Real Pp,
        Real& outm, Real& outp) // (23 MUL/ADD/SUB + 6 DIV) = 29 FLOP
{
    /* *
     * Compute upper and lower bounds of signal velocities for the Riemann
     * problem according to Einfeldt:
     *
     * 1.) Compute Rr needed for Roe averages
     * 2.) Compute speed of sound in left and right state
     * 3.) Compute speed of sound according to Einfeldt and Rr
     * 4.) Compute upper and lower signal velocities
     * */

    // 1.)
    assert(rm > 0.0f);
    assert(rp > 0.0f);
    const Real Rr   = sqrtf(rp / rm);
    const Real Rinv = 1.0f / (1.0f + Rr);

    // 2.)
    const Real cm2 = ((pm + Pm)/Gm + pm) / rm;
    const Real cp2 = ((pp + Pp)/Gp + pp) / rp;
    const Real cm  = sqrtf(cm2);
    const Real cp  = sqrtf(cp2);
    assert(!isnan(cm));
    assert(!isnan(cp));

    // 3.)
    const Real um    = vm;
    const Real up    = vp;
    const Real eta_2 = 0.5f*Rr*Rinv*Rinv;
    const Real d2    = (cm2 + Rr*cp2)*Rinv + eta_2*(up - um)*(up - um);
    const Real d     = sqrtf(d2);
    const Real u     = (um + Rr*up)*Rinv;
    assert(!isnan(d));
    assert(!isnan(u));

    // 4.)
    outm = fminf(u - d, um - cm);
    outp = fmaxf(u + d, up + cp);
}


/* *
 * Compute characteristic velocity, s^star, of the intermediate wave.  The
 * computation is based on the condition of uniform constant pressure in
 * the star region.  See P. Batten et. al., "On the choice of wavespeeds
 * for the HLLC Riemann solver", SIAM J. Sci. Comput. 18 (1997) 1553--1570
 * It is assumed s^minus and s^plus are known.
 * */
__device__
inline Real _char_vel_star(const Real rm, const Real rp,
        const Real vm, const Real vp,
        const Real pm, const Real pp,
        const Real sm, const Real sp) // (10 MUL/ADD/SUB + 1 DIV) = 11 FLOP
{
    const Real facm = rm * (sm - vm);
    const Real facp = rp * (sp - vp);
    return (pp - pm + vm*facm - vp*facp) / (facm - facp);
    /* return (pp + vm*facm - (pm + vp*facp)) / (facm - facp); */
}


__device__
inline Real _hllc_rho(const Real rm, const Real rp,
        const Real vm, const Real vp,
        const Real sm, const Real sp, const Real ss) // (21 MUL/ADD/SUB + 2 DIV) = 23 FLOP
{
    /* *
     * The flux computation is split into 4 parts:
     * 1.) Compute signum of s^*, compute s^- and s^+
     * 2.) Compute chi^* and delta of q^* and q
     * 3.) Compute trivial flux
     * 4.) Compute HLLC flux
     * */

    // 1.)
    const Real sign_star = (ss == 0.0f) ? 0.0f : ((ss < 0.0f) ? -1.0f : 1.0f);
    const Real s_minus = fminf(0.0f, sm);
    const Real s_pluss = fmaxf(0.0f, sp);

    // 2.)
    const Real chi_starm = (sm - vm) / (sm - ss);
    const Real chi_starp = (sp - vp) / (sp - ss);
    const Real qm        = rm;
    const Real qp        = rp;
    const Real q_deltam  = qm*chi_starm - qm;
    const Real q_deltap  = qp*chi_starp - qp;

    // 3.)
    const Real fm = qm*vm;
    const Real fp = qp*vp;

    // 4.)
    const Real flux = (0.5f*(1.0f + sign_star)) * (fm + s_minus*q_deltam) + (0.5f*(1.0f - sign_star)) * (fp + s_pluss*q_deltap);
    assert(!isnan(flux));
    return flux;
}


__device__
inline Real _hllc_vel(const Real rm,  const Real rp,
        const Real vm,  const Real vp,
        const Real vdm, const Real vdp,
        const Real sm,  const Real sp,  const Real ss) // (23 MUL/ADD/SUB + 2 DIV) = 25 FLOP
{
    /* *
     * The flux computation is split into 4 parts:
     * 1.) Compute signum of s^*, compute s^- and s^+
     * 2.) Compute chi^* and delta of q^* and q
     * 3.) Compute trivial flux
     * 4.) Compute HLLC flux
     * */

    // 1.)
    const Real sign_star = (ss == 0.0f) ? 0.0f : ((ss < 0.0f) ? -1.0f : 1.0f);
    const Real s_minus  = fminf(0.0f, sm);
    const Real s_pluss  = fmaxf(0.0f, sp);

    // 2.)
    const Real chi_starm = (sm - vdm) / (sm - ss);
    const Real chi_starp = (sp - vdp) / (sp - ss);
    const Real qm        = rm*vm;
    const Real qp        = rp*vp;
    const Real q_deltam  = qm*chi_starm - qm;
    const Real q_deltap  = qp*chi_starp - qp;

    // 3.)
    const Real fm = qm*vdm;
    const Real fp = qp*vdp;

    // 4.)
    const Real flux = (0.5f*(1.0f + sign_star)) * (fm + s_minus*q_deltam) + (0.5f*(1.0f - sign_star)) * (fp + s_pluss*q_deltap);
    assert(!isnan(flux));
    assert(!isnan(ss));
    assert(!isnan(sm));
    assert(!isnan(sp));
    return flux;
}


__device__
inline Real _hllc_pvel(const Real rm, const Real rp,
        const Real vm, const Real vp,
        const Real pm, const Real pp,
        const Real sm, const Real sp, const Real ss) // (27 MUL/ADD/SUB + 2 DIV) = 29 FLOP
{
    /* *
     * The flux computation is split into 4 parts:
     * 1.) Compute signum of s^*, compute s^- and s^+
     * 2.) Compute chi^* and delta of q^* and q
     * 3.) Compute trivial flux
     * 4.) Compute HLLC flux
     * */

    // 1.)
    const Real sign_star = (ss == 0.0f) ? 0.0f : ((ss < 0.0f) ? -1.0f : 1.0f);
    const Real s_minus  = fminf(0.0f, sm);
    const Real s_pluss  = fmaxf(0.0f, sp);

    // 2.)
    const Real chi_starm = (sm - vm) / (sm - ss);
    const Real chi_starp = (sp - vp) / (sp - ss);
    const Real qm        = rm*vm;
    const Real qp        = rp*vp;
    const Real q_deltam  = rm*ss*chi_starm - qm;
    const Real q_deltap  = rp*ss*chi_starp - qp;

    // 3.)
    const Real fm = qm*vm + pm;
    const Real fp = qp*vp + pp;

    // 4.)
    const Real flux = (0.5f*(1.0f + sign_star)) * (fm + s_minus*q_deltam) + (0.5f*(1.0f - sign_star)) * (fp + s_pluss*q_deltap);
    assert(!isnan(flux));
    assert(rm > 0);
    assert(rp > 0);
    return flux;
}


__device__
inline Real _hllc_e(const Real rm,  const Real rp,
        const Real vdm, const Real vdp,
        const Real v1m, const Real v1p,
        const Real v2m, const Real v2p,
        const Real pm,  const Real pp,
        const Real Gm,  const Real Gp,
        const Real Pm,  const Real Pp,
        const Real sm,  const Real sp,  const Real ss) // (55 MUL/ADD/SUB + 4 DIV) = 59 FLOP
{
    /* *
     * The flux computation is split into 4 parts:
     * 1.) Compute signum of s^*, compute s^- and s^+
     * 2.) Compute chi^* and delta of q^* and q
     * 3.) Compute trivial flux
     * 4.) Compute HLLC flux
     * */

    // 1.)
    const Real sign_star = (ss == 0.0f) ? 0.0f : ((ss < 0.0f) ? -1.0f : 1.0f);
    const Real s_minus  = fminf(0.0f, sm);
    const Real s_pluss  = fmaxf(0.0f, sp);

    // 2.)
    const Real chi_starm = (sm - vdm) / (sm - ss);
    const Real chi_starp = (sp - vdp) / (sp - ss);
    const Real qm        = Gm*pm + Pm + 0.5f*rm*(vdm*vdm + v1m*v1m + v2m*v2m);
    const Real qp        = Gp*pp + Pp + 0.5f*rp*(vdp*vdp + v1p*v1p + v2p*v2p);
    const Real q_deltam  = chi_starm*(qm + (ss - vdm)*(rm*ss + pm/(sm - vdm))) - qm;
    const Real q_deltap  = chi_starp*(qp + (ss - vdp)*(rp*ss + pp/(sp - vdp))) - qp;

    // 3.)
    const Real fm = vdm*(qm + pm);
    const Real fp = vdp*(qp + pp);

    // 4.)
    const Real flux = (0.5f*(1.0f + sign_star)) * (fm + s_minus*q_deltam) + (0.5f*(1.0f - sign_star)) * (fp + s_pluss*q_deltap);
    assert(!isnan(flux));
    return flux;
}


__device__
inline Real _extraterm_hllc_vel(const Real um, const Real up,
        const Real Gm, const Real Gp,
        const Real Pm, const Real Pp,
        const Real sm, const Real sp, const Real ss) // (17 MUL/ADD/SUB + 2 DIV) = 19 FLOP
{
    const Real sign_star = (ss == 0.0f) ? 0.0f : ((ss < 0.0f) ? -1.0f : 1.0f);
    const Real s_minus   = fminf(0.0f, sm);
    const Real s_pluss   = fmaxf(0.0f, sp);
    const Real chi_starm = (sm - um)/(sm - ss) - 1.0f;
    const Real chi_starp = (sp - up)/(sp - ss) - 1.0f;

    return (0.5f*(1.0f + sign_star))*(um + s_minus*chi_starm) + (0.5f*(1.0f - sign_star))*(up + s_pluss*chi_starp);
}


///////////////////////////////////////////////////////////////////////////////
//                                  KERNELS                                  //
///////////////////////////////////////////////////////////////////////////////
#define _STENCIL_WIDTH_ 6

template <int texID> __device__ inline float myTex3D(const int ix, const int iy, const int iz);
template <> __device__ inline float myTex3D<0>(const int ix, const int iy, const int iz) { return tex3D(tex00, ix, iy, iz); }
template <> __device__ inline float myTex3D<1>(const int ix, const int iy, const int iz) { return tex3D(tex01, ix, iy, iz); }
template <> __device__ inline float myTex3D<2>(const int ix, const int iy, const int iz) { return tex3D(tex02, ix, iy, iz); }
template <> __device__ inline float myTex3D<3>(const int ix, const int iy, const int iz) { return tex3D(tex03, ix, iy, iz); }
template <> __device__ inline float myTex3D<4>(const int ix, const int iy, const int iz) { return tex3D(tex04, ix, iy, iz); }
template <> __device__ inline float myTex3D<5>(const int ix, const int iy, const int iz) { return tex3D(tex05, ix, iy, iz); }
template <> __device__ inline float myTex3D<6>(const int ix, const int iy, const int iz) { return tex3D(tex06, ix, iy, iz); }

template <int texID, int gid0, int tid0, int gmap0, int tmap0, int ng> __device__
inline void _load_boundary_X(const uint_t iy, const uint_t iz,
        Real * const __restrict__ stencil, const Real * const __restrict__ ghost)
{
    /* *
     * load stencil data from texture and ghost mix.
     * texID = texture reference to use
     * gid0  = start index of first ghost value in stencil
     * tid0  = start index of first texture value in stencil
     * gmap0 = start index of first ghost value in ghost array
     * tmap0 = start index of first tex value in 3DArray
     * ng    = number of ghosts
     *
     * Assuming _STENCIL_WIDTH_ = 6, possible combinations are (x = ghost,
     * o = texture), stencil is processed from left to right:
     *
     * gmap=0
     * |     tmap=0
     * |     |
     * x x x o o o         tmap=NX-1 (gid0=0; tid0=3; gmap0=0; tmap0=0; ng=3)
     *   x x o o o o       | gmap=0  (gid0=0; tid0=2; gmap0=1; tmap0=0; ng=2)
     *     x o o o o o     | |       (gid0=0; tid0=1; gmap0=2; tmap0=0; ng=1)
     *             o o o o o x       (gid0=5; tid0=0; gmap0=0; tmap0=NX-5; ng=1)
     *               o o o o x x     (gid0=4; tid0=0; gmap0=0; tmap0=NX-4; ng=2)
     *                 o o o x x x   (gid0=3; tid0=0; gmap0=0; tmap0=NX-3; ng=3)
     * */
    const int giz = iz-3; // iz starts at 3 due to zghosts in texture, but not in ghost array
    for (int i = 0; i < ng; ++i)
        stencil[gid0 + i] = ghost[GHOSTMAPX(gmap0+i, iy, giz)];

    const int ntex = _STENCIL_WIDTH_ - ng;
    for (int i = 0; i < ntex; ++i)
        stencil[tid0 + i] = myTex3D<texID>(tmap0+i, iy, iz);
}

template <int texID> __device__
inline void _load_internal_X(const uint_t ix, const uint_t iy, const uint_t iz, Real * __restrict__ stencil)
{
    // fixed stencil: - - - 0 + + + + + . . .
    const int s_start = -3;
    const int s_end   = _STENCIL_WIDTH_ + s_start;
    for (int i=s_start; i < s_end; ++i)
        *stencil++ = myTex3D<texID>(ix+i, iy, iz);
}


template <int texID> __global__
void _WENO_X(Real * const __restrict__ p_minus, Real * const __restrict__ p_plus,
        const Real * const __restrict__ p_ghostL, const Real * const __restrict__ p_ghostR)
{
    // this ensures that a stencil can only contain either left ghosts or right
    // ghosts, but not a mix of left AND right ghosts.  This minimizes
    // if-conditionals below when reading the stencil. Therefore, minimum
    // number of cells in X-direction is 5
    assert(NXP1 > 5);

    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint_t iz = blockIdx.z * blockDim.z + threadIdx.z + 3; // textures are padded by 3 slices in z (zghosts)

    if (ix < NXP1 && iy < NY)
    {
        const uint_t idx = ID3(iy, ix, iz-3, NY, NXP1);

        Real s[_STENCIL_WIDTH_]; // stencil

        if (0 == ix)
            _load_boundary_X<texID, 0, 3, 0, 0, 3>(iy, iz, s, p_ghostL);
        else if (1 == ix)
            _load_boundary_X<texID, 0, 2, 1, 0, 2>(iy, iz, s, p_ghostL);
        else if (2 == ix)
            _load_boundary_X<texID, 0, 1, 2, 0, 1>(iy, iz, s, p_ghostL);
        else if (NXP1-3 == ix)
            _load_boundary_X<texID, _STENCIL_WIDTH_-1, 0, 0, NX-(_STENCIL_WIDTH_-1), 1>(iy, iz, s, p_ghostR);
        else if (NXP1-2 == ix)
            _load_boundary_X<texID, _STENCIL_WIDTH_-2, 0, 0, NX-(_STENCIL_WIDTH_-2), 2>(iy, iz, s, p_ghostR);
        else if (NXP1-1 == ix)
            _load_boundary_X<texID, _STENCIL_WIDTH_-3, 0, 0, NX-(_STENCIL_WIDTH_-3), 3>(iy, iz, s, p_ghostR);
        else
            _load_internal_X<texID>(ix, iy, iz, s);

        const Real recon_m = _weno_minus_clipped(s[0], s[1], s[2], s[3], s[4]); // 96 FLOP (6 DIV)
        const Real recon_p = _weno_pluss_clipped(s[1], s[2], s[3], s[4], s[5]); // 96 FLOP (6 DIV)
        assert(!isnan(recon_m)); assert(!isnan(recon_p));

        // write
        p_minus[idx] = recon_m;
        p_plus[idx]  = recon_p;
    }
}


__global__ void
/* __launch_bounds__(128, 16) */
_HLLC_X(DevicePointer recon_m, DevicePointer recon_p)
{
    // this ensures that a stencil can only contain either left ghosts or right
    // ghosts, but not a mix of left AND right ghosts.  This minimizes
    // if-conditionals below when reading the stencil. Therefore, minimum
    // number of cells in X-direction is 5
    assert(NXP1 > 5);

    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint_t iz = blockIdx.z * blockDim.z + threadIdx.z + 3;

    // reduce reduce register pressure
    __shared__ Real rm, rp, um, up, vm, vp, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss;

    if (ix < NXP1 && iy < NY)
    {
        const uint_t idx = ID3(iy, ix, iz-3, NY, NXP1);

        rm = recon_m.r[idx];
        rp = recon_p.r[idx];
        um = recon_m.u[idx];
        up = recon_p.u[idx];
        pm = recon_m.e[idx];
        pp = recon_p.e[idx];
        Gm = recon_m.G[idx];
        Gp = recon_p.G[idx];
        Pm = recon_m.P[idx];
        Pp = recon_p.P[idx];
        vm = recon_m.v[idx];
        vp = recon_p.v[idx];
        wm = recon_m.w[idx];
        wp = recon_p.w[idx];
        assert(rm > 0.0f); assert(rp > 0.0f);
        assert(pm > 0.0f); assert(pp > 0.0f);
        assert(Gm > 0.0f); assert(Gp > 0.0f);
        assert(Pm >= 0.0f); assert(Pp >= 0.0f);

        // TODO: inline computations below
        _char_vel_einfeldt(rm, rp, um, up, pm, pp, Gm, Gp, Pm, Pp, sm, sp); // 29 FLOP (6 DIV)
        ss = _char_vel_star(rm, rp, um, up, pm, pp, sm, sp); // 11 FLOP (1 DIV)
        assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

        const Real fr = _hllc_rho(rm, rp, um, up, sm, sp, ss); // 23 FLOP (2 DIV)
        const Real fu = _hllc_pvel(rm, rp, um, up, pm, pp, sm, sp, ss); // 29 FLOP (2 DIV)
        const Real fv = _hllc_vel(rm, rp, vm, vp, um, up, sm, sp, ss); // 25 FLOP (2 DIV)
        const Real fw = _hllc_vel(rm, rp, wm, wp, um, up, sm, sp, ss); // 25 FLOP (2 DIV)
        const Real fe = _hllc_e(rm, rp, um, up, vm, vp, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss); // 59 FLOP (4 DIV)
        const Real fG = _hllc_rho(Gm, Gp, um, up, sm, sp, ss); // 23 FLOP (2 DIV)
        const Real fP = _hllc_rho(Pm, Pp, um, up, sm, sp, ss); // 23 FLOP (2 DIV)
        assert(!isnan(fr)); assert(!isnan(fu)); assert(!isnan(fv)); assert(!isnan(fw)); assert(!isnan(fe)); assert(!isnan(fG)); assert(!isnan(fP));

        const Real hllc_vel = _extraterm_hllc_vel(um, up, Gm, Gp, Pm, Pp, sm, sp, ss); // 19 FLOP (2 DIV)

        // this is crap!
        recon_p.r[idx] = Gm;
        recon_p.u[idx] = Gp;
        recon_p.v[idx] = Pm;
        recon_p.w[idx] = Pp;

        recon_m.r[idx] = fr;
        recon_m.u[idx] = fu;
        recon_m.v[idx] = fv;
        recon_m.w[idx] = fw;
        recon_m.e[idx] = fe;
        recon_m.G[idx] = fG;
        recon_m.P[idx] = fP;

        recon_p.e[idx] = hllc_vel;
    }
}


/* __global__ void */
/* /1* __launch_bounds__(128, 16) *1/ */
/* _HLLC3D_X(const uint_t nslices, DevicePointer recon_m, DevicePointer recon_p) */
/* { */
/*     // this ensures that a stencil can only contain either left ghosts or right */
/*     // ghosts, but not a mix of left AND right ghosts.  This minimizes */
/*     // if-conditionals below when reading the stencil. Therefore, minimum */
/*     // number of cells in X-direction is 5 */
/*     assert(NXP1 > 5); */

/*     const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x; */
/*     /1* const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y; *1/ */
/*     const uint_t iy = blockIdx.y * 2 * _WARPSIZE_ + threadIdx.y; */
/*     const uint_t iz = blockIdx.z * blockDim.z + threadIdx.z + 3; */

/*     __shared__ Real rm[2], rp[2], um[2], up[2], vm[2], vp[2], wm[2], wp[2], pm[2], pp[2], Gm[2], Gp[2], Pm[2], Pp[2], sm[2], sp[2], ss[2]; */
/*     /1* __shared__ uint_t idx[2]; *1/ */

/*     if (ix < NXP1 && iy < NY) */
/*     { */
/*         for (int i = 0; i < 2; ++i) */
/*         { */
/*             const uint_t idx = ID3(iy+i*_WARPSIZE_, ix, iz-3, NY, NXP1); */
/*             rm[i] = recon_m.r[idx]; */
/*             rp[i] = recon_p.r[idx]; */
/*             um[i] = recon_m.u[idx]; */
/*             up[i] = recon_p.u[idx]; */
/*             pm[i] = recon_m.e[idx]; */
/*             pp[i] = recon_p.e[idx]; */
/*             Gm[i] = recon_m.G[idx]; */
/*             Gp[i] = recon_p.G[idx]; */
/*             Pm[i] = recon_m.P[idx]; */
/*             Pp[i] = recon_p.P[idx]; */
/*             vm[i] = recon_m.v[idx]; */
/*             vp[i] = recon_p.v[idx]; */
/*             wm[i] = recon_m.w[idx]; */
/*             wp[i] = recon_p.w[idx]; */
/*             assert(rm > 0.0f); assert(rp > 0.0f); */
/*             assert(pm > 0.0f); assert(pp > 0.0f); */
/*             assert(Gm > 0.0f); assert(Gp > 0.0f); */
/*             assert(Pm >= 0.0f); assert(Pp >= 0.0f); */
/*         } */

/*         for (int i = 0; i < 2; ++i) */
/*         { */
/*             const uint_t idx = ID3(iy+i*_WARPSIZE_, ix, iz-3, NY, NXP1); */

/*             /1* Real sm, sp; *1/ */
/*             _char_vel_einfeldt(rm[i], rp[i], um[i], up[i], pm[i], pp[i], Gm[i], Gp[i], Pm[i], Pp[i], sm[i], sp[i]); // 29 FLOP (6 DIV) */
/*             /1* const Real ss = _char_vel_star(rm[i], rp[i], um[i], up[i], pm[i], pp[i], sm[i], sp[i]); // 11 FLOP (1 DIV) *1/ */
/*             ss[i] = _char_vel_star(rm[i], rp[i], um[i], up[i], pm[i], pp[i], sm[i], sp[i]); // 11 FLOP (1 DIV) */
/*             assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss)); */

/*             const Real fr = _hllc_rho(rm[i], rp[i], um[i], up[i], sm[i], sp[i], ss[i]); // 23 FLOP (2 DIV) */
/*             const Real fu = _hllc_pvel(rm[i], rp[i], um[i], up[i], pm[i], pp[i], sm[i], sp[i], ss[i]); // 29 FLOP (2 DIV) */
/*             const Real fv = _hllc_vel(rm[i], rp[i], vm[i], vp[i], um[i], up[i], sm[i], sp[i], ss[i]); // 25 FLOP (2 DIV) */
/*             const Real fw = _hllc_vel(rm[i], rp[i], wm[i], wp[i], um[i], up[i], sm[i], sp[i], ss[i]); // 25 FLOP (2 DIV) */
/*             const Real fe = _hllc_e(rm[i], rp[i], um[i], up[i], vm[i], vp[i], wm[i], wp[i], pm[i], pp[i], Gm[i], Gp[i], Pm[i], Pp[i], sm[i], sp[i], ss[i]); // 59 FLOP (4 DIV) */
/*             const Real fG = _hllc_rho(Gm[i], Gp[i], um[i], up[i], sm[i], sp[i], ss[i]); // 23 FLOP (2 DIV) */
/*             const Real fP = _hllc_rho(Pm[i], Pp[i], um[i], up[i], sm[i], sp[i], ss[i]); // 23 FLOP (2 DIV) */
/*             assert(!isnan(fr)); assert(!isnan(fu)); assert(!isnan(fv)); assert(!isnan(fw)); assert(!isnan(fe)); assert(!isnan(fG)); assert(!isnan(fP)); */

/*             const Real hllc_vel = _extraterm_hllc_vel(um[i], up[i], Gm[i], Gp[i], Pm[i], Pp[i], sm[i], sp[i], ss[i]); // 19 FLOP (2 DIV) */

/*             // this is crap! */
/*             recon_p.r[idx] = Gm[i]; */
/*             recon_p.u[idx] = Gp[i]; */
/*             recon_p.v[idx] = Pm[i]; */
/*             recon_p.w[idx] = Pp[i]; */

/*             recon_m.r[idx] = fr; */
/*             recon_m.u[idx] = fu; */
/*             recon_m.v[idx] = fv; */
/*             recon_m.w[idx] = fw; */
/*             recon_m.e[idx] = fe; */
/*             recon_m.G[idx] = fG; */
/*             recon_m.P[idx] = fP; */

/*             recon_p.e[idx] = hllc_vel; */
/*         } */
/*     } */
/* } */


__global__
void _TEST_CONV(DevicePointer inout,
        DevicePointer xgL, DevicePointer xgR,
        DevicePointer ygL, DevicePointer ygR)
{
    const Real r_ref = 1.5f;
    const Real u_ref = 1.0f;
    const Real v_ref = 1.0f;
    const Real w_ref = 1.0f;
    const Real e_ref = 1.0f;
    const Real G_ref = 2.0f;
    const Real P_ref = 3.0f;

    // test main body
    const uint_t Ninout = NX * NY * (NodeBlock::sizeZ + 6);
    for (int i = 0; i < Ninout; ++i)
    {
        /* printf("%f\n", inout.w[i]); */
        assert(inout.r[i] == r_ref);
        assert(inout.u[i] == u_ref);
        assert(inout.v[i] == v_ref);
        assert(inout.w[i] == w_ref);
        assert(inout.e[i] == e_ref);
        assert(inout.G[i] == G_ref);
        assert(inout.P[i] == P_ref);
    }

    // test xghosts
    const uint_t Nxghost = 3*NY*(NodeBlock::sizeZ);
    for (int i = 0; i < Nxghost; ++i)
    {
        assert(xgR.r[i] == r_ref);
        assert(xgR.u[i] == u_ref);
        assert(xgR.v[i] == v_ref);
        assert(xgR.w[i] == w_ref);
        assert(xgR.e[i] == e_ref);
        assert(xgR.G[i] == G_ref);
        assert(xgR.P[i] == P_ref);

        assert(xgL.r[i] == r_ref);
        assert(xgL.u[i] == u_ref);
        assert(xgL.v[i] == v_ref);
        assert(xgL.w[i] == w_ref);
        assert(xgL.e[i] == e_ref);
        assert(xgL.G[i] == G_ref);
        assert(xgL.P[i] == P_ref);
    }

    // test yghosts
    const uint_t Nyghost = NX*3*(NodeBlock::sizeZ);
    for (int i = 0; i < Nyghost; ++i)
    {
        assert(ygR.r[i] == r_ref);
        assert(ygR.u[i] == u_ref);
        assert(ygR.v[i] == v_ref);
        assert(ygR.w[i] == w_ref);
        assert(ygR.e[i] == e_ref);
        assert(ygR.G[i] == G_ref);
        assert(ygR.P[i] == P_ref);

        assert(ygL.r[i] == r_ref);
        assert(ygL.u[i] == u_ref);
        assert(ygL.v[i] == v_ref);
        assert(ygL.w[i] == w_ref);
        assert(ygL.e[i] == e_ref);
        assert(ygL.G[i] == G_ref);
        assert(ygL.P[i] == P_ref);
    }
}


/* __global__ */
/* void _CONV(const uint_t nslices, DevicePointer data) */
/* { */
/*     const uint_t ix = blockIdx.x * _TILE_DIM_ + threadIdx.x; */
/*     const uint_t iy = blockIdx.y * _TILE_DIM_ + threadIdx.y; */
/*     const uint_t offset = _BLOCK_ROWS_ * NX; */

/*     if (ix < NX && iy < NY) */
/*     { */
/*         for (uint_t iz = 0; iz < nslices; ++iz) // zghosts inclusive */
/*         { */
/*             uint_t i0 = ID3(ix,iy,iz,NX,NY); */
/*             Real *pr = &data.r[i0]; */
/*             Real *pu = &data.u[i0]; */
/*             Real *pv = &data.v[i0]; */
/*             Real *pw = &data.w[i0]; */
/*             Real *pe = &data.e[i0]; */
/*             Real *pG = &data.G[i0]; */
/*             Real *pP = &data.P[i0]; */
/*             for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_) */
/*             { */
/*                 /1* const uint_t myidx = ID3(ix,iy+i,iz,NX,NY); *1/ */
/*                 const Real r = *pr; */
/*                 const Real u = *pu; */
/*                 const Real v = *pv; */
/*                 const Real w = *pw; */
/*                 const Real e = *pe; */
/*                 const Real G = *pG; */
/*                 const Real P = *pP; */

/*                 // convert */
/*                 const Real rinv = 1.0f/r; */
/*                 *pu = u*rinv; */
/*                 *pv = v*rinv; */
/*                 *pw = w*rinv; */
/*                 *pe = (e - 0.5f*(u*u + v*v + w*w)*rinv - P) / G; */

/*                 pr += offset; */
/*                 pu += offset; */
/*                 pv += offset; */
/*                 pw += offset; */
/*                 pe += offset; */
/*                 pG += offset; */
/*                 pP += offset; */
/*             } */
/*         } */
/*     } */
/* } */


/* __global__ */
/* void _CONV(const uint_t nslices, DevicePointer data) */
/* { */
/*     const uint_t ix = blockIdx.x * _TILE_DIM_ + threadIdx.x; */
/*     const uint_t iy = blockIdx.y * _TILE_DIM_ + threadIdx.y; */
/*     const uint_t offset = _BLOCK_ROWS_ * NX; */

/*     if (ix < NX && iy < NY) */
/*     { */
/*         for (uint_t iz = 0; iz < nslices; ++iz) // zghosts inclusive */
/*         { */
/*             uint_t i0 = ID3(ix,iy,iz,NX,NY); */
/*             for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_) */
/*             { */
/*                 const Real r = data.r[i0]; */
/*                 const Real u = data.u[i0]; */
/*                 const Real v = data.v[i0]; */
/*                 const Real w = data.w[i0]; */
/*                 const Real e = data.e[i0]; */
/*                 const Real G = data.G[i0]; */
/*                 const Real P = data.P[i0]; */

/*                 // convert */
/*                 const Real rinv = 1.0f/r; */
/*                 data.u[i0] = u*rinv; */
/*                 data.v[i0] = v*rinv; */
/*                 data.w[i0] = w*rinv; */
/*                 data.e[i0] = (e - 0.5f*(u*u + v*v + w*w)*rinv - P) / G; */

/*                 i0 += offset; */
/*             } */
/*         } */
/*     } */
/* } */


__global__
void _CONV(DevicePointer data)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;
    const uint_t iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix < NX && iy < NY)
    {
        const uint_t i0 = ID3(ix,iy,iz,NX,NY);

        const Real r = data.r[i0];
        const Real u = data.u[i0];
        const Real v = data.v[i0];
        const Real w = data.w[i0];
        const Real e = data.e[i0];
        const Real G = data.G[i0];
        const Real P = data.P[i0];

        // convert
        const Real rinv = 1.0f/r;
        data.u[i0] = u*rinv;
        data.v[i0] = v*rinv;
        data.w[i0] = w*rinv;
        data.e[i0] = (e - 0.5f*(u*u + v*v + w*w)*rinv - P) / G;
    }
}


__global__
void _xextraterm_hllc(const uint_t nslices, DevicePointer divF, DevicePointer flux,
        const Real * const __restrict__ Gm, const Real * const __restrict__ Gp,
        const Real * const __restrict__ Pm, const Real * const __restrict__ Pp,
        const Real * const __restrict__ vel,
        Real * const __restrict__ sumG, Real * const __restrict__ sumP, Real * const __restrict__ divU)
{
    const uint_t ix = blockIdx.x * _TILE_DIM_ + threadIdx.x;
    const uint_t iy = blockIdx.y * _TILE_DIM_ + threadIdx.y;

    // limiting resource
    __shared__ Real smem1[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem2[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem3[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem4[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem5[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem6[_TILE_DIM_][_TILE_DIM_+1];

    if (ix < NX && iy < NY)
    {
        // transpose
        const uint_t iyT = blockIdx.y * _TILE_DIM_ + threadIdx.x;
        const uint_t ixT = blockIdx.x * _TILE_DIM_ + threadIdx.y;

        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                const uint_t idxm = ID3(iyT,ixT+i,iz,NY,NXP1);
                const uint_t idxp = ID3(iyT,(ixT+1)+i,iz,NY,NXP1);

                // pre-fetch
                Real _sumG = Gp[idxm];
                Real _sumP = Pp[idxm];
                Real _divU = vel[idxp];
                _sumG = _sumG + Gm[idxp];
                _sumP = _sumP + Pm[idxp];
                _divU = _divU - vel[idxm];
                // read first batch
                smem1[threadIdx.x][threadIdx.y+i] = _sumG;
                smem2[threadIdx.x][threadIdx.y+i] = _sumP;
                smem3[threadIdx.x][threadIdx.y+i] = _divU;
            }
            __syncthreads();

            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                const uint_t idxm = ID3(iyT,ixT+i,iz,NY,NXP1);
                const uint_t idxp = ID3(iyT,(ixT+1)+i,iz,NY,NXP1);
                const uint_t idx = ID3(ix,iy+i,iz,NX,NY);

                // pre-fetch
                Real _divFr = flux.r[idxp];
                Real _divFu = flux.u[idxp];
                Real _divFv = flux.v[idxp];
                _divFr = _divFr - flux.r[idxm];
                _divFu = _divFu - flux.u[idxm];
                _divFv = _divFv - flux.v[idxm];
                // write first batch
                sumG[idx] = smem1[threadIdx.y+i][threadIdx.x];
                sumP[idx] = smem2[threadIdx.y+i][threadIdx.x];
                divU[idx] = smem3[threadIdx.y+i][threadIdx.x];
                // read second batch
                smem4[threadIdx.x][threadIdx.y+i] = _divFr;
                smem5[threadIdx.x][threadIdx.y+i] = _divFu;
                smem6[threadIdx.x][threadIdx.y+i] = _divFv;
            }
            __syncthreads();

            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                const uint_t idxm = ID3(iyT,ixT+i,iz,NY,NXP1);
                const uint_t idxp = ID3(iyT,(ixT+1)+i,iz,NY,NXP1);
                const uint_t idx = ID3(ix,iy+i,iz,NX,NY);

                // pre-fetch
                Real _divFw = flux.w[idxp];
                Real _divFe = flux.e[idxp];
                Real _divFG = flux.G[idxp];
                _divFw = _divFw - flux.w[idxm];
                _divFe = _divFe - flux.e[idxm];
                _divFG = _divFG - flux.G[idxm];
                // write second batch
                divF.r[idx] = smem4[threadIdx.y+i][threadIdx.x];
                divF.u[idx] = smem5[threadIdx.y+i][threadIdx.x];
                divF.v[idx] = smem6[threadIdx.y+i][threadIdx.x];
                // read third batch
                smem1[threadIdx.x][threadIdx.y+i] = _divFw;
                smem2[threadIdx.x][threadIdx.y+i] = _divFe;
                smem3[threadIdx.x][threadIdx.y+i] = _divFG;
            }
            __syncthreads();

            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                const uint_t idxm = ID3(iyT,ixT+i,iz,NY,NXP1);
                const uint_t idxp = ID3(iyT,(ixT+1)+i,iz,NY,NXP1);
                const uint_t idx = ID3(ix,iy+i,iz,NX,NY);

                // pre-fetch
                Real _divFP = flux.P[idxp];
                _divFP = _divFP - flux.P[idxm];
                // write third batch
                divF.w[idx] = smem1[threadIdx.y+i][threadIdx.x];
                divF.e[idx] = smem2[threadIdx.y+i][threadIdx.x];
                divF.G[idx] = smem3[threadIdx.y+i][threadIdx.x];
                // read fourth batch
                smem4[threadIdx.x][threadIdx.y+i] = _divFP;
            }
            __syncthreads();

            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                const uint_t idx = ID3(ix,iy+i,iz,NX,NY);
                // write fourth batch
                divF.P[idx] = smem4[threadIdx.y+i][threadIdx.x];
            }
            // NOTE: __syncthreads() can be omitted since it will not be
            // touched until next synchronization point
        }
    }
}


__global__
void _yextraterm_hllc(const uint_t nslices, DevicePointer divF, DevicePointer flux,
        const Real * const __restrict__ Gm, const Real * const __restrict__ Gp,
        const Real * const __restrict__ Pm, const Real * const __restrict__ Pp,
        const Real * const __restrict__ vel,
        Real * const __restrict__ sumG, Real * const __restrict__ sumP, Real * const __restrict__ divU)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx  = ID3(ix,iy,iz,NX,NY);
            const uint_t idxm = ID3(ix,iy,iz,NX,NYP1);
            const uint_t idxp = ID3(ix,(iy+1),iz,NX,NYP1);

            Real _sumG = Gp[idxm];
            Real _sumP = Pp[idxm];
            Real _divU = vel[idxp];
            Real _divFr = flux.r[idxp];
            Real _divFu = flux.u[idxp];
            Real _divFv = flux.v[idxp];
            Real _divFw = flux.w[idxp];
            Real _divFe = flux.e[idxp];
            Real _divFG = flux.G[idxp];
            Real _divFP = flux.P[idxp];
            _sumG = _sumG + Gm[idxp];
            _sumP = _sumP + Pm[idxp];
            _divU = _divU - vel[idxm];
            _divFr = _divFr - flux.r[idxm];
            _divFu = _divFu - flux.u[idxm];
            _divFv = _divFv - flux.v[idxm];
            _divFw = _divFw - flux.w[idxm];
            _divFe = _divFe - flux.e[idxm];
            _divFG = _divFG - flux.G[idxm];
            _divFP = _divFP - flux.P[idxm];

            sumG[idx] += _sumG;
            sumP[idx] += _sumP;
            divU[idx] += _divU;
            divF.r[idx] += _divFr;
            divF.u[idx] += _divFu;
            divF.v[idx] += _divFv;
            divF.w[idx] += _divFw;
            divF.e[idx] += _divFe;
            divF.G[idx] += _divFG;
            divF.P[idx] += _divFP;
        }
    }
}


__global__
void _zextraterm_hllc(const uint_t nslices, DevicePointer divF, DevicePointer flux,
        const Real * const __restrict__ Gm, const Real * const __restrict__ Gp,
        const Real * const __restrict__ Pm, const Real * const __restrict__ Pp,
        const Real * const __restrict__ vel,
        Real * const __restrict__ sumG, Real * const __restrict__ sumP, Real * const __restrict__ divU)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx  = ID3(ix,iy,iz,NX,NY);
            const uint_t idxm = ID3(ix,iy,iz,NX,NY);
            const uint_t idxp = ID3(ix,iy,(iz+1),NX,NY);

            const Real inv6 = 1.0f/6.0f;

            // cummulative sums of x and y
            const Real cumm_sumG = sumG[idx];
            const Real cumm_sumP = sumP[idx];
            const Real cumm_divU = divU[idx];

            Real _sumG = Gp[idxm];
            Real _sumP = Pp[idxm];
            Real _divU = vel[idxp];
            Real _divFr = flux.r[idxp];
            Real _divFu = flux.u[idxp];
            Real _divFv = flux.v[idxp];
            Real _divFw = flux.w[idxp];
            Real _divFe = flux.e[idxp];
            Real _divFG = flux.G[idxp];
            Real _divFP = flux.P[idxp];
            _sumG = _sumG + Gm[idxp] + cumm_sumG;
            _sumP = _sumP + Pm[idxp] + cumm_sumP;
            _divU = _divU - vel[idxm]+ cumm_divU;
            _divFr = _divFr - flux.r[idxm];
            _divFu = _divFu - flux.u[idxm];
            _divFv = _divFv - flux.v[idxm];
            _divFw = _divFw - flux.w[idxm];
            _divFe = _divFe - flux.e[idxm];
            _divFG = _divFG - flux.G[idxm];
            _divFP = _divFP - flux.P[idxm];

            // final divF
            divF.r[idx] += _divFr;
            divF.u[idx] += _divFu;
            divF.v[idx] += _divFv;
            divF.w[idx] += _divFw;
            divF.e[idx] += _divFe;
            divF.G[idx] += _divFG - inv6*_divU*_sumG;
            divF.P[idx] += _divFP - inv6*_divU*_sumP;
        }
    }
}


__global__
void _xflux00(const uint_t nslices, const uint_t global_iz,
        const DevicePointer ghostL, const DevicePointer ghostR, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NXP1 = NX + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * 6.) Reads textures 00
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    assert(NXP1 > 5);

    if (ix < NXP1 && iy < NY)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            // GMEM transactions are cached, effective GMEM accesses are 7*3
            // (according to nvvp)
            if (0 == ix)
                _load_3X00<0,0,3,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(3*GMEM + 3*tex_start)
            else if (1 == ix)
                _load_2X00<0,0,2,1>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(2*GMEM + 4*tex_start)
            else if (2 == ix)
                _load_1X00<0,0,1,2>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(1*GMEM + 5*tex_start)
            else if (NXP1-3 == ix)
                _load_1X00<NXP1-6,5,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NXP1-2 == ix)
                _load_2X00<NXP1-5,4,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NXP1-1 == ix)
                _load_3X00<NXP1-4,3,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else
                _load_internal_X00(ix, iy, iz, r, u, v, w, e, G, P, global_iz, NULL); // load 7*(6*tex_start)

            // compute body
#           include "xflux_body.cu"

            /* if (global_iz) */
            /* { */
/* #pragma unroll 6 */
            /*     for (uint_t i = 0; i < 6; ++i) */
            /*     { */
            /*         r[0] += r[i]; */
            /*         u[0] += u[i]; */
            /*         v[0] += v[i]; */
            /*         w[0] += w[i]; */
            /*         e[0] += e[i]; */
            /*         G[0] += G[i]; */
            /*         P[0] += P[i]; */
            /*     } */
            /* } */
            /* const uint_t idx = ID3(iy, ix, iz-3, NY, NXP1); */
            /* flux.r[idx] = r[0]; */
            /* flux.u[idx] = u[0]; */
            /* flux.v[idx] = v[0]; */
            /* flux.w[idx] = w[0]; */
            /* flux.e[idx] = e[0]; */
            /* flux.G[idx] = G[0]; */
            /* flux.P[idx] = P[0]; */
            /* xtra_vel[idx] = r[0]; */
            /* xtra_Gm[idx]  = w[0]; */
            /* xtra_Gp[idx]  = e[0]; */
            /* xtra_Pm[idx]  = P[0]; */
            /* xtra_Pp[idx]  = u[0]; */

            const uint_t idx = ID3(iy, ix, iz-3, NY, NXP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _xflux01(const uint_t nslices, const uint_t global_iz,
        const DevicePointer ghostL, const DevicePointer ghostR, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NXP1 = NX + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * 6.) Reads textures 01
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    assert(NXP1 > 5);

    if (ix < NXP1 && iy < NY)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            // GMEM transactions are cached, effective GMEM accesses are 7*3
            // (according to nvvp)
            if (0 == ix)
                _load_3X01<0,0,3,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(3*GMEM + 3*tex_start)
            else if (1 == ix)
                _load_2X01<0,0,2,1>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(2*GMEM + 4*tex_start)
            else if (2 == ix)
                _load_1X01<0,0,1,2>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(1*GMEM + 5*tex_start)
            else if (NXP1-3 == ix)
                _load_1X01<NXP1-6,5,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NXP1-2 == ix)
                _load_2X01<NXP1-5,4,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NXP1-1 == ix)
                _load_3X01<NXP1-4,3,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else
                _load_internal_X01(ix, iy, iz, r, u, v, w, e, G, P, global_iz, NULL); // load 7*(6*tex_start)

            // compute body
#           include "xflux_body.cu"

            const uint_t idx = ID3(iy, ix, iz-3, NY, NXP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _yflux00(const uint_t nslices, const uint_t global_iz,
        const DevicePointer ghostL, const DevicePointer ghostR, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NYP1 = NY + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * 6.) Reads texture 00
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    assert(NYP1 > 5);

    if (ix < NX && iy < NYP1)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            // GMEM transactions are cached, effective GMEM accesses are 7*3
            // (according to nvvp)
            if (0 == iy)
                _load_3Y00<0,0,3,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(3*GMEM + 3*TEX)
            else if (1 == iy)
                _load_2Y00<0,0,2,1>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(2*GMEM + 4*TEX)
            else if (2 == iy)
                _load_1Y00<0,0,1,2>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(1*GMEM + 5*TEX)
            else if (NYP1-3 == iy)
                _load_1Y00<NYP1-6,5,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NYP1-2 == iy)
                _load_2Y00<NYP1-5,4,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NYP1-1 == iy)
                _load_3Y00<NYP1-4,3,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else
                _load_internal_Y00(ix, iy, iz, r, u, v, w, e, G, P, global_iz, NULL); // load 7*(6*TEX)

            // compute body
#           include "yflux_body.cu"

            /* if (global_iz) */
            /* { */
/* #pragma unroll 6 */
            /*     for (uint_t i = 0; i < 6; ++i) */
            /*     { */
            /*         r[0] += r[i]; */
            /*         u[0] += u[i]; */
            /*         v[0] += v[i]; */
            /*         w[0] += w[i]; */
            /*         e[0] += e[i]; */
            /*         G[0] += G[i]; */
            /*         P[0] += P[i]; */
            /*     } */
            /* } */
            /* const uint_t idx = ID3(ix, iy, iz-3, NX, NYP1); */
            /* flux.r[idx] = r[0]; */
            /* flux.u[idx] = u[0]; */
            /* flux.v[idx] = v[0]; */
            /* flux.w[idx] = w[0]; */
            /* flux.e[idx] = e[0]; */
            /* flux.G[idx] = G[0]; */
            /* flux.P[idx] = P[0]; */
            /* xtra_vel[idx] = r[0]; */
            /* xtra_Gm[idx]  = w[0]; */
            /* xtra_Gp[idx]  = e[0]; */
            /* xtra_Pm[idx]  = P[0]; */
            /* xtra_Pp[idx]  = u[0]; */

            const uint_t idx = ID3(ix, iy, iz-3, NX, NYP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _yflux01(const uint_t nslices, const uint_t global_iz,
        const DevicePointer ghostL, const DevicePointer ghostR, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NYP1 = NY + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * 6.) Reads texture 01
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    assert(NYP1 > 5);

    if (ix < NX && iy < NYP1)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            // GMEM transactions are cached, effective GMEM accesses are 7*3
            // (according to nvvp)
            if (0 == iy)
                _load_3Y01<0,0,3,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(3*GMEM + 3*TEX)
            else if (1 == iy)
                _load_2Y01<0,0,2,1>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(2*GMEM + 4*TEX)
            else if (2 == iy)
                _load_1Y01<0,0,1,2>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(1*GMEM + 5*TEX)
            else if (NYP1-3 == iy)
                _load_1Y01<NYP1-6,5,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NYP1-2 == iy)
                _load_2Y01<NYP1-5,4,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NYP1-1 == iy)
                _load_3Y01<NYP1-4,3,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else
                _load_internal_Y01(ix, iy, iz, r, u, v, w, e, G, P, global_iz, NULL); // load 7*(6*TEX)

            // compute body
#           include "yflux_body.cu"

            const uint_t idx = ID3(ix, iy, iz-3, NX, NYP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _zflux00(const uint_t nslices, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NX = NodeBlock::sizeX
     * 2.) NY = NodeBlock::sizeY
     * 3.) NZ = NodeBlock::sizeZ
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) Reads texture 00
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    // depends on boundary condition in z-direction
    assert(NodeBlock::sizeZ > 0);

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 3; iz < (nslices+1)+3; ++iz) // first and last 3 slices are zghosts; need to compute nslices+1 fluxes in z-direction
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            _load_internal_Z00(ix, iy, iz, r, u, v, w, e, G, P, 0, NULL); // load 7*(6*TEX)

            // compute body
#           include "zflux_body.cu"

            /* if (global_iz) */
            /* { */
/* #pragma unroll 6 */
            /*     for (uint_t i = 0; i < 6; ++i) */
            /*     { */
            /*         r[0] += r[i]; */
            /*         u[0] += u[i]; */
            /*         v[0] += v[i]; */
            /*         w[0] += w[i]; */
            /*         e[0] += e[i]; */
            /*         G[0] += G[i]; */
            /*         P[0] += P[i]; */
            /*     } */
            /* } */
            /* const uint_t idx = ID3(ix, iy, iz-3, NX, NY); */
            /* flux.r[idx] = r[0]; */
            /* flux.u[idx] = u[0]; */
            /* flux.v[idx] = v[0]; */
            /* flux.w[idx] = w[0]; */
            /* flux.e[idx] = e[0]; */
            /* flux.G[idx] = G[0]; */
            /* flux.P[idx] = P[0]; */
            /* xtra_vel[idx] = r[0]; */
            /* xtra_Gm[idx]  = w[0]; */
            /* xtra_Gp[idx]  = e[0]; */
            /* xtra_Pm[idx]  = P[0]; */
            /* xtra_Pp[idx]  = u[0]; */

            const uint_t idx = ID3(ix, iy, iz-3, NX, NY);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _zflux01(const uint_t nslices, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NX = NodeBlock::sizeX
     * 2.) NY = NodeBlock::sizeY
     * 3.) NZ = NodeBlock::sizeZ
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) Reads texture 01
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    // depends on boundary condition in z-direction
    assert(NodeBlock::sizeZ > 0);

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 3; iz < (nslices+1)+3; ++iz) // first and last 3 slices are zghosts; need to compute nslices+1 fluxes in z-direction
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            _load_internal_Z01(ix, iy, iz, r, u, v, w, e, G, P, 0, NULL); // load 7*(6*TEX)

            // compute body
#           include "zflux_body.cu"

            const uint_t idx = ID3(ix, iy, iz-3, NX, NY);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _maxSOS(const uint_t nslices, int* g_maxSOS)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    const uint_t loc_idx = blockDim.x * threadIdx.y + threadIdx.x;
    __shared__ Real block_sos[_NTHREADS_];
    block_sos[loc_idx] = 0.0f;

    if (ix < NX && iy < NY)
    {
        Real sos = 0.0f;

        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            // TODO: used both buffers here
            const Real r = tex3D(texR00, ix, iy, iz);
            const Real u = tex3D(texU00, ix, iy, iz);
            const Real v = tex3D(texV00, ix, iy, iz);
            const Real w = tex3D(texW00, ix, iy, iz);
            const Real e = tex3D(texE00, ix, iy, iz);
            const Real G = tex3D(texG00, ix, iy, iz);
            const Real P = tex3D(texP00, ix, iy, iz);

            const Real p = (e - (u*u + v*v + w*w)*(0.5f/r) - P) / G;
            const Real c = sqrtf(((p + P) / G + p) / r);

            sos = fmaxf(sos, c + fmaxf(fmaxf(fabsf(u), fabsf(v)), fabsf(w)) / r);
        }
        block_sos[loc_idx] = sos;
        __syncthreads();

        if (0 == loc_idx)
        {
            for (int i = 1; i < _NTHREADS_; ++i)
                sos = fmaxf(sos, block_sos[i]);
            assert(sos > 0.0f);
            atomicMax(g_maxSOS, __float_as_int(sos));
        }
    }
}

///////////////////////////////////////////////////////////////////////////////
//                              KERNEL WRAPPERS                              //
///////////////////////////////////////////////////////////////////////////////

void _TEST_dump(const Real * const d_data, const size_t bytes, const string fname = "data.bin")
{
    Real *h_data = (Real *)malloc(bytes);
    hipMemcpy(h_data, d_data, bytes, hipMemcpyDeviceToHost);
    ofstream out(fname.c_str(), std::ofstream::binary);
    out.write((char *)h_data, bytes);
    out.close();
    free(h_data);
}

static void _bindTexture(texture<float, 3, hipReadModeElementType> * const tex, hipArray_t d_ptr)
{
    hipChannelFormatDesc fmt = hipCreateChannelDesc<Real>();
    tex->addressMode[0]       = hipAddressModeClamp;
    tex->addressMode[1]       = hipAddressModeClamp;
    tex->addressMode[2]       = hipAddressModeClamp;
    tex->channelDesc          = fmt;
    tex->filterMode           = hipFilterModePoint;
    tex->mipmapFilterMode     = hipFilterModePoint;
    tex->normalized           = false;
    hipBindTextureToArray(tex, d_ptr, &fmt);
}


void GPU::compute_pipe_divF(const uint_t nslices, const uint_t global_iz,
        const uint_t gbuf_id, const int chunk_id)
{
    assert(gbuf_id < _NUM_GPU_BUF_);

    /* *
     * Compute div(F)
     * */

    // my stream
    const uint_t s_id = chunk_id % _NUM_STREAMS_;

    // my data
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    // my input/output
    DevicePointer inout(mybuf->d_inout);

    // previous stream has priority
    const uint_t s_idm1 = ((chunk_id-1) + _NUM_STREAMS_) % _NUM_STREAMS_;
    assert(s_idm1 < _NUM_STREAMS_);
    hipStreamWaitEvent(stream[s_id], event_compute[s_idm1], 0);

    char prof_item[256];

    // before we do anything, we convert to primitive variables and prepare
    // texture buffers
    const dim3 CONV_blocks(_WARPSIZE_, 4, 1);
    const dim3 CONV_grid((NX + _WARPSIZE_ - 1)/_WARPSIZE_, (NY + 4 - 1)/4, nslices+6);

    sprintf(prof_item, "_CONV (%d)", s_id);
    GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
    _CONV<<<CONV_grid, CONV_blocks, 0, stream[s_id]>>>(inout);
    GPU::profiler.pop_stopCUDA();

    // TODO: REMOVE THIS
    /* sprintf(prof_item, "_TEST_CONV (%d)", s_id); */
    /* GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /* _TEST_CONV<<<1,1,0,stream[s_id]>>>(inout, xghostL, xghostR, yghostL, yghostR); */
    /* GPU::profiler.pop_stopCUDA(); */

    // copy to tex buffers
    for (uint_t i = 0; i < VSIZE; ++i)
    {
        hipMemcpy3DParms copyParams = {0};
        copyParams.extent            = make_hipExtent(NX, NY, nslices+6);
        copyParams.kind              = hipMemcpyDeviceToDevice;
        copyParams.srcPtr            = make_hipPitchedPtr((void *)mybuf->d_inout[i], NX * sizeof(Real), NX, NY);
        copyParams.dstArray          = mybuf->d_GPU3D[i];
        hipMemcpy3DAsync(&copyParams, stream[s_id]);
    }
    _bindTexture(&tex00, mybuf->d_GPU3D[0]);
    _bindTexture(&tex01, mybuf->d_GPU3D[1]);
    _bindTexture(&tex02, mybuf->d_GPU3D[2]);
    _bindTexture(&tex03, mybuf->d_GPU3D[3]);
    _bindTexture(&tex04, mybuf->d_GPU3D[4]);
    _bindTexture(&tex05, mybuf->d_GPU3D[5]);
    _bindTexture(&tex06, mybuf->d_GPU3D[6]);

    // my reconstruction
    DevicePointer recon_m(d_recon_m);
    DevicePointer recon_p(d_recon_p);

    // ========================================================================
    // X
    // ========================================================================
    // TODO: check that nslices is an integer multiple of 4 (or 1)
    const dim3 X_blocks(1, _WARPSIZE_, 4);
    const dim3 X_grid(NXP1, (NY + _WARPSIZE_ - 1)/_WARPSIZE_, (nslices + 4 - 1)/4);

    // reconstruct
    _WENO_X<0><<<X_grid, X_blocks, 0, stream[s_id]>>>(d_recon_m[0], d_recon_p[0], mybuf->d_xgl[0], mybuf->d_xgr[0]);
    _WENO_X<1><<<X_grid, X_blocks, 0, stream[s_id]>>>(d_recon_m[1], d_recon_p[1], mybuf->d_xgl[1], mybuf->d_xgr[1]);
    _WENO_X<2><<<X_grid, X_blocks, 0, stream[s_id]>>>(d_recon_m[2], d_recon_p[2], mybuf->d_xgl[2], mybuf->d_xgr[2]);
    _WENO_X<3><<<X_grid, X_blocks, 0, stream[s_id]>>>(d_recon_m[3], d_recon_p[3], mybuf->d_xgl[3], mybuf->d_xgr[3]);
    _WENO_X<4><<<X_grid, X_blocks, 0, stream[s_id]>>>(d_recon_m[4], d_recon_p[4], mybuf->d_xgl[4], mybuf->d_xgr[4]);
    _WENO_X<5><<<X_grid, X_blocks, 0, stream[s_id]>>>(d_recon_m[5], d_recon_p[5], mybuf->d_xgl[5], mybuf->d_xgr[5]);
    _WENO_X<6><<<X_grid, X_blocks, 0, stream[s_id]>>>(d_recon_m[6], d_recon_p[6], mybuf->d_xgl[6], mybuf->d_xgr[6]);

    // hllc fluxes
    _HLLC_X<<<X_grid, X_blocks, 0, stream[s_id]>>>(recon_m, recon_p);

    // flux divegence X + extra term contribution
    const dim3 X_xtraBlocks(_TILE_DIM_, _BLOCK_ROWS_, 1);
    const dim3 X_xtraGrid((NX + _TILE_DIM_ - 1)/_TILE_DIM_, (NY + _TILE_DIM_ - 1)/_TILE_DIM_, 1);
    _xextraterm_hllc<<<X_xtraGrid, X_xtraBlocks, 0, stream[s_id]>>>(nslices, inout, recon_m, recon_p.r, recon_p.u, recon_p.v, recon_p.w, recon_p.e, d_sumG, d_sumP, d_divU);

    hipDeviceSynchronize();
    /* std::exit(3); */

    _TEST_dump(inout.u, 256*256*256*sizeof(Real), "split_xrhs.u.bin");


    // my ghosts TODO: don't need them
    DevicePointer xghostL(mybuf->d_xgl);
    DevicePointer xghostR(mybuf->d_xgr);
    DevicePointer yghostL(mybuf->d_ygl);
    DevicePointer yghostR(mybuf->d_ygr);



    /* // my launch config */
    /* const dim3 X_blocks(1, _NTHREADS_, 1); */
    /* const dim3 X_grid(NXP1, (NY + _NTHREADS_ -1)/_NTHREADS_, 1); */
    /* const dim3 X_xtraBlocks(_TILE_DIM_, _BLOCK_ROWS_, 1); */
    /* const dim3 X_xtraGrid((NX + _TILE_DIM_ - 1)/_TILE_DIM_, (NY + _TILE_DIM_ - 1)/_TILE_DIM_, 1); */

    /* const dim3 Y_blocks(_NTHREADS_, 1, 1); */
    /* const dim3 Y_grid((NX + _NTHREADS_ -1) / _NTHREADS_, NYP1, 1); */
    /* const dim3 Y_xtraGrid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1); */

    /* const dim3 Z_blocks(_NTHREADS_, 1, 1); */
    /* const dim3 Z_grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1); */



    /* // queue kernels in pipe */
    /* switch (gbuf_id) */
    /* { */
    /*     case 0: */
    /*         _bindTexture(&texR00, mybuf->d_GPUin[0]); */
    /*         _bindTexture(&texU00, mybuf->d_GPUin[1]); */
    /*         _bindTexture(&texV00, mybuf->d_GPUin[2]); */
    /*         _bindTexture(&texW00, mybuf->d_GPUin[3]); */
    /*         _bindTexture(&texE00, mybuf->d_GPUin[4]); */
    /*         _bindTexture(&texG00, mybuf->d_GPUin[5]); */
    /*         _bindTexture(&texP00, mybuf->d_GPUin[6]); */
    /*         // --- X --- */
    /*         sprintf(prof_item, "_XFLUX (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _xflux00<<<X_grid, X_blocks, 0, stream[s_id]>>>(nslices, global_iz, xghostL, xghostR, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp); */
    /*         GPU::profiler.pop_stopCUDA(); */

    /*         sprintf(prof_item, "_XEXTRATERM (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _xextraterm_hllc<<<X_xtraGrid, X_xtraBlocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU); */
    /*         GPU::profiler.pop_stopCUDA(); */

    /*         // --- Y --- */
    /*         sprintf(prof_item, "_YFLUX (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _yflux00<<<Y_grid, Y_blocks, 0, stream[s_id]>>>(nslices, global_iz, yghostL, yghostR, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp); */
    /*         GPU::profiler.pop_stopCUDA(); */

    /*         sprintf(prof_item, "_YEXTRATERM (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _yextraterm_hllc<<<Y_xtraGrid, Y_blocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU); */
    /*         GPU::profiler.pop_stopCUDA(); */

    /*         // --- Z --- */
    /*         sprintf(prof_item, "_ZFLUX (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _zflux00<<<Z_grid, Z_blocks, 0, stream[s_id]>>>(nslices, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp); */
    /*         GPU::profiler.pop_stopCUDA(); */

    /*         sprintf(prof_item, "_ZEXTRATERM (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _zextraterm_hllc<<<Z_grid, Z_blocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU); */
    /*         GPU::profiler.pop_stopCUDA(); */
    /*         break; */

    /*     case 1: */
    /*         _bindTexture(&texR01, mybuf->d_GPUin[0]); */
    /*         _bindTexture(&texU01, mybuf->d_GPUin[1]); */
    /*         _bindTexture(&texV01, mybuf->d_GPUin[2]); */
    /*         _bindTexture(&texW01, mybuf->d_GPUin[3]); */
    /*         _bindTexture(&texE01, mybuf->d_GPUin[4]); */
    /*         _bindTexture(&texG01, mybuf->d_GPUin[5]); */
    /*         _bindTexture(&texP01, mybuf->d_GPUin[6]); */
    /*         // --- X --- */
    /*         sprintf(prof_item, "_XFLUX (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _xflux01<<<X_grid, X_blocks, 0, stream[s_id]>>>(nslices, global_iz, xghostL, xghostR, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp); */
    /*         GPU::profiler.pop_stopCUDA(); */

    /*         sprintf(prof_item, "_XEXTRATERM (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _xextraterm_hllc<<<X_xtraGrid, X_xtraBlocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU); */
    /*         GPU::profiler.pop_stopCUDA(); */

    /*         // --- Y --- */
    /*         sprintf(prof_item, "_YFLUX (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _yflux01<<<Y_grid, Y_blocks, 0, stream[s_id]>>>(nslices, global_iz, yghostL, yghostR, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp); */
    /*         GPU::profiler.pop_stopCUDA(); */

    /*         sprintf(prof_item, "_YEXTRATERM (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _yextraterm_hllc<<<Y_xtraGrid, Y_blocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU); */
    /*         GPU::profiler.pop_stopCUDA(); */

    /*         // --- Z --- */
    /*         sprintf(prof_item, "_ZFLUX (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _zflux01<<<Z_grid, Z_blocks, 0, stream[s_id]>>>(nslices, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp); */
    /*         GPU::profiler.pop_stopCUDA(); */

    /*         sprintf(prof_item, "_ZEXTRATERM (s_id=%d)", s_id); */
    /*         GPU::profiler.push_startCUDA(prof_item, &stream[s_id]); */
    /*         _zextraterm_hllc<<<Z_grid, Z_blocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU); */
    /*         GPU::profiler.pop_stopCUDA(); */
    /*         break; */
    /* } */

    /* hipEventRecord(event_compute[s_id], stream[s_id]); */
}


void GPU::MaxSpeedOfSound(const uint_t nslices, const uint_t gbuf_id, const int chunk_id)
{
    assert(gbuf_id < _NUM_GPU_BUF_);

    // my stream
    const uint_t s_id = chunk_id % _NUM_STREAMS_;

    // my data
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    _bindTexture(&texR00, mybuf->d_GPU3D[0]);
    _bindTexture(&texU00, mybuf->d_GPU3D[1]);
    _bindTexture(&texV00, mybuf->d_GPU3D[2]);
    _bindTexture(&texW00, mybuf->d_GPU3D[3]);
    _bindTexture(&texE00, mybuf->d_GPU3D[4]);
    _bindTexture(&texG00, mybuf->d_GPU3D[5]);
    _bindTexture(&texP00, mybuf->d_GPU3D[6]);

    // my launch config
    const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
    const dim3 blocks(_NTHREADS_, 1, 1);

    char prof_item[256];

    sprintf(prof_item, "_MAXSOS (%d)", s_id);
    GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
    _maxSOS<<<grid, blocks, 0, stream[s_id]>>>(nslices, d_maxSOS);
    GPU::profiler.pop_stopCUDA();
}

///////////////////////////////////////////////////////////////////////////
// TEST SECTION
///////////////////////////////////////////////////////////////////////////
void GPU::TestKernel()
{
    const uint_t gbuf_id = 0;
    const uint_t s_id = 0;

    // my data
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    _bindTexture(&texR00, mybuf->d_GPU3D[0]);
    _bindTexture(&texU00, mybuf->d_GPU3D[1]);
    _bindTexture(&texV00, mybuf->d_GPU3D[2]);
    _bindTexture(&texW00, mybuf->d_GPU3D[3]);
    _bindTexture(&texE00, mybuf->d_GPU3D[4]);
    _bindTexture(&texG00, mybuf->d_GPU3D[5]);
    _bindTexture(&texP00, mybuf->d_GPU3D[6]);

    /* // my ghosts */
    /* DevicePointer xghostL(mybuf->d_xgl); */
    /* DevicePointer xghostR(mybuf->d_xgr); */
    /* DevicePointer yghostL(mybuf->d_ygl); */
    /* DevicePointer yghostR(mybuf->d_ygr); */

    /* // my output */
    /* DevicePointer divF(mybuf->d_divF); */

    /* // my tmp storage */
    /* DevicePointer flux(d_flux); */

    /* hipFree(d_sumG); */
    /* hipFree(d_sumP); */
    /* hipFree(d_divU); */

    /* const uint_t nslices = NodeBlock::sizeZ; */
    /* const uint_t xflxSize = (NodeBlock::sizeX+1)*NodeBlock::sizeY*nslices; */
    /* const uint_t yflxSize = NodeBlock::sizeX*(NodeBlock::sizeY+1)*nslices; */
    /* const uint_t zflxSize = NodeBlock::sizeX*NodeBlock::sizeY*(nslices+1); */

    /* Real *d_extra_X[5]; */
    /* Real *d_extra_Y[5]; */
    /* Real *d_extra_Z[5]; */
    /* for (int i = 0; i < 5; ++i) */
    /* { */
    /*     hipMalloc(&(d_extra_X[i]), xflxSize * sizeof(Real)); */
    /*     hipMalloc(&(d_extra_Y[i]), yflxSize * sizeof(Real)); */
    /*     hipMalloc(&(d_extra_Z[i]), zflxSize * sizeof(Real)); */
    /* } */
    /* GPU::tell_memUsage_GPU(); */


    /* { */

    /*     const dim3 xblocks(1, _NTHREADS_, 1); */
    /*     const dim3 yblocks(_NTHREADS_, 1, 1); */
    /*     const dim3 zblocks(_NTHREADS_, 1, 1); */
    /*     const dim3 xgrid(NXP1, (NY + _NTHREADS_ - 1) / _NTHREADS_,   1); */
    /*     const dim3 ygrid((NX   + _NTHREADS_ - 1) / _NTHREADS_, NYP1, 1); */
    /*     const dim3 zgrid((NX   + _NTHREADS_ - 1) / _NTHREADS_, NY,   1); */

    /*     GPU::profiler.push_startCUDA("_XFLUX", &stream[s_id]); */
    /*     _xflux00<<<xgrid, xblocks, 0, stream[s_id]>>>(nslices, 0, xghostL, xghostR, flux, d_extra_X[0], d_extra_X[1], d_extra_X[2], d_extra_X[3], d_extra_X[4]); */
    /*     GPU::profiler.pop_stopCUDA(); */

    /*     /1* GPU::profiler.push_startCUDA("_YFLUX", &_s[0]); *1/ */
    /*     /1* _yflux<<<ygrid, yblocks, 0, _s[0]>>>(nslices, 0, yghostL, yghostR, flux, d_extra_Y[0], d_extra_Y[1], d_extra_Y[2], d_extra_Y[3], d_extra_Y[4]); *1/ */
    /*     /1* GPU::profiler.pop_stopCUDA(); *1/ */

    /*     /1* GPU::profiler.push_startCUDA("_ZFLUX", &_s[0]); *1/ */
    /*     /1* _zflux<<<zgrid, zblocks, 0, _s[0]>>>(nslices, flux, d_extra_Z[0], d_extra_Z[1], d_extra_Z[2], d_extra_Z[3], d_extra_Z[4]); *1/ */
    /*     /1* GPU::profiler.pop_stopCUDA(); *1/ */

    /*     /1* _xflux<<<xgrid, xblocks, 0, _s[0]>>>(nslices, 0, xghostL, xghostR, flux, d_extra_X[0], d_extra_X[1], d_extra_X[2], d_extra_X[3], d_extra_X[4]); *1/ */
    /*     /1* _yflux<<<ygrid, yblocks, 0, _s[1]>>>(nslices, 0, yghostL, yghostR, flux, d_extra_Y[0], d_extra_Y[1], d_extra_Y[2], d_extra_Y[3], d_extra_Y[4]); *1/ */
    /*     /1* _zflux<<<zgrid, zblocks, 0, _s[2]>>>(nslices, flux, d_extra_Z[0], d_extra_Z[1], d_extra_Z[2], d_extra_Z[3], d_extra_Z[4]); *1/ */

    /*     hipDeviceSynchronize(); */
    /* } */
}
