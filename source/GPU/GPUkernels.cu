#include "hip/hip_runtime.h"
/* *
 * GPUkernels.cu
 *
 * Created by Fabian Wermelinger on 6/25/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */
#include <assert.h>
#include <stdio.h>
#include <vector>

#include "GPU.h" // includes Types.h & wrapper declarations
#include "GPUonly.cuh"

#if _BLOCKSIZEX_ < 5
#error Minimum _BLOCKSIZEX_ is 5
#elif _BLOCKSIZEY_ < 5
#error Minimum _BLOCKSIZEY_ is 5
#elif _BLOCKSIZEZ_ < 1
#error Minimum _BLOCKSIZEZ_ is 1
#endif

#if NX % _TILE_DIM_ != 0
#error _BLOCKSIZEX_ should be an integer multiple of _TILE_DIM_
#endif
#if NY % _TILE_DIM_ != 0
#error _BLOCKSIZEY_ should be an integer multiple of _TILE_DIM_
#endif

///////////////////////////////////////////////////////////////////////////////
//                             DEVICE FUNCTIONS                              //
///////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////////////////////////////////////////
//                                  KERNELS                                  //
///////////////////////////////////////////////////////////////////////////////
#define _NFLUXES_ 3
#define _STENCIL_WIDTH_ 8 // 6 + _NFLUXES_ - 1

__global__
void _xextraterm_hllc(const uint_t nslices,
        const Real * const __restrict__ Gm, const Real * const __restrict__ Gp,
        const Real * const __restrict__ Pm, const Real * const __restrict__ Pp,
        const Real * const __restrict__ vel,
        Real * const __restrict__ sumG, Real * const __restrict__ sumP, Real * const __restrict__ divU)
{
    const uint_t ix = blockIdx.x * _TILE_DIM_ + threadIdx.x;
    const uint_t iy = blockIdx.y * _TILE_DIM_ + threadIdx.y;

    // limiting resource, but runs faster by using 3 buffers
    __shared__ Real smem1[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem2[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem3[_TILE_DIM_][_TILE_DIM_+1];

    if (ix < NX && iy < NY)
    {
        // transpose
        const uint_t iyT = blockIdx.y * _TILE_DIM_ + threadIdx.x;
        const uint_t ixT = blockIdx.x * _TILE_DIM_ + threadIdx.y;

        // per thread:
        // LOADS  = nslices * (6 * _TILE_DIM_/_BLOCK_ROWS_)
        // STORES = nslices * (3 * _TILE_DIM_/_BLOCK_ROWS_)
        // total words transferred per thread:
        // WORDS  = nslices * (9 * _TILE_DIM_/_BLOCK_ROWS_)
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                smem1[threadIdx.x][threadIdx.y+i] = Gp[ID3(iyT,ixT+i,iz,NY,NXP1)]      + Gm[ID3(iyT,(ixT+1)+i,iz,NY,NXP1)];
                smem2[threadIdx.x][threadIdx.y+i] = Pp[ID3(iyT,ixT+i,iz,NY,NXP1)]      + Pm[ID3(iyT,(ixT+1)+i,iz,NY,NXP1)];
                smem3[threadIdx.x][threadIdx.y+i] = vel[ID3(iyT,(ixT+1)+i,iz,NY,NXP1)] - vel[ID3(iyT,ixT+i,iz,NY,NXP1)];
            }
            __syncthreads();
            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                sumG[ID3(ix,iy+i,iz,NX,NY)] = smem1[threadIdx.y+i][threadIdx.x];
                sumP[ID3(ix,iy+i,iz,NX,NY)] = smem2[threadIdx.y+i][threadIdx.x];
                divU[ID3(ix,iy+i,iz,NX,NY)] = smem3[threadIdx.y+i][threadIdx.x];
            }
            __syncthreads();
        }
    }
}


__global__
void _yextraterm_hllc(const uint_t nslices,
        const Real * const __restrict__ Gm, const Real * const __restrict__ Gp,
        const Real * const __restrict__ Pm, const Real * const __restrict__ Pp,
        const Real * const __restrict__ vel,
        Real * const __restrict__ sumG, Real * const __restrict__ sumP, Real * const __restrict__ divU)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        // per thread:
        // LOADS  = nslices * 9
        // STORES = nslices * 3
        // total words transferred per thread:
        // WORDS  = nslices * 12
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx  = ID3(ix,iy,iz,NX,NY);
            const uint_t idxm = ID3(ix,iy,iz,NX,NYP1);
            const uint_t idxp = ID3(ix,(iy+1),iz,NX,NYP1);
            Real tq = Gp[idxm];
            Real tr = Pp[idxm];
            Real ts = vel[idxp];
            tq = tq + Gm[idxp];
            tr = tr + Pm[idxp];
            ts = ts - vel[idxm];
            sumG[idx] += tq;
            sumP[idx] += tr;
            divU[idx] += ts;
        }
    }
}


__global__
void _zextraterm_hllc(const uint_t nslices,
        const Real * const __restrict__ Gm, const Real * const __restrict__ Gp,
        const Real * const __restrict__ Pm, const Real * const __restrict__ Pp,
        const Real * const __restrict__ vel,
        Real * const __restrict__ sumG, Real * const __restrict__ sumP, Real * const __restrict__ divU)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        // per thread:
        // LOADS  = nslices * 9
        // STORES = nslices * 3
        // total words transferred per thread:
        // WORDS  = nslices * 12
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx  = ID3(ix,iy,iz,NX,NY);
            const uint_t idxm = ID3(ix,iy,iz,NX,NY);
            const uint_t idxp = ID3(ix,iy,(iz+1),NX,NY);
            Real tq = Gp[idxm];
            Real tr = Pp[idxm];
            Real ts = vel[idxp];
            tq = tq + Gm[idxp];
            tr = tr + Pm[idxp];
            ts = ts - vel[idxm];
            sumG[idx] += tq;
            sumP[idx] += tr;
            divU[idx] += ts;
        }
    }
}


__global__
void _xflux(const uint_t nslices, const uint_t global_iz,
        const DevicePointer ghostL, const DevicePointer ghostR, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NXP1 = NX + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    assert(NXP1 > 5);

    if (ix < NXP1 && iy < NY)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            // GMEM transactions are cached, effective GMEM accesses are 7*3
            // (according to nvvp)
            if (0 == ix)
                _load_3X<0,0,3,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(3*GMEM + 3*TEX)
            else if (1 == ix)
                _load_2X<0,0,2,1>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(2*GMEM + 4*TEX)
            else if (2 == ix)
                _load_1X<0,0,1,2>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(1*GMEM + 5*TEX)
            else if (NXP1-3 == ix)
                _load_1X<NXP1-6,5,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NXP1-2 == ix)
                _load_2X<NXP1-5,4,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NXP1-1 == ix)
                _load_3X<NXP1-4,3,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else
                _load_internal_X(ix, iy, iz, r, u, v, w, e, G, P, global_iz, NULL); // load 7*(6*TEX)

            // 2.)
            // convert to primitive variables
#pragma unroll 6
            for (uint_t i = 0; i < 6; ++i)
            {
                e[i] = (e[i] - 0.5f*(u[i]*u[i] + v[i]*v[i] + w[i]*w[i])/r[i] - P[i]) / G[i];
                u[i] = u[i]/r[i];
                v[i] = v[i]/r[i];
                w[i] = w[i]/r[i];
            } // 6 x (8 MUL/ADD/SUB + 5 DIV) = 78 FLOPS

///////////////////////////////////////////////////////////////////////////////
            // TEST MORE SOFTWARE PIPELINE //
            /* Real rm, rp, um, up, vm, vp, wm, wp, pm, pp, Gm, Gp, Pm, Pp; */
            /* _weno_reconstruction(rm, rp, r); */
            /* _weno_reconstruction(Gm, Gp, G); */
            /* _weno_reconstruction(Pm, Pp, P); */
            /* _weno_reconstruction(pm, pp, e); */
            /* _weno_reconstruction(um, up, u); */
            /* _weno_reconstruction(vm, vp, v); */
            /* _weno_reconstruction(wm, wp, w); */
            /* rm = _weno_clip_minus(rm, r[1], r[2], r[3]); */
            /* rp = _weno_clip_pluss(rp, r[2], r[3], r[4]); */
            /* Gm = _weno_clip_minus(Gm, G[1], G[2], G[3]); */
            /* Gp = _weno_clip_pluss(Gp, G[2], G[3], G[4]); */
            /* Pm = _weno_clip_minus(Pm, P[1], P[2], P[3]); */
            /* Pp = _weno_clip_pluss(Pp, P[2], P[3], P[4]); */
            /* pm = _weno_clip_minus(pm, e[1], e[2], e[3]); */
            /* pp = _weno_clip_pluss(pp, e[2], e[3], e[4]); */
            /* um = _weno_clip_minus(um, u[1], u[2], u[3]); */
            /* up = _weno_clip_pluss(up, u[2], u[3], u[4]); */
            /* vm = _weno_clip_minus(vm, v[1], v[2], v[3]); */
            /* vp = _weno_clip_pluss(vp, v[2], v[3], v[4]); */
            /* wm = _weno_clip_minus(wm, w[1], w[2], w[3]); */
            /* wp = _weno_clip_pluss(wp, w[2], w[3], w[4]); */
            /* assert(!isnan(rp)); assert(!isnan(rm)); */
            /* assert(!isnan(Gp)); assert(!isnan(Gm)); */
            /* assert(!isnan(Pp)); assert(!isnan(Pm)); */
            /* assert(!isnan(pp)); assert(!isnan(pm)); */
            /* assert(!isnan(up)); assert(!isnan(um)); */
            /* assert(!isnan(vp)); assert(!isnan(vm)); */
            /* assert(!isnan(wp)); assert(!isnan(wm)); */
///////////////////////////////////////////////////////////////////////////////

            const Real rm = _weno_minus_clipped(r[0], r[1], r[2], r[3], r[4]); // 96 FLOP (6 DIV)
            const Real rp = _weno_pluss_clipped(r[1], r[2], r[3], r[4], r[5]); // 96 FLOP (6 DIV)
            assert(!isnan(rp)); assert(!isnan(rm));

            const Real Gm = _weno_minus_clipped(G[0], G[1], G[2], G[3], G[4]); // 96 FLOP (6 DIV)
            const Real Gp = _weno_pluss_clipped(G[1], G[2], G[3], G[4], G[5]); // 96 FLOP (6 DIV)
            assert(!isnan(Gp)); assert(!isnan(Gm));

            const Real Pm = _weno_minus_clipped(P[0], P[1], P[2], P[3], P[4]); // 96 FLOP (6 DIV)
            const Real Pp = _weno_pluss_clipped(P[1], P[2], P[3], P[4], P[5]); // 96 FLOP (6 DIV)
            assert(!isnan(Pp)); assert(!isnan(Pm));

            const Real pm = _weno_minus_clipped(e[0], e[1], e[2], e[3], e[4]); // 96 FLOP (6 DIV)
            const Real pp = _weno_pluss_clipped(e[1], e[2], e[3], e[4], e[5]); // 96 FLOP (6 DIV)
            assert(!isnan(pp)); assert(!isnan(pm));

            const Real um = _weno_minus_clipped(u[0], u[1], u[2], u[3], u[4]); // 96 FLOP (6 DIV)
            const Real up = _weno_pluss_clipped(u[1], u[2], u[3], u[4], u[5]); // 96 FLOP (6 DIV)
            assert(!isnan(up)); assert(!isnan(um));

            const Real vm = _weno_minus_clipped(v[0], v[1], v[2], v[3], v[4]); // 96 FLOP (6 DIV)
            const Real vp = _weno_pluss_clipped(v[1], v[2], v[3], v[4], v[5]); // 96 FLOP (6 DIV)
            assert(!isnan(vp)); assert(!isnan(vm));

            const Real wm = _weno_minus_clipped(w[0], w[1], w[2], w[3], w[4]); // 96 FLOP (6 DIV)
            const Real wp = _weno_pluss_clipped(w[1], w[2], w[3], w[4], w[5]); // 96 FLOP (6 DIV)
            assert(!isnan(wp)); assert(!isnan(wm));

            // 3.)
            Real sm, sp;
            _char_vel_einfeldt(rm, rp, um, up, pm, pp, Gm, Gp, Pm, Pp, sm, sp); // 29 FLOP (6 DIV)
            const Real ss = _char_vel_star(rm, rp, um, up, pm, pp, sm, sp); // 11 FLOP (1 DIV)
            assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

            // 4.)
            const Real fr = _hllc_rho(rm, rp, um, up, sm, sp, ss); // 23 FLOP (2 DIV)
            const Real fu = _hllc_pvel(rm, rp, um, up, pm, pp, sm, sp, ss); // 29 FLOP (2 DIV)
            const Real fv = _hllc_vel(rm, rp, vm, vp, um, up, sm, sp, ss); // 25 FLOP (2 DIV)
            const Real fw = _hllc_vel(rm, rp, wm, wp, um, up, sm, sp, ss); // 25 FLOP (2 DIV)
            const Real fe = _hllc_e(rm, rp, um, up, vm, vp, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss); // 59 FLOP (4 DIV)
            const Real fG = _hllc_rho(Gm, Gp, um, up, sm, sp, ss); // 23 FLOP (2 DIV)
            const Real fP = _hllc_rho(Pm, Pp, um, up, sm, sp, ss); // 23 FLOP (2 DIV)
            assert(!isnan(fr)); assert(!isnan(fu)); assert(!isnan(fv)); assert(!isnan(fw)); assert(!isnan(fe)); assert(!isnan(fG)); assert(!isnan(fP));

            const Real hllc_vel = _extraterm_hllc_vel(um, up, Gm, Gp, Pm, Pp, sm, sp, ss); // 19 FLOP (2 DIV)

            /* if (global_iz) */
            /* { */
/* #pragma unroll 6 */
            /*     for (uint_t i = 0; i < 6; ++i) */
            /*     { */
            /*         r[0] += r[i]; */
            /*         u[0] += u[i]; */
            /*         v[0] += v[i]; */
            /*         w[0] += w[i]; */
            /*         e[0] += e[i]; */
            /*         G[0] += G[i]; */
            /*         P[0] += P[i]; */
            /*     } */
            /* } */
            /* const uint_t idx = ID3(iy, ix, iz-3, NY, NXP1); */
            /* flux.r[idx] = r[0]; */
            /* flux.u[idx] = u[0]; */
            /* flux.v[idx] = v[0]; */
            /* flux.w[idx] = w[0]; */
            /* flux.e[idx] = e[0]; */
            /* flux.G[idx] = G[0]; */
            /* flux.P[idx] = P[0]; */
            /* xtra_vel[idx] = r[0]; */
            /* xtra_Gm[idx]  = w[0]; */
            /* xtra_Gp[idx]  = e[0]; */
            /* xtra_Pm[idx]  = P[0]; */
            /* xtra_Pp[idx]  = u[0]; */

            const uint_t idx = ID3(iy, ix, iz-3, NY, NXP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}



__global__
void _yflux(const uint_t nslices, const uint_t global_iz,
        const DevicePointer ghostL, const DevicePointer ghostR, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NYP1 = NY + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    assert(NYP1 > 5);

    if (ix < NX && iy < NYP1)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            // GMEM transactions are cached, effective GMEM accesses are 7*3
            // (according to nvvp)
            if (0 == iy)
                _load_3Y<0,0,3,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(3*GMEM + 3*TEX)
            else if (1 == iy)
                _load_2Y<0,0,2,1>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(2*GMEM + 4*TEX)
            else if (2 == iy)
                _load_1Y<0,0,1,2>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(1*GMEM + 5*TEX)
            else if (NYP1-3 == iy)
                _load_1Y<NYP1-6,5,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NYP1-2 == iy)
                _load_2Y<NYP1-5,4,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NYP1-1 == iy)
                _load_3Y<NYP1-4,3,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else
                _load_internal_Y(ix, iy, iz, r, u, v, w, e, G, P, global_iz, NULL); // load 7*(6*TEX)

            // 2.)
            // convert to primitive variables
#pragma unroll 6
            for (uint_t i = 0; i < 6; ++i)
            {
                e[i] = (e[i] - 0.5f*(u[i]*u[i] + v[i]*v[i] + w[i]*w[i])/r[i] - P[i]) / G[i];
                u[i] = u[i]/r[i];
                v[i] = v[i]/r[i];
                w[i] = w[i]/r[i];
            } // 6 x (8 MUL/ADD/SUB + 5 DIV) = 78 FLOPS

            const Real rm = _weno_minus_clipped(r[0], r[1], r[2], r[3], r[4]); // 96 FLOP (6 DIV)
            const Real rp = _weno_pluss_clipped(r[1], r[2], r[3], r[4], r[5]); // 96 FLOP (6 DIV)
            assert(!isnan(rp)); assert(!isnan(rm));

            const Real Gm = _weno_minus_clipped(G[0], G[1], G[2], G[3], G[4]); // 96 FLOP (6 DIV)
            const Real Gp = _weno_pluss_clipped(G[1], G[2], G[3], G[4], G[5]); // 96 FLOP (6 DIV)
            assert(!isnan(Gp)); assert(!isnan(Gm));

            const Real Pm = _weno_minus_clipped(P[0], P[1], P[2], P[3], P[4]); // 96 FLOP (6 DIV)
            const Real Pp = _weno_pluss_clipped(P[1], P[2], P[3], P[4], P[5]); // 96 FLOP (6 DIV)
            assert(!isnan(Pp)); assert(!isnan(Pm));

            const Real pm = _weno_minus_clipped(e[0], e[1], e[2], e[3], e[4]); // 96 FLOP (6 DIV)
            const Real pp = _weno_pluss_clipped(e[1], e[2], e[3], e[4], e[5]); // 96 FLOP (6 DIV)
            assert(!isnan(pp)); assert(!isnan(pm));

            const Real vm = _weno_minus_clipped(v[0], v[1], v[2], v[3], v[4]); // 96 FLOP (6 DIV)
            const Real vp = _weno_pluss_clipped(v[1], v[2], v[3], v[4], v[5]); // 96 FLOP (6 DIV)
            assert(!isnan(vp)); assert(!isnan(vm));

            const Real um = _weno_minus_clipped(u[0], u[1], u[2], u[3], u[4]); // 96 FLOP (6 DIV)
            const Real up = _weno_pluss_clipped(u[1], u[2], u[3], u[4], u[5]); // 96 FLOP (6 DIV)
            assert(!isnan(up)); assert(!isnan(um));

            const Real wm = _weno_minus_clipped(w[0], w[1], w[2], w[3], w[4]); // 96 FLOP (6 DIV)
            const Real wp = _weno_pluss_clipped(w[1], w[2], w[3], w[4], w[5]); // 96 FLOP (6 DIV)
            assert(!isnan(wp)); assert(!isnan(wm));

            // 3.)
            Real sm, sp;
            _char_vel_einfeldt(rm, rp, vm, vp, pm, pp, Gm, Gp, Pm, Pp, sm, sp); // 29 FLOP (6 DIV)
            const Real ss = _char_vel_star(rm, rp, vm, vp, pm, pp, sm, sp); // 11 FLOP (1 DIV)
            assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

            // 4.)
            const Real fr = _hllc_rho(rm, rp, vm, vp, sm, sp, ss); // 23 FLOP (2 DIV)
            const Real fu = _hllc_vel(rm, rp, um, up, vm, vp, sm, sp, ss); // 25 FLOP (2 DIV)
            const Real fv = _hllc_pvel(rm, rp, vm, vp, pm, pp, sm, sp, ss); // 29 FLOP (2 DIV)
            const Real fw = _hllc_vel(rm, rp, wm, wp, vm, vp, sm, sp, ss); // 25 FLOP (2 DIV)
            const Real fe = _hllc_e(rm, rp, vm, vp, um, up, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss); // 59 FLOP (4 DIV)
            const Real fG = _hllc_rho(Gm, Gp, vm, vp, sm, sp, ss); // 23 FLOP (2 DIV)
            const Real fP = _hllc_rho(Pm, Pp, vm, vp, sm, sp, ss); // 23 FLOP (2 DIV)
            assert(!isnan(fr)); assert(!isnan(fu)); assert(!isnan(fv)); assert(!isnan(fw)); assert(!isnan(fe)); assert(!isnan(fG)); assert(!isnan(fP));

            const Real hllc_vel = _extraterm_hllc_vel(vm, vp, Gm, Gp, Pm, Pp, sm, sp, ss); // 19 FLOP (2 DIV)

            /* if (global_iz) */
            /* { */
/* #pragma unroll 6 */
            /*     for (uint_t i = 0; i < 6; ++i) */
            /*     { */
            /*         r[0] += r[i]; */
            /*         u[0] += u[i]; */
            /*         v[0] += v[i]; */
            /*         w[0] += w[i]; */
            /*         e[0] += e[i]; */
            /*         G[0] += G[i]; */
            /*         P[0] += P[i]; */
            /*     } */
            /* } */
            /* const uint_t idx = ID3(ix, iy, iz-3, NX, NYP1); */
            /* flux.r[idx] = r[0]; */
            /* flux.u[idx] = u[0]; */
            /* flux.v[idx] = v[0]; */
            /* flux.w[idx] = w[0]; */
            /* flux.e[idx] = e[0]; */
            /* flux.G[idx] = G[0]; */
            /* flux.P[idx] = P[0]; */
            /* xtra_vel[idx] = r[0]; */
            /* xtra_Gm[idx]  = w[0]; */
            /* xtra_Gp[idx]  = e[0]; */
            /* xtra_Pm[idx]  = P[0]; */
            /* xtra_Pp[idx]  = u[0]; */

            const uint_t idx = ID3(ix, iy, iz-3, NX, NYP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}



__global__
void _zflux(const uint_t nslices, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp,
        const uint_t global_iz = 0)
{
    /* *
     * Notes:
     * ======
     * 1.) NX = NodeBlock::sizeX
     * 2.) NY = NodeBlock::sizeY
     * 3.) NZ = NodeBlock::sizeZ
     * 4.) nslices = number of slices for currently processed chunk
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    // depends on boundary condition in z-direction
    assert(NodeBlock::sizeZ > 0);

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 3; iz < (nslices+1)+3; ++iz) // first and last 3 slices are zghosts; need to compute nslices+1 fluxes in z-direction
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            _load_internal_Z(ix, iy, iz, r, u, v, w, e, G, P, 0, NULL); // load 7*(6*TEX)

            // 2.)
            // convert to primitive variables
#pragma unroll 6
            for (uint_t i = 0; i < 6; ++i)
            {
                e[i] = (e[i] - 0.5f*(u[i]*u[i] + v[i]*v[i] + w[i]*w[i])/r[i] - P[i]) / G[i];
                u[i] = u[i]/r[i];
                v[i] = v[i]/r[i];
                w[i] = w[i]/r[i];
            } // 6 x (8 MUL/ADD/SUB + 5 DIV) = 78 FLOPS

            const Real rm = _weno_minus_clipped(r[0], r[1], r[2], r[3], r[4]); // 96 FLOP (6 DIV)
            const Real rp = _weno_pluss_clipped(r[1], r[2], r[3], r[4], r[5]); // 96 FLOP (6 DIV)
            assert(!isnan(rp)); assert(!isnan(rm));

            const Real Gm = _weno_minus_clipped(G[0], G[1], G[2], G[3], G[4]); // 96 FLOP (6 DIV)
            const Real Gp = _weno_pluss_clipped(G[1], G[2], G[3], G[4], G[5]); // 96 FLOP (6 DIV)
            assert(!isnan(Gp)); assert(!isnan(Gm));

            const Real Pm = _weno_minus_clipped(P[0], P[1], P[2], P[3], P[4]); // 96 FLOP (6 DIV)
            const Real Pp = _weno_pluss_clipped(P[1], P[2], P[3], P[4], P[5]); // 96 FLOP (6 DIV)
            assert(!isnan(Pp)); assert(!isnan(Pm));

            const Real pm = _weno_minus_clipped(e[0], e[1], e[2], e[3], e[4]); // 96 FLOP (6 DIV)
            const Real pp = _weno_pluss_clipped(e[1], e[2], e[3], e[4], e[5]); // 96 FLOP (6 DIV)
            assert(!isnan(pp)); assert(!isnan(pm));

            const Real wm = _weno_minus_clipped(w[0], w[1], w[2], w[3], w[4]); // 96 FLOP (6 DIV)
            const Real wp = _weno_pluss_clipped(w[1], w[2], w[3], w[4], w[5]); // 96 FLOP (6 DIV)
            assert(!isnan(wp)); assert(!isnan(wm));

            const Real um = _weno_minus_clipped(u[0], u[1], u[2], u[3], u[4]); // 96 FLOP (6 DIV)
            const Real up = _weno_pluss_clipped(u[1], u[2], u[3], u[4], u[5]); // 96 FLOP (6 DIV)
            assert(!isnan(up)); assert(!isnan(um));

            const Real vm = _weno_minus_clipped(v[0], v[1], v[2], v[3], v[4]); // 96 FLOP (6 DIV)
            const Real vp = _weno_pluss_clipped(v[1], v[2], v[3], v[4], v[5]); // 96 FLOP (6 DIV)
            assert(!isnan(vp)); assert(!isnan(vm));

            // 3.)
            Real sm, sp;
            _char_vel_einfeldt(rm, rp, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp); // 29 FLOP (6 DIV)
            const Real ss = _char_vel_star(rm, rp, wm, wp, pm, pp, sm, sp); // 11 FLOP (1 DIV)
            assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

            // 4.)
            const Real fr = _hllc_rho(rm, rp, wm, wp, sm, sp, ss); // 23 FLOP (2 DIV)
            const Real fu = _hllc_vel(rm, rp, um, up, wm, wp, sm, sp, ss); // 25 FLOP (2 DIV)
            const Real fv = _hllc_vel(rm, rp, vm, vp, wm, wp, sm, sp, ss); // 25 FLOP (2 DIV)
            const Real fw = _hllc_pvel(rm, rp, wm, wp, pm, pp, sm, sp, ss); // 29 FLOP (2 DIV)
            const Real fe = _hllc_e(rm, rp, wm, wp, um, up, vm, vp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss); // 59 FLOP (4 DIV)
            const Real fG = _hllc_rho(Gm, Gp, wm, wp, sm, sp, ss); // 23 FLOP (2 DIV)
            const Real fP = _hllc_rho(Pm, Pp, wm, wp, sm, sp, ss); // 23 FLOP (2 DIV)

            const Real hllc_vel = _extraterm_hllc_vel(wm, wp, Gm, Gp, Pm, Pp, sm, sp, ss); // 19 FLOP (2 DIV)

            /* if (global_iz) */
            /* { */
/* #pragma unroll 6 */
            /*     for (uint_t i = 0; i < 6; ++i) */
            /*     { */
            /*         r[0] += r[i]; */
            /*         u[0] += u[i]; */
            /*         v[0] += v[i]; */
            /*         w[0] += w[i]; */
            /*         e[0] += e[i]; */
            /*         G[0] += G[i]; */
            /*         P[0] += P[i]; */
            /*     } */
            /* } */
            /* const uint_t idx = ID3(ix, iy, iz-3, NX, NY); */
            /* flux.r[idx] = r[0]; */
            /* flux.u[idx] = u[0]; */
            /* flux.v[idx] = v[0]; */
            /* flux.w[idx] = w[0]; */
            /* flux.e[idx] = e[0]; */
            /* flux.G[idx] = G[0]; */
            /* flux.P[idx] = P[0]; */
            /* xtra_vel[idx] = r[0]; */
            /* xtra_Gm[idx]  = w[0]; */
            /* xtra_Gp[idx]  = e[0]; */
            /* xtra_Pm[idx]  = P[0]; */
            /* xtra_Pp[idx]  = u[0]; */


            const uint_t idx = ID3(ix, iy, iz-3, NX, NY);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _divergence(const uint_t nslices,
        const DevicePointer xflux, const DevicePointer yflux, const DevicePointer zflux,
        DevicePointer rhs, const Real a, const Real dtinvh, const DevicePointer tmp,
        const Real * const sumG, const Real * const sumP, const Real * const divU)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        Real fxp, fxm, fyp, fym, fzp, fzm;
        const Real factor6 = 1.0f / 6.0f;

        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx = ID3(ix, iy, iz, NX, NY);

            _fetch_flux(ix, iy, iz, xflux.r, yflux.r, zflux.r, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_r = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm);
            rhs.r[idx] = a*tmp.r[idx] - rhs_r;

            _fetch_flux(ix, iy, iz, xflux.u, yflux.u, zflux.u, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_u = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm);
            rhs.u[idx] = a*tmp.u[idx] - rhs_u;

            _fetch_flux(ix, iy, iz, xflux.v, yflux.v, zflux.v, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_v = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm);
            rhs.v[idx] = a*tmp.v[idx] - rhs_v;

            _fetch_flux(ix, iy, iz, xflux.w, yflux.w, zflux.w, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_w = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm);
            rhs.w[idx] = a*tmp.w[idx] - rhs_w;

            _fetch_flux(ix, iy, iz, xflux.e, yflux.e, zflux.e, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_e = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm);
            rhs.e[idx] = a*tmp.e[idx] - rhs_e;

            _fetch_flux(ix, iy, iz, xflux.G, yflux.G, zflux.G, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_G = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm   - divU[idx] * sumG[idx] * factor6);
            rhs.G[idx] = a*tmp.G[idx] - rhs_G;

            _fetch_flux(ix, iy, iz, xflux.P, yflux.P, zflux.P, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_P = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm   - divU[idx] * sumP[idx] * factor6);
            rhs.P[idx] = a*tmp.P[idx] - rhs_P;
        }
    }
}


__global__
void _update(const uint_t nslices, const Real b, DevicePointer tmp, const DevicePointer rhs)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx = ID3(ix, iy, iz, NX, NY);

            const Real r = tex3D(texR, ix, iy, iz+3);
            const Real u = tex3D(texU, ix, iy, iz+3);
            const Real v = tex3D(texV, ix, iy, iz+3);
            const Real w = tex3D(texW, ix, iy, iz+3);
            const Real e = tex3D(texE, ix, iy, iz+3);
            const Real G = tex3D(texG, ix, iy, iz+3);
            const Real P = tex3D(texP, ix, iy, iz+3);

            // this overwrites the rhs from the previous stage, stored in tmp,
            // with the updated solution.
            tmp.r[idx] = b*rhs.r[idx] + r;
            tmp.u[idx] = b*rhs.u[idx] + u;
            tmp.v[idx] = b*rhs.v[idx] + v;
            tmp.w[idx] = b*rhs.w[idx] + w;
            tmp.e[idx] = b*rhs.e[idx] + e;
            tmp.G[idx] = b*rhs.G[idx] + G;
            tmp.P[idx] = b*rhs.P[idx] + P;
            assert(tmp.r[idx] > 0);
            assert(tmp.e[idx] > 0);
            assert(tmp.G[idx] > 0);
            assert(tmp.P[idx] >= 0);
            /* if (tmp.P[idx] < 0) */
            /*     printf("(%d, %d, %d):\trhs.P = %f, tmp.P = %f, P = %f\n", ix, iy, iz, rhs.P[idx], tmp.P[idx], P); */
        }
    }
}


__global__
void _maxSOS(const uint_t nslices, int* g_maxSOS)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    const uint_t loc_idx = blockDim.x * threadIdx.y + threadIdx.x;
    __shared__ Real block_sos[_NTHREADS_];
    block_sos[loc_idx] = 0.0f;

    if (ix < NX && iy < NY)
    {
        Real sos = 0.0f;

        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const Real r = tex3D(texR, ix, iy, iz);
            const Real u = tex3D(texU, ix, iy, iz);
            const Real v = tex3D(texV, ix, iy, iz);
            const Real w = tex3D(texW, ix, iy, iz);
            const Real e = tex3D(texE, ix, iy, iz);
            const Real G = tex3D(texG, ix, iy, iz);
            const Real P = tex3D(texP, ix, iy, iz);

            const Real p = (e - (u*u + v*v + w*w)*(0.5f/r) - P) / G;
            const Real c = sqrtf(((p + P) / G + p) / r);

            sos = fmaxf(sos, c + fmaxf(fmaxf(fabsf(u), fabsf(v)), fabsf(w)) / r);
        }
        block_sos[loc_idx] = sos;
        __syncthreads();

        if (0 == loc_idx)
        {
            for (int i = 1; i < _NTHREADS_; ++i)
                sos = fmaxf(sos, block_sos[i]);
            assert(sos > 0);
            atomicMax(g_maxSOS, __float_as_int(sos));
        }
    }
}

///////////////////////////////////////////////////////////////////////////////
//                              KERNEL WRAPPERS                              //
///////////////////////////////////////////////////////////////////////////////
void GPU::xflux(const uint_t nslices, const uint_t global_iz)
{
#ifndef _MUTE_GPU_
    DevicePointer xghostL(d_xgl);
    DevicePointer xghostR(d_xgr);
    DevicePointer xflux(d_xflux);

    {
        const dim3 blocks(1, _NTHREADS_, 1);
        const dim3 grid(NXP1, (NY + _NTHREADS_ -1)/_NTHREADS_, 1);
        GPU::profiler.push_startCUDA("_XFLUX", &stream1);
        _xflux<<<grid, blocks, 0, stream1>>>(nslices, global_iz, xghostL, xghostR, xflux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
        GPU::profiler.pop_stopCUDA();
    }

    {
        const dim3 xtraBlocks(_TILE_DIM_, _BLOCK_ROWS_, 1);
        const dim3 xtraGrid((NX + _TILE_DIM_ - 1)/_TILE_DIM_, (NY + _TILE_DIM_ - 1)/_TILE_DIM_, 1);
        GPU::profiler.push_startCUDA("_XEXTRATERM", &stream1);
        _xextraterm_hllc<<<xtraGrid, xtraBlocks, 0, stream1>>>(nslices, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
        GPU::profiler.pop_stopCUDA();

        // 70.1% of Peak BW (w/ ECC) on K20c
        const uint_t PTS_PER_SLICE = NX * NY;
        const uint_t total_words = PTS_PER_SLICE * (nslices * 9);
        /* printf("XEXTRA GB = %f\n", total_words*4./1024./1024./1024.); */
    }
#endif
}


void GPU::yflux(const uint_t nslices, const uint_t global_iz)
{
#ifndef _MUTE_GPU_
    DevicePointer yghostL(d_ygl);
    DevicePointer yghostR(d_ygr);
    DevicePointer yflux(d_yflux);

    const dim3 blocks(_NTHREADS_, 1, 1);

    {
        const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NYP1, 1);
        GPU::profiler.push_startCUDA("_YFLUX", &stream1);
        _yflux<<<grid, blocks, 0, stream1>>>(nslices, global_iz, yghostL, yghostR, yflux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
        GPU::profiler.pop_stopCUDA();
    }

    {
        const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
        GPU::profiler.push_startCUDA("_YEXTRATERM", &stream1);
        _yextraterm_hllc<<<grid, blocks, 0, stream1>>>(nslices, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
        GPU::profiler.pop_stopCUDA();

        // 78.6% of Peak BW (w/ ECC) on K20c
        const uint_t PTS_PER_SLICE = NX * NY;
        const uint_t total_words = PTS_PER_SLICE * (nslices * 12);
        /* printf("YEXTRA GB = %f\n", total_words*4./1024./1024./1024.); */
    }
#endif
}


void GPU::zflux(const uint_t nslices)
{
#ifndef _MUTE_GPU_
    DevicePointer zflux(d_zflux);

    const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
    const dim3 blocks(_NTHREADS_, 1, 1);

    GPU::profiler.push_startCUDA("_ZFLUX", &stream1);
    _zflux<<<grid, blocks, 0, stream1>>>(nslices, zflux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
    GPU::profiler.pop_stopCUDA();

    GPU::profiler.push_startCUDA("_ZEXTRATERM", &stream1);
    _zextraterm_hllc<<<grid, blocks, 0, stream1>>>(nslices, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
    GPU::profiler.pop_stopCUDA();

    // 76.7% of Peak BW (w/ ECC) on K20c
    const uint_t PTS_PER_SLICE = NX * NY;
    const uint_t total_words = PTS_PER_SLICE * (nslices * 12);
    /* printf("ZEXTRA GB = %f\n", total_words*4./1024./1024./1024.); */
#endif
}


void GPU::divergence(const Real a, const Real dtinvh, const uint_t nslices)
{
#ifndef _MUTE_GPU_
    hipStreamWaitEvent(stream1, h2d_tmp_completed, 0);

    DevicePointer xflux(d_xflux);
    DevicePointer yflux(d_yflux);
    DevicePointer zflux(d_zflux);
    DevicePointer rhs(d_rhs);
    DevicePointer tmp(d_tmp);

    const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
    const dim3 blocks(_NTHREADS_, 1, 1);

    GPU::profiler.push_startCUDA("_DIVERGENCE", &stream1);
    _divergence<<<grid, blocks, 0, stream1>>>(nslices, xflux, yflux, zflux, rhs, a, dtinvh, tmp, d_sumG, d_sumP, d_divU);
    GPU::profiler.pop_stopCUDA();

    hipEventRecord(divergence_completed, stream1);
#endif
}


void GPU::update(const Real b, const uint_t nslices)
{
#ifndef _MUTE_GPU_
    DevicePointer tmp(d_tmp);
    DevicePointer rhs(d_rhs);

    const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
    const dim3 blocks(_NTHREADS_, 1, 1);

    GPU::profiler.push_startCUDA("_UPDATE", &stream1);
    _update<<<grid, blocks, 0, stream1>>>(nslices, b, tmp, rhs);
    GPU::profiler.pop_stopCUDA();

    hipEventRecord(update_completed, stream1);
#endif
}


void GPU::MaxSpeedOfSound(const uint_t nslices)
{
#ifndef _MUTE_GPU_
    const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
    const dim3 blocks(_NTHREADS_, 1, 1);

    GPU::profiler.push_startCUDA("_MAXSOS", &stream1);
    _maxSOS<<<grid, blocks, 0, stream1>>>(nslices, d_maxSOS);
    GPU::profiler.pop_stopCUDA();
#endif
}

///////////////////////////////////////////////////////////////////////////
// TEST SECTION
///////////////////////////////////////////////////////////////////////////
void GPU::TestKernel()
{
    DevicePointer xghostL(d_xgl);
    DevicePointer xghostR(d_xgr);
    DevicePointer xflux(d_xflux);

    DevicePointer yghostL(d_ygl);
    DevicePointer yghostR(d_ygr);
    DevicePointer yflux(d_yflux);

    DevicePointer zflux(d_zflux);


    // rearrange GPU memory for TEST
    for (int var = 0; var < 7; ++var)
    {
        hipFree(d_tmp[var]);
        hipFree(d_rhs[var]);
    }
    hipFree(d_Gm);
    hipFree(d_Gp);
    hipFree(d_Pm);
    hipFree(d_Pp);
    hipFree(d_hllc_vel);
    hipFree(d_sumG);
    hipFree(d_sumP);
    hipFree(d_divU);

    const uint_t nslices = NodeBlock::sizeZ;
    const uint_t xflxSize = (NodeBlock::sizeX+1)*NodeBlock::sizeY*nslices;
    const uint_t yflxSize = NodeBlock::sizeX*(NodeBlock::sizeY+1)*nslices;
    const uint_t zflxSize = NodeBlock::sizeX*NodeBlock::sizeY*(nslices+1);

    Real *d_extra_X[5];
    Real *d_extra_Y[5];
    Real *d_extra_Z[5];
    for (int i = 0; i < 5; ++i)
    {
        hipMalloc(&(d_extra_X[i]), xflxSize * sizeof(Real));
        hipMalloc(&(d_extra_Y[i]), yflxSize * sizeof(Real));
        hipMalloc(&(d_extra_Z[i]), zflxSize * sizeof(Real));
    }
    GPU::tell_memUsage_GPU();


    {

        const dim3 xblocks(1, _NTHREADS_, 1);
        const dim3 yblocks(_NTHREADS_, 1, 1);
        const dim3 zblocks(_NTHREADS_, 1, 1);
        const dim3 xgrid(NXP1, (NY + _NTHREADS_ - 1) / _NTHREADS_,   1);
        const dim3 ygrid((NX   + _NTHREADS_ - 1) / _NTHREADS_, NYP1, 1);
        const dim3 zgrid((NX   + _NTHREADS_ - 1) / _NTHREADS_, NY,   1);

        hipStream_t *_s = (hipStream_t *) malloc(3*sizeof(hipStream_t));
        for (int i = 0; i < 3; ++i)
            hipStreamCreate(&(_s[i]));

        GPU::profiler.push_startCUDA("_XFLUX", &_s[0]);
        _xflux<<<xgrid, xblocks, 0, _s[0]>>>(nslices, 0, xghostL, xghostR, xflux, d_extra_X[0], d_extra_X[1], d_extra_X[2], d_extra_X[3], d_extra_X[4]);
        GPU::profiler.pop_stopCUDA();

        GPU::profiler.push_startCUDA("_YFLUX", &_s[0]);
        _yflux<<<ygrid, yblocks, 0, _s[0]>>>(nslices, 0, yghostL, yghostR, yflux, d_extra_Y[0], d_extra_Y[1], d_extra_Y[2], d_extra_Y[3], d_extra_Y[4]);
        GPU::profiler.pop_stopCUDA();

        GPU::profiler.push_startCUDA("_ZFLUX", &_s[0]);
        _zflux<<<zgrid, zblocks, 0, _s[0]>>>(nslices, zflux, d_extra_Z[0], d_extra_Z[1], d_extra_Z[2], d_extra_Z[3], d_extra_Z[4]);
        GPU::profiler.pop_stopCUDA();

        /* _xflux<<<xgrid, xblocks, 0, _s[0]>>>(nslices, 0, xghostL, xghostR, xflux, d_extra_X[0], d_extra_X[1], d_extra_X[2], d_extra_X[3], d_extra_X[4]); */
        /* _yflux<<<ygrid, yblocks, 0, _s[1]>>>(nslices, 0, yghostL, yghostR, yflux, d_extra_Y[0], d_extra_Y[1], d_extra_Y[2], d_extra_Y[3], d_extra_Y[4]); */
        /* _zflux<<<zgrid, zblocks, 0, _s[2]>>>(nslices, zflux, d_extra_Z[0], d_extra_Z[1], d_extra_Z[2], d_extra_Z[3], d_extra_Z[4]); */

        hipDeviceSynchronize();

        for (int i = 0; i < 3; ++i)
            hipStreamDestroy(_s[i]);
    }
}


///////////////////////////////////////////////////////////////////////////////
//                                   UTILS                                   //
///////////////////////////////////////////////////////////////////////////////
static void _bindTexture(texture<float, 3, hipReadModeElementType> * const tex, hipArray_t d_ptr)
{
    hipChannelFormatDesc fmt = hipCreateChannelDesc<Real>();
    tex->addressMode[0]       = hipAddressModeClamp;
    tex->addressMode[1]       = hipAddressModeClamp;
    tex->addressMode[2]       = hipAddressModeClamp;
    tex->channelDesc          = fmt;
    tex->filterMode           = hipFilterModePoint;
    tex->mipmapFilterMode     = hipFilterModePoint;
    tex->normalized           = false;

    hipBindTextureToArray(tex, d_ptr, &fmt);
}


void GPU::bind_textures()
{
#ifndef _MUTE_GPU_
    _bindTexture(&texR, d_GPUin[0]);
    _bindTexture(&texU, d_GPUin[1]);
    _bindTexture(&texV, d_GPUin[2]);
    _bindTexture(&texW, d_GPUin[3]);
    _bindTexture(&texE, d_GPUin[4]);
    _bindTexture(&texG, d_GPUin[5]);
    _bindTexture(&texP, d_GPUin[6]);
#endif
}


void GPU::unbind_textures()
{
#ifndef _MUTE_GPU_
    hipUnbindTexture(&texR);
    hipUnbindTexture(&texU);
    hipUnbindTexture(&texV);
    hipUnbindTexture(&texW);
    hipUnbindTexture(&texE);
    hipUnbindTexture(&texG);
    hipUnbindTexture(&texP);
#endif
}
