#include "hip/hip_runtime.h"
/* *
 * GPUkernels.cu
 *
 * Created by Fabian Wermelinger on 6/25/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */
#include <assert.h>
#include <stdio.h>

#include "GPU.cuh"

#if _BLOCKSIZEX_ < 5
#error Minimum _BLOCKSIZEX_ is 5
#elif _BLOCKSIZEY_ < 5
#error Minimum _BLOCKSIZEY_ is 5
#elif _BLOCKSIZEZ_ < 1
#error Minimum _BLOCKSIZEZ_ is 1
#endif

#if NX % _TILE_DIM_ != 0
#error _BLOCKSIZEX_ should be an integer multiple of _TILE_DIM_
#endif
#if NY % _TILE_DIM_ != 0
#error _BLOCKSIZEY_ should be an integer multiple of _TILE_DIM_
#endif


///////////////////////////////////////////////////////////////////////////////
//                           GLOBAL VARIABLES                                //
///////////////////////////////////////////////////////////////////////////////
// helper storage
extern real_vector_t d_flux;
extern Real *d_Gm, *d_Gp;
extern Real *d_Pm, *d_Pp;
extern Real *d_hllc_vel;
extern Real *d_sumG, *d_sumP, *d_divU;

// max SOS
extern int *d_maxSOS;

// GPU input/output
extern struct GPU_COMM gpu_comm[_NUM_GPU_BUF_];

// use non-null stream (async)
extern hipStream_t *stream;

// compute events
extern hipEvent_t *event_compute;

// texture references
#include "Texture.cu"

///////////////////////////////////////////////////////////////////////////////
//                             DEVICE FUNCTIONS                              //
///////////////////////////////////////////////////////////////////////////////
__device__
inline Real _weno_pluss(const Real b, const Real c, const Real d, const Real e, const Real f)
{
    const Real wenoeps_f = (Real)WENOEPS;
#ifndef _WENO3_
    // (90 MUL/ADD/SUB + 6 DIV) = 96 FLOP
    const Real inv6 = 1.0f/6.0f;
    const Real inv3 = 1.0f/3.0f;
    const Real q1 =  10.0f*inv3;
    const Real q2 =  31.0f*inv3;
    const Real q3 =  11.0f*inv3;
    const Real q4 =  25.0f*inv3;
    const Real q5 =  19.0f*inv3;
    const Real q6 =   4.0f*inv3;
    const Real q7 =  13.0f*inv3;
    const Real q8 =   5.0f*inv3;

    const Real sum0 =  inv3*f - 7.0f*inv6*e + 11.0f*inv6*d;
    const Real sum1 = -inv6*e + 5.0f*inv6*d + inv3*c;
    const Real sum2 =  inv3*d + 5.0f*inv6*c - inv6*b;

    const Real is0 = d*(d*q1 - e*q2 + f*q3) + e*(e*q4 - f*q5) + f*f*q6;
    const Real is1 = c*(c*q6 - d*q7 + e*q8) + d*(d*q7 - e*q7) + e*e*q6;
    const Real is2 = b*(b*q6 - c*q5 + d*q3) + c*(c*q4 - d*q2) + d*d*q1;

    const Real is0plus = is0 + wenoeps_f;
    const Real is1plus = is1 + wenoeps_f;
    const Real is2plus = is2 + wenoeps_f;

    const Real alpha0 = 1.0f / (10.0f*is0plus*is0plus);
    const Real alpha1 = 6.0f * (1.0f / (10.0f*is1plus*is1plus));
    const Real alpha2 = 3.0f * (1.0f / (10.0f*is2plus*is2plus));
    const Real alphasumInv = 1.0f / (alpha0+alpha1+alpha2);

    const Real omega0 = alpha0 * alphasumInv;
    const Real omega1 = alpha1 * alphasumInv;
    const Real omega2 = 1.0f - omega0 - omega1;

    return omega0*sum0 + omega1*sum1 + omega2*sum2;

#else
    // 28 FLOP
    const Real sum0 = 1.5f*d - 0.5f*e;
    const Real sum1 = 0.5f*(d + c);

    const Real is0 = (d-e)*(d-e);
    const Real is1 = (d-c)*(d-c);

    const Real alpha0 = 1.0f / (3.0f * (is0+wenoeps_f)*(is0+wenoeps_f));
    const Real alpha1 = 2.0f * (1.0f / (3.0f * (is1+wenoeps_f)*(is1+wenoeps_f)));

    const Real omega0 = alpha0 / (alpha0+alpha1);
    const Real omega1 = 1.0f - omega0;

    return omega0*sum0 + omega1*sum1;

#endif
}


__device__
inline Real _weno_minus(const Real a, const Real b, const Real c, const Real d, const Real e)
{
    const Real wenoeps_f = (Real)WENOEPS;
#ifndef _WENO3_
    // (90 MUL/ADD/SUB + 6 DIV) = 96 FLOP
    const Real inv6 = 1.0f/6.0f;
    const Real inv3 = 1.0f/3.0f;
    const Real q1 =   4.0f*inv3;
    const Real q2 =  19.0f*inv3;
    const Real q3 =  11.0f*inv3;
    const Real q4 =  25.0f*inv3;
    const Real q5 =  31.0f*inv3;
    const Real q6 =  10.0f*inv3;
    const Real q7 =  13.0f*inv3;
    const Real q8 =   5.0f*inv3;

    const Real sum0 =  inv3*a - 7.0f*inv6*b + 11.0f*inv6*c;
    const Real sum1 = -inv6*b + 5.0f*inv6*c + inv3*d;
    const Real sum2 =  inv3*c + 5.0f*inv6*d - inv6*e;

    const Real is0 = a*(a*q1 - b*q2 + c*q3) + b*(b*q4 - c*q5) + c*c*q6;
    const Real is1 = b*(b*q1 - c*q7 + d*q8) + c*(c*q7 - d*q7) + d*d*q1;
    const Real is2 = c*(c*q6 - d*q5 + e*q3) + d*(d*q4 - e*q2) + e*e*q1;

    const Real is0plus = is0 + wenoeps_f;
    const Real is1plus = is1 + wenoeps_f;
    const Real is2plus = is2 + wenoeps_f;

    const Real alpha0 = 1.0f / (10.0f*is0plus*is0plus);
    const Real alpha1 = 6.0f * (1.0f / (10.0f*is1plus*is1plus));
    const Real alpha2 = 3.0f * (1.0f / (10.0f*is2plus*is2plus));
    const Real alphasumInv = 1.0f / (alpha0+alpha1+alpha2);

    const Real omega0 = alpha0 * alphasumInv;
    const Real omega1 = alpha1 * alphasumInv;
    const Real omega2 = 1.0f - omega0 - omega1;

    return omega0*sum0 + omega1*sum1 + omega2*sum2;

#else
    // 28 FLOP
    const Real sum0 = 1.5f*c - 0.5f*b;
    const Real sum1 = 0.5f*(c + d);

    const Real is0 = (c-b)*(c-b);
    const Real is1 = (d-c)*(d-c);

    const Real alpha0 = 1.0f / (3.0f * (is0+wenoeps_f)*(is0+wenoeps_f));
    const Real alpha1 = 2.0f * (1.0f / (3.0f * (is1+wenoeps_f)*(is1+wenoeps_f)));

    const Real omega0 = alpha0 / (alpha0+alpha1);
    const Real omega1 = 1.0f - omega0;

    return omega0*sum0 + omega1*sum1;

#endif
}


__device__
inline Real _weno_pluss_clipped(const Real b, const Real c, const Real d, const Real e, const Real f)
{
    const Real retval = _weno_pluss(b,c,d,e,f);
    const Real min_in = fminf( fminf(c,d), e );
    const Real max_in = fmaxf( fmaxf(c,d), e );
    return fminf(fmaxf(retval, min_in), max_in);
}


__device__
inline Real _weno_minus_clipped(const Real a, const Real b, const Real c, const Real d, const Real e)
{
    const Real retval = _weno_minus(a,b,c,d,e);
    const Real min_in = fminf( fminf(b,c), d );
    const Real max_in = fmaxf( fmaxf(b,c), d );
    return fminf(fmaxf(retval, min_in), max_in);
}


__device__
inline void _char_vel_einfeldt(const Real rm, const Real rp,
        const Real vm, const Real vp,
        const Real pm, const Real pp,
        const Real Gm, const Real Gp,
        const Real Pm, const Real Pp,
        Real& outm, Real& outp) // (23 MUL/ADD/SUB + 6 DIV) = 29 FLOP
{
    /* *
     * Compute upper and lower bounds of signal velocities for the Riemann
     * problem according to Einfeldt:
     *
     * 1.) Compute Rr needed for Roe averages
     * 2.) Compute speed of sound in left and right state
     * 3.) Compute speed of sound according to Einfeldt and Rr
     * 4.) Compute upper and lower signal velocities
     * */

    // 1.)
    assert(rm > 0.0f);
    assert(rp > 0.0f);
    const Real Rr   = sqrtf(rp / rm);
    const Real Rinv = 1.0f / (1.0f + Rr);

    // 2.)
    const Real cm2 = ((pm + Pm)/Gm + pm) / rm;
    const Real cp2 = ((pp + Pp)/Gp + pp) / rp;
    const Real cm  = sqrtf(cm2);
    const Real cp  = sqrtf(cp2);
    assert(!isnan(cm));
    assert(!isnan(cp));

    // 3.)
    const Real um    = vm;
    const Real up    = vp;
    const Real eta_2 = 0.5f*Rr*Rinv*Rinv;
    const Real d2    = (cm2 + Rr*cp2)*Rinv + eta_2*(up - um)*(up - um);
    const Real d     = sqrtf(d2);
    const Real u     = (um + Rr*up)*Rinv;
    assert(!isnan(d));
    assert(!isnan(u));

    // 4.)
    outm = fminf(u - d, um - cm);
    outp = fmaxf(u + d, up + cp);
}


/* *
 * Compute characteristic velocity, s^star, of the intermediate wave.  The
 * computation is based on the condition of uniform constant pressure in
 * the star region.  See P. Batten et. al., "On the choice of wavespeeds
 * for the HLLC Riemann solver", SIAM J. Sci. Comput. 18 (1997) 1553--1570
 * It is assumed s^minus and s^plus are known.
 * */
__device__
inline Real _char_vel_star(const Real rm, const Real rp,
        const Real vm, const Real vp,
        const Real pm, const Real pp,
        const Real sm, const Real sp) // (10 MUL/ADD/SUB + 1 DIV) = 11 FLOP
{
    const Real facm = rm * (sm - vm);
    const Real facp = rp * (sp - vp);
    return (pp - pm + vm*facm - vp*facp) / (facm - facp);
    /* return (pp + vm*facm - (pm + vp*facp)) / (facm - facp); */
}


__device__
inline Real _hllc_rho(const Real rm, const Real rp,
        const Real vm, const Real vp,
        const Real sm, const Real sp, const Real ss) // (21 MUL/ADD/SUB + 2 DIV) = 23 FLOP
{
    /* *
     * The flux computation is split into 4 parts:
     * 1.) Compute signum of s^*, compute s^- and s^+
     * 2.) Compute chi^* and delta of q^* and q
     * 3.) Compute trivial flux
     * 4.) Compute HLLC flux
     * */

    // 1.)
    const Real sign_star = (ss == 0.0f) ? 0.0f : ((ss < 0.0f) ? -1.0f : 1.0f);
    const Real s_minus = fminf(0.0f, sm);
    const Real s_pluss = fmaxf(0.0f, sp);

    // 2.)
    const Real chi_starm = (sm - vm) / (sm - ss);
    const Real chi_starp = (sp - vp) / (sp - ss);
    const Real qm        = rm;
    const Real qp        = rp;
    const Real q_deltam  = qm*chi_starm - qm;
    const Real q_deltap  = qp*chi_starp - qp;

    // 3.)
    const Real fm = qm*vm;
    const Real fp = qp*vp;

    // 4.)
    const Real flux = (0.5f*(1.0f + sign_star)) * (fm + s_minus*q_deltam) + (0.5f*(1.0f - sign_star)) * (fp + s_pluss*q_deltap);
    assert(!isnan(flux));
    return flux;
}


__device__
inline Real _hllc_vel(const Real rm,  const Real rp,
        const Real vm,  const Real vp,
        const Real vdm, const Real vdp,
        const Real sm,  const Real sp,  const Real ss) // (23 MUL/ADD/SUB + 2 DIV) = 25 FLOP
{
    /* *
     * The flux computation is split into 4 parts:
     * 1.) Compute signum of s^*, compute s^- and s^+
     * 2.) Compute chi^* and delta of q^* and q
     * 3.) Compute trivial flux
     * 4.) Compute HLLC flux
     * */

    // 1.)
    const Real sign_star = (ss == 0.0f) ? 0.0f : ((ss < 0.0f) ? -1.0f : 1.0f);
    const Real s_minus  = fminf(0.0f, sm);
    const Real s_pluss  = fmaxf(0.0f, sp);

    // 2.)
    const Real chi_starm = (sm - vdm) / (sm - ss);
    const Real chi_starp = (sp - vdp) / (sp - ss);
    const Real qm        = rm*vm;
    const Real qp        = rp*vp;
    const Real q_deltam  = qm*chi_starm - qm;
    const Real q_deltap  = qp*chi_starp - qp;

    // 3.)
    const Real fm = qm*vdm;
    const Real fp = qp*vdp;

    // 4.)
    const Real flux = (0.5f*(1.0f + sign_star)) * (fm + s_minus*q_deltam) + (0.5f*(1.0f - sign_star)) * (fp + s_pluss*q_deltap);
    assert(!isnan(flux));
    assert(!isnan(ss));
    assert(!isnan(sm));
    assert(!isnan(sp));
    return flux;
}


__device__
inline Real _hllc_pvel(const Real rm, const Real rp,
        const Real vm, const Real vp,
        const Real pm, const Real pp,
        const Real sm, const Real sp, const Real ss) // (27 MUL/ADD/SUB + 2 DIV) = 29 FLOP
{
    /* *
     * The flux computation is split into 4 parts:
     * 1.) Compute signum of s^*, compute s^- and s^+
     * 2.) Compute chi^* and delta of q^* and q
     * 3.) Compute trivial flux
     * 4.) Compute HLLC flux
     * */

    // 1.)
    const Real sign_star = (ss == 0.0f) ? 0.0f : ((ss < 0.0f) ? -1.0f : 1.0f);
    const Real s_minus  = fminf(0.0f, sm);
    const Real s_pluss  = fmaxf(0.0f, sp);

    // 2.)
    const Real chi_starm = (sm - vm) / (sm - ss);
    const Real chi_starp = (sp - vp) / (sp - ss);
    const Real qm        = rm*vm;
    const Real qp        = rp*vp;
    const Real q_deltam  = rm*ss*chi_starm - qm;
    const Real q_deltap  = rp*ss*chi_starp - qp;

    // 3.)
    const Real fm = qm*vm + pm;
    const Real fp = qp*vp + pp;

    // 4.)
    const Real flux = (0.5f*(1.0f + sign_star)) * (fm + s_minus*q_deltam) + (0.5f*(1.0f - sign_star)) * (fp + s_pluss*q_deltap);
    assert(!isnan(flux));
    assert(rm > 0);
    assert(rp > 0);
    return flux;
}


__device__
inline Real _hllc_e(const Real rm,  const Real rp,
        const Real vdm, const Real vdp,
        const Real v1m, const Real v1p,
        const Real v2m, const Real v2p,
        const Real pm,  const Real pp,
        const Real Gm,  const Real Gp,
        const Real Pm,  const Real Pp,
        const Real sm,  const Real sp,  const Real ss) // (55 MUL/ADD/SUB + 4 DIV) = 59 FLOP
{
    /* *
     * The flux computation is split into 4 parts:
     * 1.) Compute signum of s^*, compute s^- and s^+
     * 2.) Compute chi^* and delta of q^* and q
     * 3.) Compute trivial flux
     * 4.) Compute HLLC flux
     * */

    // 1.)
    const Real sign_star = (ss == 0.0f) ? 0.0f : ((ss < 0.0f) ? -1.0f : 1.0f);
    const Real s_minus  = fminf(0.0f, sm);
    const Real s_pluss  = fmaxf(0.0f, sp);

    // 2.)
    const Real chi_starm = (sm - vdm) / (sm - ss);
    const Real chi_starp = (sp - vdp) / (sp - ss);
    const Real qm        = Gm*pm + Pm + 0.5f*rm*(vdm*vdm + v1m*v1m + v2m*v2m);
    const Real qp        = Gp*pp + Pp + 0.5f*rp*(vdp*vdp + v1p*v1p + v2p*v2p);
    const Real q_deltam  = chi_starm*(qm + (ss - vdm)*(rm*ss + pm/(sm - vdm))) - qm;
    const Real q_deltap  = chi_starp*(qp + (ss - vdp)*(rp*ss + pp/(sp - vdp))) - qp;

    // 3.)
    const Real fm = vdm*(qm + pm);
    const Real fp = vdp*(qp + pp);

    // 4.)
    const Real flux = (0.5f*(1.0f + sign_star)) * (fm + s_minus*q_deltam) + (0.5f*(1.0f - sign_star)) * (fp + s_pluss*q_deltap);
    assert(!isnan(flux));
    return flux;
}


__device__
inline Real _extraterm_hllc_vel(const Real um, const Real up,
        const Real Gm, const Real Gp,
        const Real Pm, const Real Pp,
        const Real sm, const Real sp, const Real ss) // (17 MUL/ADD/SUB + 2 DIV) = 19 FLOP
{
    const Real sign_star = (ss == 0.0f) ? 0.0f : ((ss < 0.0f) ? -1.0f : 1.0f);
    const Real s_minus   = fminf(0.0f, sm);
    const Real s_pluss   = fmaxf(0.0f, sp);
    const Real chi_starm = (sm - um)/(sm - ss) - 1.0f;
    const Real chi_starp = (sp - up)/(sp - ss) - 1.0f;

    return (0.5f*(1.0f + sign_star))*(um + s_minus*chi_starm) + (0.5f*(1.0f - sign_star))*(up + s_pluss*chi_starp);
}


///////////////////////////////////////////////////////////////////////////////
//                                  KERNELS                                  //
///////////////////////////////////////////////////////////////////////////////
__global__
void _xextraterm_hllc(const uint_t nslices, DevicePointer divF, DevicePointer flux,
        const Real * const __restrict__ Gm, const Real * const __restrict__ Gp,
        const Real * const __restrict__ Pm, const Real * const __restrict__ Pp,
        const Real * const __restrict__ vel,
        Real * const __restrict__ sumG, Real * const __restrict__ sumP, Real * const __restrict__ divU)
{
    const uint_t ix = blockIdx.x * _TILE_DIM_ + threadIdx.x;
    const uint_t iy = blockIdx.y * _TILE_DIM_ + threadIdx.y;

    // limiting resource
    __shared__ Real smem1[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem2[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem3[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem4[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem5[_TILE_DIM_][_TILE_DIM_+1];
    __shared__ Real smem6[_TILE_DIM_][_TILE_DIM_+1];

    if (ix < NX && iy < NY)
    {
        // transpose
        const uint_t iyT = blockIdx.y * _TILE_DIM_ + threadIdx.x;
        const uint_t ixT = blockIdx.x * _TILE_DIM_ + threadIdx.y;

        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                const uint_t idxm = ID3(iyT,ixT+i,iz,NY,NXP1);
                const uint_t idxp = ID3(iyT,(ixT+1)+i,iz,NY,NXP1);

                // pre-fetch
                Real _sumG = Gp[idxm];
                Real _sumP = Pp[idxm];
                Real _divU = vel[idxp];
                _sumG = _sumG + Gm[idxp];
                _sumP = _sumP + Pm[idxp];
                _divU = _divU - vel[idxm];
                // read first batch
                smem1[threadIdx.x][threadIdx.y+i] = _sumG;
                smem2[threadIdx.x][threadIdx.y+i] = _sumP;
                smem3[threadIdx.x][threadIdx.y+i] = _divU;
            }
            __syncthreads();

            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                const uint_t idxm = ID3(iyT,ixT+i,iz,NY,NXP1);
                const uint_t idxp = ID3(iyT,(ixT+1)+i,iz,NY,NXP1);
                const uint_t idx = ID3(ix,iy+i,iz,NX,NY);

                // pre-fetch
                Real _divFr = flux.r[idxp];
                Real _divFu = flux.u[idxp];
                Real _divFv = flux.v[idxp];
                _divFr = _divFr - flux.r[idxm];
                _divFu = _divFu - flux.u[idxm];
                _divFv = _divFv - flux.v[idxm];
                // write first batch
                sumG[idx] = smem1[threadIdx.y+i][threadIdx.x];
                sumP[idx] = smem2[threadIdx.y+i][threadIdx.x];
                divU[idx] = smem3[threadIdx.y+i][threadIdx.x];
                // read second batch
                smem4[threadIdx.x][threadIdx.y+i] = _divFr;
                smem5[threadIdx.x][threadIdx.y+i] = _divFu;
                smem6[threadIdx.x][threadIdx.y+i] = _divFv;
            }
            __syncthreads();

            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                const uint_t idxm = ID3(iyT,ixT+i,iz,NY,NXP1);
                const uint_t idxp = ID3(iyT,(ixT+1)+i,iz,NY,NXP1);
                const uint_t idx = ID3(ix,iy+i,iz,NX,NY);

                // pre-fetch
                Real _divFw = flux.w[idxp];
                Real _divFe = flux.e[idxp];
                Real _divFG = flux.G[idxp];
                _divFw = _divFw - flux.w[idxm];
                _divFe = _divFe - flux.e[idxm];
                _divFG = _divFG - flux.G[idxm];
                // write second batch
                divF.r[idx] = smem4[threadIdx.y+i][threadIdx.x];
                divF.u[idx] = smem5[threadIdx.y+i][threadIdx.x];
                divF.v[idx] = smem6[threadIdx.y+i][threadIdx.x];
                // read third batch
                smem1[threadIdx.x][threadIdx.y+i] = _divFw;
                smem2[threadIdx.x][threadIdx.y+i] = _divFe;
                smem3[threadIdx.x][threadIdx.y+i] = _divFG;
            }
            __syncthreads();

            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                const uint_t idxm = ID3(iyT,ixT+i,iz,NY,NXP1);
                const uint_t idxp = ID3(iyT,(ixT+1)+i,iz,NY,NXP1);
                const uint_t idx = ID3(ix,iy+i,iz,NX,NY);

                // pre-fetch
                Real _divFP = flux.P[idxp];
                _divFP = _divFP - flux.P[idxm];
                // write third batch
                divF.w[idx] = smem1[threadIdx.y+i][threadIdx.x];
                divF.e[idx] = smem2[threadIdx.y+i][threadIdx.x];
                divF.G[idx] = smem3[threadIdx.y+i][threadIdx.x];
                // read fourth batch
                smem4[threadIdx.x][threadIdx.y+i] = _divFP;
            }
            __syncthreads();

            for (int i = 0; i < _TILE_DIM_; i += _BLOCK_ROWS_)
            {
                const uint_t idx = ID3(ix,iy+i,iz,NX,NY);
                // write fourth batch
                divF.P[idx] = smem4[threadIdx.y+i][threadIdx.x];
            }
            // NOTE: __syncthreads() can be omitted since it will not be
            // touched until next synchronization point
        }
    }
}


__global__
void _yextraterm_hllc(const uint_t nslices, DevicePointer divF, DevicePointer flux,
        const Real * const __restrict__ Gm, const Real * const __restrict__ Gp,
        const Real * const __restrict__ Pm, const Real * const __restrict__ Pp,
        const Real * const __restrict__ vel,
        Real * const __restrict__ sumG, Real * const __restrict__ sumP, Real * const __restrict__ divU)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx  = ID3(ix,iy,iz,NX,NY);
            const uint_t idxm = ID3(ix,iy,iz,NX,NYP1);
            const uint_t idxp = ID3(ix,(iy+1),iz,NX,NYP1);

            Real _sumG = Gp[idxm];
            Real _sumP = Pp[idxm];
            Real _divU = vel[idxp];
            Real _divFr = flux.r[idxp];
            Real _divFu = flux.u[idxp];
            Real _divFv = flux.v[idxp];
            Real _divFw = flux.w[idxp];
            Real _divFe = flux.e[idxp];
            Real _divFG = flux.G[idxp];
            Real _divFP = flux.P[idxp];
            _sumG = _sumG + Gm[idxp];
            _sumP = _sumP + Pm[idxp];
            _divU = _divU - vel[idxm];
            _divFr = _divFr - flux.r[idxm];
            _divFu = _divFu - flux.u[idxm];
            _divFv = _divFv - flux.v[idxm];
            _divFw = _divFw - flux.w[idxm];
            _divFe = _divFe - flux.e[idxm];
            _divFG = _divFG - flux.G[idxm];
            _divFP = _divFP - flux.P[idxm];

            sumG[idx] += _sumG;
            sumP[idx] += _sumP;
            divU[idx] += _divU;
            divF.r[idx] += _divFr;
            divF.u[idx] += _divFu;
            divF.v[idx] += _divFv;
            divF.w[idx] += _divFw;
            divF.e[idx] += _divFe;
            divF.G[idx] += _divFG;
            divF.P[idx] += _divFP;
        }
    }
}


__global__
void _zextraterm_hllc(const uint_t nslices, DevicePointer divF, DevicePointer flux,
        const Real * const __restrict__ Gm, const Real * const __restrict__ Gp,
        const Real * const __restrict__ Pm, const Real * const __restrict__ Pp,
        const Real * const __restrict__ vel,
        Real * const __restrict__ sumG, Real * const __restrict__ sumP, Real * const __restrict__ divU)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx  = ID3(ix,iy,iz,NX,NY);
            const uint_t idxm = ID3(ix,iy,iz,NX,NY);
            const uint_t idxp = ID3(ix,iy,(iz+1),NX,NY);

            const Real inv6 = 1.0f/6.0f;

            // cummulative sums of x and y
            const Real cumm_sumG = sumG[idx];
            const Real cumm_sumP = sumP[idx];
            const Real cumm_divU = divU[idx];

            Real _sumG = Gp[idxm];
            Real _sumP = Pp[idxm];
            Real _divU = vel[idxp];
            Real _divFr = flux.r[idxp];
            Real _divFu = flux.u[idxp];
            Real _divFv = flux.v[idxp];
            Real _divFw = flux.w[idxp];
            Real _divFe = flux.e[idxp];
            Real _divFG = flux.G[idxp];
            Real _divFP = flux.P[idxp];
            _sumG = _sumG + Gm[idxp] + cumm_sumG;
            _sumP = _sumP + Pm[idxp] + cumm_sumP;
            _divU = _divU - vel[idxm]+ cumm_divU;
            _divFr = _divFr - flux.r[idxm];
            _divFu = _divFu - flux.u[idxm];
            _divFv = _divFv - flux.v[idxm];
            _divFw = _divFw - flux.w[idxm];
            _divFe = _divFe - flux.e[idxm];
            _divFG = _divFG - flux.G[idxm];
            _divFP = _divFP - flux.P[idxm];

            // final divF
            divF.r[idx] += _divFr;
            divF.u[idx] += _divFu;
            divF.v[idx] += _divFv;
            divF.w[idx] += _divFw;
            divF.e[idx] += _divFe;
            divF.G[idx] += _divFG - inv6*_divU*_sumG;
            divF.P[idx] += _divFP - inv6*_divU*_sumP;
        }
    }
}


__global__
void _xflux00(const uint_t nslices, const uint_t global_iz,
        const DevicePointer ghostL, const DevicePointer ghostR, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NXP1 = NX + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * 6.) Reads textures 00
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    assert(NXP1 > 5);

    if (ix < NXP1 && iy < NY)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            // GMEM transactions are cached, effective GMEM accesses are 7*3
            // (according to nvvp)
            if (0 == ix)
                _load_3X00<0,0,3,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(3*GMEM + 3*tex_start)
            else if (1 == ix)
                _load_2X00<0,0,2,1>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(2*GMEM + 4*tex_start)
            else if (2 == ix)
                _load_1X00<0,0,1,2>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(1*GMEM + 5*tex_start)
            else if (NXP1-3 == ix)
                _load_1X00<NXP1-6,5,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NXP1-2 == ix)
                _load_2X00<NXP1-5,4,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NXP1-1 == ix)
                _load_3X00<NXP1-4,3,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else
                _load_internal_X00(ix, iy, iz, r, u, v, w, e, G, P, global_iz, NULL); // load 7*(6*tex_start)

            // compute body
#           include "xflux_body.cu"

            /* if (global_iz) */
            /* { */
/* #pragma unroll 6 */
            /*     for (uint_t i = 0; i < 6; ++i) */
            /*     { */
            /*         r[0] += r[i]; */
            /*         u[0] += u[i]; */
            /*         v[0] += v[i]; */
            /*         w[0] += w[i]; */
            /*         e[0] += e[i]; */
            /*         G[0] += G[i]; */
            /*         P[0] += P[i]; */
            /*     } */
            /* } */
            /* const uint_t idx = ID3(iy, ix, iz-3, NY, NXP1); */
            /* flux.r[idx] = r[0]; */
            /* flux.u[idx] = u[0]; */
            /* flux.v[idx] = v[0]; */
            /* flux.w[idx] = w[0]; */
            /* flux.e[idx] = e[0]; */
            /* flux.G[idx] = G[0]; */
            /* flux.P[idx] = P[0]; */
            /* xtra_vel[idx] = r[0]; */
            /* xtra_Gm[idx]  = w[0]; */
            /* xtra_Gp[idx]  = e[0]; */
            /* xtra_Pm[idx]  = P[0]; */
            /* xtra_Pp[idx]  = u[0]; */

            const uint_t idx = ID3(iy, ix, iz-3, NY, NXP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _xflux01(const uint_t nslices, const uint_t global_iz,
        const DevicePointer ghostL, const DevicePointer ghostR, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NXP1 = NX + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * 6.) Reads textures 01
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    assert(NXP1 > 5);

    if (ix < NXP1 && iy < NY)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            // GMEM transactions are cached, effective GMEM accesses are 7*3
            // (according to nvvp)
            if (0 == ix)
                _load_3X01<0,0,3,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(3*GMEM + 3*tex_start)
            else if (1 == ix)
                _load_2X01<0,0,2,1>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(2*GMEM + 4*tex_start)
            else if (2 == ix)
                _load_1X01<0,0,1,2>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(1*GMEM + 5*tex_start)
            else if (NXP1-3 == ix)
                _load_1X01<NXP1-6,5,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NXP1-2 == ix)
                _load_2X01<NXP1-5,4,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NXP1-1 == ix)
                _load_3X01<NXP1-4,3,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else
                _load_internal_X01(ix, iy, iz, r, u, v, w, e, G, P, global_iz, NULL); // load 7*(6*tex_start)

            // compute body
#           include "xflux_body.cu"

            const uint_t idx = ID3(iy, ix, iz-3, NY, NXP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _yflux00(const uint_t nslices, const uint_t global_iz,
        const DevicePointer ghostL, const DevicePointer ghostR, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NYP1 = NY + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * 6.) Reads texture 00
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    assert(NYP1 > 5);

    if (ix < NX && iy < NYP1)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            // GMEM transactions are cached, effective GMEM accesses are 7*3
            // (according to nvvp)
            if (0 == iy)
                _load_3Y00<0,0,3,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(3*GMEM + 3*TEX)
            else if (1 == iy)
                _load_2Y00<0,0,2,1>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(2*GMEM + 4*TEX)
            else if (2 == iy)
                _load_1Y00<0,0,1,2>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(1*GMEM + 5*TEX)
            else if (NYP1-3 == iy)
                _load_1Y00<NYP1-6,5,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NYP1-2 == iy)
                _load_2Y00<NYP1-5,4,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NYP1-1 == iy)
                _load_3Y00<NYP1-4,3,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else
                _load_internal_Y00(ix, iy, iz, r, u, v, w, e, G, P, global_iz, NULL); // load 7*(6*TEX)

            // compute body
#           include "yflux_body.cu"

            /* if (global_iz) */
            /* { */
/* #pragma unroll 6 */
            /*     for (uint_t i = 0; i < 6; ++i) */
            /*     { */
            /*         r[0] += r[i]; */
            /*         u[0] += u[i]; */
            /*         v[0] += v[i]; */
            /*         w[0] += w[i]; */
            /*         e[0] += e[i]; */
            /*         G[0] += G[i]; */
            /*         P[0] += P[i]; */
            /*     } */
            /* } */
            /* const uint_t idx = ID3(ix, iy, iz-3, NX, NYP1); */
            /* flux.r[idx] = r[0]; */
            /* flux.u[idx] = u[0]; */
            /* flux.v[idx] = v[0]; */
            /* flux.w[idx] = w[0]; */
            /* flux.e[idx] = e[0]; */
            /* flux.G[idx] = G[0]; */
            /* flux.P[idx] = P[0]; */
            /* xtra_vel[idx] = r[0]; */
            /* xtra_Gm[idx]  = w[0]; */
            /* xtra_Gp[idx]  = e[0]; */
            /* xtra_Pm[idx]  = P[0]; */
            /* xtra_Pp[idx]  = u[0]; */

            const uint_t idx = ID3(ix, iy, iz-3, NX, NYP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _yflux01(const uint_t nslices, const uint_t global_iz,
        const DevicePointer ghostL, const DevicePointer ghostR, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NYP1 = NY + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * 6.) Reads texture 01
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    assert(NYP1 > 5);

    if (ix < NX && iy < NYP1)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            // GMEM transactions are cached, effective GMEM accesses are 7*3
            // (according to nvvp)
            if (0 == iy)
                _load_3Y01<0,0,3,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(3*GMEM + 3*TEX)
            else if (1 == iy)
                _load_2Y01<0,0,2,1>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(2*GMEM + 4*TEX)
            else if (2 == iy)
                _load_1Y01<0,0,1,2>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostL); // load 7*(1*GMEM + 5*TEX)
            else if (NYP1-3 == iy)
                _load_1Y01<NYP1-6,5,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NYP1-2 == iy)
                _load_2Y01<NYP1-5,4,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else if (NYP1-1 == iy)
                _load_3Y01<NYP1-4,3,0,0>(ix, iy, iz, r, u, v, w, e, G, P, global_iz, &ghostR);
            else
                _load_internal_Y01(ix, iy, iz, r, u, v, w, e, G, P, global_iz, NULL); // load 7*(6*TEX)

            // compute body
#           include "yflux_body.cu"

            const uint_t idx = ID3(ix, iy, iz-3, NX, NYP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _zflux00(const uint_t nslices, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NX = NodeBlock::sizeX
     * 2.) NY = NodeBlock::sizeY
     * 3.) NZ = NodeBlock::sizeZ
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) Reads texture 00
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    // depends on boundary condition in z-direction
    assert(NodeBlock::sizeZ > 0);

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 3; iz < (nslices+1)+3; ++iz) // first and last 3 slices are zghosts; need to compute nslices+1 fluxes in z-direction
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            _load_internal_Z00(ix, iy, iz, r, u, v, w, e, G, P, 0, NULL); // load 7*(6*TEX)

            // compute body
#           include "zflux_body.cu"

            /* if (global_iz) */
            /* { */
/* #pragma unroll 6 */
            /*     for (uint_t i = 0; i < 6; ++i) */
            /*     { */
            /*         r[0] += r[i]; */
            /*         u[0] += u[i]; */
            /*         v[0] += v[i]; */
            /*         w[0] += w[i]; */
            /*         e[0] += e[i]; */
            /*         G[0] += G[i]; */
            /*         P[0] += P[i]; */
            /*     } */
            /* } */
            /* const uint_t idx = ID3(ix, iy, iz-3, NX, NY); */
            /* flux.r[idx] = r[0]; */
            /* flux.u[idx] = u[0]; */
            /* flux.v[idx] = v[0]; */
            /* flux.w[idx] = w[0]; */
            /* flux.e[idx] = e[0]; */
            /* flux.G[idx] = G[0]; */
            /* flux.P[idx] = P[0]; */
            /* xtra_vel[idx] = r[0]; */
            /* xtra_Gm[idx]  = w[0]; */
            /* xtra_Gp[idx]  = e[0]; */
            /* xtra_Pm[idx]  = P[0]; */
            /* xtra_Pp[idx]  = u[0]; */

            const uint_t idx = ID3(ix, iy, iz-3, NX, NY);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _zflux01(const uint_t nslices, DevicePointer flux,
        Real * const __restrict__ xtra_vel,
        Real * const __restrict__ xtra_Gm, Real * const __restrict__ xtra_Gp,
        Real * const __restrict__ xtra_Pm, Real * const __restrict__ xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NX = NodeBlock::sizeX
     * 2.) NY = NodeBlock::sizeY
     * 3.) NZ = NodeBlock::sizeZ
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) Reads texture 01
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    // depends on boundary condition in z-direction
    assert(NodeBlock::sizeZ > 0);

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 3; iz < (nslices+1)+3; ++iz) // first and last 3 slices are zghosts; need to compute nslices+1 fluxes in z-direction
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load stencils
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // stencils (7 * _STENCIL_WIDTH_ registers per thread)
            Real r[6];
            Real u[6];
            Real v[6];
            Real w[6];
            Real e[6];
            Real G[6];
            Real P[6];

            // 1.)
            _load_internal_Z01(ix, iy, iz, r, u, v, w, e, G, P, 0, NULL); // load 7*(6*TEX)

            // compute body
#           include "zflux_body.cu"

            const uint_t idx = ID3(ix, iy, iz-3, NX, NY);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = hllc_vel;
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
}


__global__
void _maxSOS(const uint_t nslices, int* g_maxSOS)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    const uint_t loc_idx = blockDim.x * threadIdx.y + threadIdx.x;
    __shared__ Real block_sos[_NTHREADS_];
    block_sos[loc_idx] = 0.0f;

    if (ix < NX && iy < NY)
    {
        Real sos = 0.0f;

        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            // TODO: used both buffers here
            const Real r = tex3D(texR00, ix, iy, iz);
            const Real u = tex3D(texU00, ix, iy, iz);
            const Real v = tex3D(texV00, ix, iy, iz);
            const Real w = tex3D(texW00, ix, iy, iz);
            const Real e = tex3D(texE00, ix, iy, iz);
            const Real G = tex3D(texG00, ix, iy, iz);
            const Real P = tex3D(texP00, ix, iy, iz);

            const Real p = (e - (u*u + v*v + w*w)*(0.5f/r) - P) / G;
            const Real c = sqrtf(((p + P) / G + p) / r);

            sos = fmaxf(sos, c + fmaxf(fmaxf(fabsf(u), fabsf(v)), fabsf(w)) / r);
        }
        block_sos[loc_idx] = sos;
        __syncthreads();

        if (0 == loc_idx)
        {
            for (int i = 1; i < _NTHREADS_; ++i)
                sos = fmaxf(sos, block_sos[i]);
            assert(sos > 0.0f);
            atomicMax(g_maxSOS, __float_as_int(sos));
        }
    }
}

///////////////////////////////////////////////////////////////////////////////
//                              KERNEL WRAPPERS                              //
///////////////////////////////////////////////////////////////////////////////
static void _bindTexture(texture<float, 3, hipReadModeElementType> * const tex, hipArray_t d_ptr)
{
    hipChannelFormatDesc fmt = hipCreateChannelDesc<Real>();
    tex->addressMode[0]       = hipAddressModeClamp;
    tex->addressMode[1]       = hipAddressModeClamp;
    tex->addressMode[2]       = hipAddressModeClamp;
    tex->channelDesc          = fmt;
    tex->filterMode           = hipFilterModePoint;
    tex->mipmapFilterMode     = hipFilterModePoint;
    tex->normalized           = false;

    hipBindTextureToArray(tex, d_ptr, &fmt);
}


void GPU::compute_pipe_divF(const uint_t nslices, const uint_t global_iz,
        const uint_t gbuf_id, const int chunk_id)
{
#ifndef _MUTE_GPU_
    assert(gbuf_id < _NUM_GPU_BUF_);

    /* *
     * Compute div(F)
     * */

    // my stream
    const uint_t s_id = chunk_id % _NUM_STREAMS_;

    // my data
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    // my ghosts
    DevicePointer xghostL(mybuf->d_xgl);
    DevicePointer xghostR(mybuf->d_xgr);
    DevicePointer yghostL(mybuf->d_ygl);
    DevicePointer yghostR(mybuf->d_ygr);

    // my output
    DevicePointer divF(mybuf->d_divF);

    // my tmp storage
    DevicePointer flux(d_flux);

    // my launch config
    const dim3 X_blocks(1, _NTHREADS_, 1);
    const dim3 X_grid(NXP1, (NY + _NTHREADS_ -1)/_NTHREADS_, 1);
    const dim3 X_xtraBlocks(_TILE_DIM_, _BLOCK_ROWS_, 1);
    const dim3 X_xtraGrid((NX + _TILE_DIM_ - 1)/_TILE_DIM_, (NY + _TILE_DIM_ - 1)/_TILE_DIM_, 1);

    const dim3 Y_blocks(_NTHREADS_, 1, 1);
    const dim3 Y_grid((NX + _NTHREADS_ -1) / _NTHREADS_, NYP1, 1);
    const dim3 Y_xtraGrid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);

    const dim3 Z_blocks(_NTHREADS_, 1, 1);
    const dim3 Z_grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);

    // previous stream has priority. Since resources are limited, concurrent
    // kernels make less sense
    const uint_t s_idm1 = ((chunk_id-1) + _NUM_STREAMS_) % _NUM_STREAMS_;
    assert(s_idm1 < _NUM_STREAMS_);
    hipStreamWaitEvent(stream[s_id], event_compute[s_idm1], 0);

    char prof_item[256];

    // queue kernels in pipe
    switch (gbuf_id)
    {
        case 0:
            _bindTexture(&texR00, mybuf->d_GPUin[0]);
            _bindTexture(&texU00, mybuf->d_GPUin[1]);
            _bindTexture(&texV00, mybuf->d_GPUin[2]);
            _bindTexture(&texW00, mybuf->d_GPUin[3]);
            _bindTexture(&texE00, mybuf->d_GPUin[4]);
            _bindTexture(&texG00, mybuf->d_GPUin[5]);
            _bindTexture(&texP00, mybuf->d_GPUin[6]);
            // --- X ---
            sprintf(prof_item, "_XFLUX (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _xflux00<<<X_grid, X_blocks, 0, stream[s_id]>>>(nslices, global_iz, xghostL, xghostR, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
            GPU::profiler.pop_stopCUDA();

            sprintf(prof_item, "_XEXTRATERM (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _xextraterm_hllc<<<X_xtraGrid, X_xtraBlocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
            GPU::profiler.pop_stopCUDA();

            // --- Y ---
            sprintf(prof_item, "_YFLUX (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _yflux00<<<Y_grid, Y_blocks, 0, stream[s_id]>>>(nslices, global_iz, yghostL, yghostR, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
            GPU::profiler.pop_stopCUDA();

            sprintf(prof_item, "_YEXTRATERM (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _yextraterm_hllc<<<Y_xtraGrid, Y_blocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
            GPU::profiler.pop_stopCUDA();

            // --- Z ---
            sprintf(prof_item, "_ZFLUX (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _zflux00<<<Z_grid, Z_blocks, 0, stream[s_id]>>>(nslices, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
            GPU::profiler.pop_stopCUDA();

            sprintf(prof_item, "_ZEXTRATERM (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _zextraterm_hllc<<<Z_grid, Z_blocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
            GPU::profiler.pop_stopCUDA();
            break;

        case 1:
            _bindTexture(&texR01, mybuf->d_GPUin[0]);
            _bindTexture(&texU01, mybuf->d_GPUin[1]);
            _bindTexture(&texV01, mybuf->d_GPUin[2]);
            _bindTexture(&texW01, mybuf->d_GPUin[3]);
            _bindTexture(&texE01, mybuf->d_GPUin[4]);
            _bindTexture(&texG01, mybuf->d_GPUin[5]);
            _bindTexture(&texP01, mybuf->d_GPUin[6]);
            // --- X ---
            sprintf(prof_item, "_XFLUX (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _xflux01<<<X_grid, X_blocks, 0, stream[s_id]>>>(nslices, global_iz, xghostL, xghostR, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
            GPU::profiler.pop_stopCUDA();

            sprintf(prof_item, "_XEXTRATERM (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _xextraterm_hllc<<<X_xtraGrid, X_xtraBlocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
            GPU::profiler.pop_stopCUDA();

            // --- Y ---
            sprintf(prof_item, "_YFLUX (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _yflux01<<<Y_grid, Y_blocks, 0, stream[s_id]>>>(nslices, global_iz, yghostL, yghostR, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
            GPU::profiler.pop_stopCUDA();

            sprintf(prof_item, "_YEXTRATERM (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _yextraterm_hllc<<<Y_xtraGrid, Y_blocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
            GPU::profiler.pop_stopCUDA();

            // --- Z ---
            sprintf(prof_item, "_ZFLUX (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _zflux01<<<Z_grid, Z_blocks, 0, stream[s_id]>>>(nslices, flux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
            GPU::profiler.pop_stopCUDA();

            sprintf(prof_item, "_ZEXTRATERM (s_id=%d)", s_id);
            GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
            _zextraterm_hllc<<<Z_grid, Z_blocks, 0, stream[s_id]>>>(nslices, divF, flux, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
            GPU::profiler.pop_stopCUDA();
            break;
    }

    hipEventRecord(event_compute[s_id], stream[s_id]);
#endif
}


void GPU::MaxSpeedOfSound(const uint_t nslices, const uint_t gbuf_id, const int chunk_id)
{
#ifndef _MUTE_GPU_
    assert(gbuf_id < _NUM_GPU_BUF_);

    // my stream
    const uint_t s_id = chunk_id % _NUM_STREAMS_;

    // my data
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    _bindTexture(&texR00, mybuf->d_GPUin[0]);
    _bindTexture(&texU00, mybuf->d_GPUin[1]);
    _bindTexture(&texV00, mybuf->d_GPUin[2]);
    _bindTexture(&texW00, mybuf->d_GPUin[3]);
    _bindTexture(&texE00, mybuf->d_GPUin[4]);
    _bindTexture(&texG00, mybuf->d_GPUin[5]);
    _bindTexture(&texP00, mybuf->d_GPUin[6]);

    // my launch config
    const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
    const dim3 blocks(_NTHREADS_, 1, 1);

    char prof_item[256];

    sprintf(prof_item, "_MAXSOS (s_id=%d)", s_id);
    GPU::profiler.push_startCUDA(prof_item, &stream[s_id]);
    _maxSOS<<<grid, blocks, 0, stream[s_id]>>>(nslices, d_maxSOS);
    GPU::profiler.pop_stopCUDA();
#endif
}

///////////////////////////////////////////////////////////////////////////
// TEST SECTION
///////////////////////////////////////////////////////////////////////////
void GPU::TestKernel()
{
    const uint_t gbuf_id = 0;
    const uint_t s_id = 0;

    // my data
    GPU_COMM * const mybuf = &gpu_comm[gbuf_id];

    _bindTexture(&texR00, mybuf->d_GPUin[0]);
    _bindTexture(&texU00, mybuf->d_GPUin[1]);
    _bindTexture(&texV00, mybuf->d_GPUin[2]);
    _bindTexture(&texW00, mybuf->d_GPUin[3]);
    _bindTexture(&texE00, mybuf->d_GPUin[4]);
    _bindTexture(&texG00, mybuf->d_GPUin[5]);
    _bindTexture(&texP00, mybuf->d_GPUin[6]);

    // my ghosts
    DevicePointer xghostL(mybuf->d_xgl);
    DevicePointer xghostR(mybuf->d_xgr);
    DevicePointer yghostL(mybuf->d_ygl);
    DevicePointer yghostR(mybuf->d_ygr);

    // my output
    DevicePointer divF(mybuf->d_divF);

    // my tmp storage
    DevicePointer flux(d_flux);

    hipFree(d_Gm);
    hipFree(d_Gp);
    hipFree(d_Pm);
    hipFree(d_Pp);
    hipFree(d_hllc_vel);
    hipFree(d_sumG);
    hipFree(d_sumP);
    hipFree(d_divU);

    const uint_t nslices = NodeBlock::sizeZ;
    const uint_t xflxSize = (NodeBlock::sizeX+1)*NodeBlock::sizeY*nslices;
    const uint_t yflxSize = NodeBlock::sizeX*(NodeBlock::sizeY+1)*nslices;
    const uint_t zflxSize = NodeBlock::sizeX*NodeBlock::sizeY*(nslices+1);

    Real *d_extra_X[5];
    Real *d_extra_Y[5];
    Real *d_extra_Z[5];
    for (int i = 0; i < 5; ++i)
    {
        hipMalloc(&(d_extra_X[i]), xflxSize * sizeof(Real));
        hipMalloc(&(d_extra_Y[i]), yflxSize * sizeof(Real));
        hipMalloc(&(d_extra_Z[i]), zflxSize * sizeof(Real));
    }
    GPU::tell_memUsage_GPU();


    {

        const dim3 xblocks(1, _NTHREADS_, 1);
        const dim3 yblocks(_NTHREADS_, 1, 1);
        const dim3 zblocks(_NTHREADS_, 1, 1);
        const dim3 xgrid(NXP1, (NY + _NTHREADS_ - 1) / _NTHREADS_,   1);
        const dim3 ygrid((NX   + _NTHREADS_ - 1) / _NTHREADS_, NYP1, 1);
        const dim3 zgrid((NX   + _NTHREADS_ - 1) / _NTHREADS_, NY,   1);

        GPU::profiler.push_startCUDA("_XFLUX", &stream[s_id]);
        _xflux00<<<xgrid, xblocks, 0, stream[s_id]>>>(nslices, 0, xghostL, xghostR, flux, d_extra_X[0], d_extra_X[1], d_extra_X[2], d_extra_X[3], d_extra_X[4]);
        GPU::profiler.pop_stopCUDA();

        /* GPU::profiler.push_startCUDA("_YFLUX", &_s[0]); */
        /* _yflux<<<ygrid, yblocks, 0, _s[0]>>>(nslices, 0, yghostL, yghostR, flux, d_extra_Y[0], d_extra_Y[1], d_extra_Y[2], d_extra_Y[3], d_extra_Y[4]); */
        /* GPU::profiler.pop_stopCUDA(); */

        /* GPU::profiler.push_startCUDA("_ZFLUX", &_s[0]); */
        /* _zflux<<<zgrid, zblocks, 0, _s[0]>>>(nslices, flux, d_extra_Z[0], d_extra_Z[1], d_extra_Z[2], d_extra_Z[3], d_extra_Z[4]); */
        /* GPU::profiler.pop_stopCUDA(); */

        /* _xflux<<<xgrid, xblocks, 0, _s[0]>>>(nslices, 0, xghostL, xghostR, flux, d_extra_X[0], d_extra_X[1], d_extra_X[2], d_extra_X[3], d_extra_X[4]); */
        /* _yflux<<<ygrid, yblocks, 0, _s[1]>>>(nslices, 0, yghostL, yghostR, flux, d_extra_Y[0], d_extra_Y[1], d_extra_Y[2], d_extra_Y[3], d_extra_Y[4]); */
        /* _zflux<<<zgrid, zblocks, 0, _s[2]>>>(nslices, flux, d_extra_Z[0], d_extra_Z[1], d_extra_Z[2], d_extra_Z[3], d_extra_Z[4]); */

        hipDeviceSynchronize();
    }
}
