#include "hip/hip_runtime.h"
/* *
 * GPUkernels.cu
 *
 * Created by Fabian Wermelinger on 6/25/14.
 * Copyright 2014 ETH Zurich. All rights reserved.
 * */
#include <assert.h>
#include <stdio.h>
#include <vector>

#include "GPU.h" // includes Types.h & wrapper declarations
#include "GPUonly.cuh"

#ifdef _CUDA_TIMER_
#include "CUDA_Timer.cuh"
#endif


///////////////////////////////////////////////////////////////////////////////
//                             DEVICE FUNCTIONS                              //
///////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////////////////////////////////////////
//                                  KERNELS                                  //
///////////////////////////////////////////////////////////////////////////////
__global__
void _xextraterm_hllc(const uint_t nslices,
        const Real * const Gm, const Real * const Gp,
        const Real * const Pm, const Real * const Pp,
        const Real * const vel,
        Real * const sumG, Real * const sumP, Real * const divU)
{
    /* *
     * Computes x-contribution for the right hand side of the advection
     * equations.  Maps two values on cell faces to one value at the cell
     * center.  NOTE: The assignment here is "="
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx  = ID3(ix,   iy, iz, NX,   NY);
            const uint_t idxm = ID3(ix,   iy, iz, NXP1, NY);
            const uint_t idxp = ID3(ix+1, iy, iz, NXP1, NY);
            sumG[idx] = Gp[idxm]  + Gm[idxp];
            sumP[idx] = Pp[idxm]  + Pm[idxp];
            divU[idx] = vel[idxp] - vel[idxm];
        }
    }
}


__global__
void _xflux(const uint_t nslices, const uint_t global_iz,
        devPtrSet ghostL, devPtrSet ghostR, devPtrSet flux,
        Real * const xtra_vel,
        Real * const xtra_Gm, Real * const xtra_Gp,
        Real * const xtra_Pm, Real * const xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NXP1 = NX + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    /* *
     * The following code requires 1D thread blocks with dimension
     * dim3(_NTHREADS_, 1, 1)
     * on a 2D grid with dimension
     * dim3((NXP1 + _NTHREADS_ - 1) / _NTHREADS_, NY, 1)
     *
     * The load of ghosts is organized into a switch block with 6 cases. Blocks
     * affected by this are the first three on the left boundary and the last
     * three on the right. Given the layout of thread blocks above, warps do
     * not diverge because of the switch.
     *
     * NOTE: To minimize the switch cases to 6 (and simplify code) the
     * following requires that NX >= 5
     * */
    assert(NXP1 > 5);

#if 1
    if (ix < NXP1 && iy < NY)
    {
        Stencil r, u, v, w, e, G, P;
        Stencil p; // for reconstruction
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load ghosts from GMEM or tex3D into stencil (do this 7x, for each
             *     quantity)
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // 1.)
#if 1
            // more conditionals
            _read_stencil_X(r, texR, ghostL.r, ghostR.r, ix, iy, iz, global_iz);
            _read_stencil_X(u, texU, ghostL.u, ghostR.u, ix, iy, iz, global_iz);
            _read_stencil_X(v, texV, ghostL.v, ghostR.v, ix, iy, iz, global_iz);
            _read_stencil_X(w, texW, ghostL.w, ghostR.w, ix, iy, iz, global_iz);
            _read_stencil_X(e, texE, ghostL.e, ghostR.e, ix, iy, iz, global_iz);
            _read_stencil_X(G, texG, ghostL.G, ghostR.G, ix, iy, iz, global_iz);
            _read_stencil_X(P, texP, ghostL.P, ghostR.P, ix, iy, iz, global_iz);
#else
            // less conditionals, additional tex3D fetches
            _load_stencil_tex3D_X(r, texR, ix, iy, iz);
            _load_stencil_tex3D_X(u, texU, ix, iy, iz);
            _load_stencil_tex3D_X(v, texV, ix, iy, iz);
            _load_stencil_tex3D_X(w, texW, ix, iy, iz);
            _load_stencil_tex3D_X(e, texE, ix, iy, iz);
            _load_stencil_tex3D_X(G, texG, ix, iy, iz);
            _load_stencil_tex3D_X(P, texP, ix, iy, iz);
            switch (ix)
            {
                case 0:
                    _load_3ghosts_X(r.im3, r.im2, r.im1, ghostL.r, iy, iz-3+global_iz);
                    _load_3ghosts_X(u.im3, u.im2, u.im1, ghostL.u, iy, iz-3+global_iz);
                    _load_3ghosts_X(v.im3, v.im2, v.im1, ghostL.v, iy, iz-3+global_iz);
                    _load_3ghosts_X(w.im3, w.im2, w.im1, ghostL.w, iy, iz-3+global_iz);
                    _load_3ghosts_X(e.im3, e.im2, e.im1, ghostL.e, iy, iz-3+global_iz);
                    _load_3ghosts_X(G.im3, G.im2, G.im1, ghostL.G, iy, iz-3+global_iz);
                    _load_3ghosts_X(P.im3, P.im2, P.im1, ghostL.P, iy, iz-3+global_iz);
                    break;
                case 1:
                    _load_2ghosts_X(r.im3, r.im2, 1, 2, ghostL.r, iy, iz-3+global_iz);
                    _load_2ghosts_X(u.im3, u.im2, 1, 2, ghostL.u, iy, iz-3+global_iz);
                    _load_2ghosts_X(v.im3, v.im2, 1, 2, ghostL.v, iy, iz-3+global_iz);
                    _load_2ghosts_X(w.im3, w.im2, 1, 2, ghostL.w, iy, iz-3+global_iz);
                    _load_2ghosts_X(e.im3, e.im2, 1, 2, ghostL.e, iy, iz-3+global_iz);
                    _load_2ghosts_X(G.im3, G.im2, 1, 2, ghostL.G, iy, iz-3+global_iz);
                    _load_2ghosts_X(P.im3, P.im2, 1, 2, ghostL.P, iy, iz-3+global_iz);
                    break;
                case 2:
                    _load_1ghost_X(r.im3, 2, ghostL.r, iy, iz-3+global_iz);
                    _load_1ghost_X(u.im3, 2, ghostL.u, iy, iz-3+global_iz);
                    _load_1ghost_X(v.im3, 2, ghostL.v, iy, iz-3+global_iz);
                    _load_1ghost_X(w.im3, 2, ghostL.w, iy, iz-3+global_iz);
                    _load_1ghost_X(e.im3, 2, ghostL.e, iy, iz-3+global_iz);
                    _load_1ghost_X(G.im3, 2, ghostL.G, iy, iz-3+global_iz);
                    _load_1ghost_X(P.im3, 2, ghostL.P, iy, iz-3+global_iz);
                    break;
                case (NXP1-3):
                    _load_1ghost_X(r.ip2, 0, ghostR.r, iy, iz-3+global_iz);
                    _load_1ghost_X(u.ip2, 0, ghostR.u, iy, iz-3+global_iz);
                    _load_1ghost_X(v.ip2, 0, ghostR.v, iy, iz-3+global_iz);
                    _load_1ghost_X(w.ip2, 0, ghostR.w, iy, iz-3+global_iz);
                    _load_1ghost_X(e.ip2, 0, ghostR.e, iy, iz-3+global_iz);
                    _load_1ghost_X(G.ip2, 0, ghostR.G, iy, iz-3+global_iz);
                    _load_1ghost_X(P.ip2, 0, ghostR.P, iy, iz-3+global_iz);
                    break;
                case (NXP1-2):
                    _load_2ghosts_X(r.ip1, r.ip2, 0, 1, ghostR.r, iy, iz-3+global_iz);
                    _load_2ghosts_X(u.ip1, u.ip2, 0, 1, ghostR.u, iy, iz-3+global_iz);
                    _load_2ghosts_X(v.ip1, v.ip2, 0, 1, ghostR.v, iy, iz-3+global_iz);
                    _load_2ghosts_X(w.ip1, w.ip2, 0, 1, ghostR.w, iy, iz-3+global_iz);
                    _load_2ghosts_X(e.ip1, e.ip2, 0, 1, ghostR.e, iy, iz-3+global_iz);
                    _load_2ghosts_X(G.ip1, G.ip2, 0, 1, ghostR.G, iy, iz-3+global_iz);
                    _load_2ghosts_X(P.ip1, P.ip2, 0, 1, ghostR.P, iy, iz-3+global_iz);
                    break;
                case (NXP1-1):
                    _load_3ghosts_X(r.i, r.ip1, r.ip2, ghostR.r, iy, iz-3+global_iz);
                    _load_3ghosts_X(u.i, u.ip1, u.ip2, ghostR.u, iy, iz-3+global_iz);
                    _load_3ghosts_X(v.i, v.ip1, v.ip2, ghostR.v, iy, iz-3+global_iz);
                    _load_3ghosts_X(w.i, w.ip1, w.ip2, ghostR.w, iy, iz-3+global_iz);
                    _load_3ghosts_X(e.i, e.ip1, e.ip2, ghostR.e, iy, iz-3+global_iz);
                    _load_3ghosts_X(G.i, G.ip1, G.ip2, ghostR.G, iy, iz-3+global_iz);
                    _load_3ghosts_X(P.i, P.ip1, P.ip2, ghostR.P, iy, iz-3+global_iz);
                    break;
            } // end switch
#endif
            assert(r > 0);
            assert(e > 0);
            assert(G > 0);
            assert(P >= 0);

            // 2.)
            // rho
            const Real rp = _weno_pluss_clipped(r.im2, r.im1, r.i, r.ip1, r.ip2);
            const Real rm = _weno_minus_clipped(r.im3, r.im2, r.im1, r.i, r.ip1);
            assert(!isnan(rp)); assert(!isnan(rm));
            // u (convert primitive variable u = (rho*u) / rho)
            u.im3 /= r.im3;
            u.im2 /= r.im2;
            u.im1 /= r.im1;
            u.i   /= r.i;
            u.ip1 /= r.ip1;
            u.ip2 /= r.ip2;
            const Real up = _weno_pluss_clipped(u.im2, u.im1, u.i, u.ip1, u.ip2);
            const Real um = _weno_minus_clipped(u.im3, u.im2, u.im1, u.i, u.ip1);
            assert(!isnan(up)); assert(!isnan(um));
            // v (convert primitive variable v = (rho*v) / rho)
            v.im3 /= r.im3;
            v.im2 /= r.im2;
            v.im1 /= r.im1;
            v.i   /= r.i;
            v.ip1 /= r.ip1;
            v.ip2 /= r.ip2;
            const Real vp = _weno_pluss_clipped(v.im2, v.im1, v.i, v.ip1, v.ip2);
            const Real vm = _weno_minus_clipped(v.im3, v.im2, v.im1, v.i, v.ip1);
            assert(!isnan(vp)); assert(!isnan(vm));
            // w (convert primitive variable w = (rho*w) / rho)
            w.im3 /= r.im3;
            w.im2 /= r.im2;
            w.im1 /= r.im1;
            w.i   /= r.i;
            w.ip1 /= r.ip1;
            w.ip2 /= r.ip2;
            const Real wp = _weno_pluss_clipped(w.im2, w.im1, w.i, w.ip1, w.ip2);
            const Real wm = _weno_minus_clipped(w.im3, w.im2, w.im1, w.i, w.ip1);
            assert(!isnan(wp)); assert(!isnan(wm));
            // p (convert primitive variable p = (e - 0.5*rho*(u*u + v*v + w*w) - P) / G
            p.im3 = (e.im3 - 0.5f*r.im3*(u.im3*u.im3 + v.im3*v.im3 + w.im3*w.im3) - P.im3) / G.im3;
            p.im2 = (e.im2 - 0.5f*r.im2*(u.im2*u.im2 + v.im2*v.im2 + w.im2*w.im2) - P.im2) / G.im2;
            p.im1 = (e.im1 - 0.5f*r.im1*(u.im1*u.im1 + v.im1*v.im1 + w.im1*w.im1) - P.im1) / G.im1;
            p.i   = (e.i   - 0.5f*r.i*(u.i*u.i       + v.i*v.i     + w.i*w.i)     - P.i)   / G.i;
            p.ip1 = (e.ip1 - 0.5f*r.ip1*(u.ip1*u.ip1 + v.ip1*v.ip1 + w.ip1*w.ip1) - P.ip1) / G.ip1;
            p.ip2 = (e.ip2 - 0.5f*r.ip2*(u.ip2*u.ip2 + v.ip2*v.ip2 + w.ip2*w.ip2) - P.ip2) / G.ip2;
            const Real pp = _weno_pluss_clipped(p.im2, p.im1, p.i, p.ip1, p.ip2);
            const Real pm = _weno_minus_clipped(p.im3, p.im2, p.im1, p.i, p.ip1);
            assert(!isnan(pp)); assert(!isnan(pm));
            // G
            const Real Gp = _weno_pluss_clipped(G.im2, G.im1, G.i, G.ip1, G.ip2);
            const Real Gm = _weno_minus_clipped(G.im3, G.im2, G.im1, G.i, G.ip1);
            assert(!isnan(Gp)); assert(!isnan(Gm));
            // P
            const Real Pp = _weno_pluss_clipped(P.im2, P.im1, P.i, P.ip1, P.ip2);
            const Real Pm = _weno_minus_clipped(P.im3, P.im2, P.im1, P.i, P.ip1);
            assert(!isnan(Pp)); assert(!isnan(Pm));

            // 3.)
            Real sm, sp;
            _char_vel_einfeldt(rm, rp, um, up, pm, pp, Gm, Gp, Pm, Pp, sm, sp);
            const Real ss = _char_vel_star(rm, rp, um, up, pm, pp, sm, sp);
            assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

            // 4.)
            const Real fr = _hllc_rho(rm, rp, um, up, sm, sp, ss);
            const Real fu = _hllc_pvel(rm, rp, um, up, pm, pp, sm, sp, ss);
            const Real fv = _hllc_vel(rm, rp, vm, vp, um, up, sm, sp, ss);
            const Real fw = _hllc_vel(rm, rp, wm, wp, um, up, sm, sp, ss);
            const Real fe = _hllc_e(rm, rp, um, up, vm, vp, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss);
            const Real fG = _hllc_rho(Gm, Gp, um, up, sm, sp, ss);
            const Real fP = _hllc_rho(Pm, Pp, um, up, sm, sp, ss);
            assert(!isnan(fr)); assert(!isnan(fu)); assert(!isnan(fv)); assert(!isnan(fw)); assert(!isnan(fe)); assert(!isnan(fG)); assert(!isnan(fP));

            const uint_t idx = ID3(ix, iy, iz-3, NXP1, NY);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = _extraterm_hllc_vel(um, up, Gm, Gp, Pm, Pp, sm, sp, ss);
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
#endif


#if 0
    if (ix < NXP1 && iy < NY)
    {
        // Process nslices of current chunk
        // iz = 0, 1, 2: left zghost slices
        // iz = nslices+3, nslices+4, nslices+5: right zghost slices
        for (uint_t iz = 3; iz < nslices+3; ++iz)
        {
            /* *
             * 1.) Get cell values
             * 2.) Reconstruct face values (in primitive variables)
             * 3.) Compute characteristic velocities
             * 4.) Compute 7 flux contributions
             * 5.) Compute right hand side for the advection equations
             * */

            ///////////////////////////////////////////////////////////////////
            // 1.) Load data
            ///////////////////////////////////////////////////////////////////
            Real rm3, rm2, rm1, rp1, rp2, rp3;
            _xfetch_data(texR, ghostL.r, ghostR.r, ix, iy, iz, global_iz, NXP1, NY, rm3, rm2, rm1, rp1, rp2, rp3);
            assert(rm3 > 0); assert(rm2 > 0); assert(rm1 > 0); assert(rp1 > 0); assert(rp2 > 0); assert(rp3 > 0);

            Real um3, um2, um1, up1, up2, up3;
            _xfetch_data(texU, ghostL.u, ghostR.u, ix, iy, iz, global_iz, NXP1, NY, um3, um2, um1, up1, up2, up3);

            Real vm3, vm2, vm1, vp1, vp2, vp3;
            _xfetch_data(texV, ghostL.v, ghostR.v, ix, iy, iz, global_iz, NXP1, NY, vm3, vm2, vm1, vp1, vp2, vp3);

            Real wm3, wm2, wm1, wp1, wp2, wp3;
            _xfetch_data(texW, ghostL.w, ghostR.w, ix, iy, iz, global_iz, NXP1, NY, wm3, wm2, wm1, wp1, wp2, wp3);

            Real em3, em2, em1, ep1, ep2, ep3;
            _xfetch_data(texE, ghostL.e, ghostR.e, ix, iy, iz, global_iz, NXP1, NY, em3, em2, em1, ep1, ep2, ep3);
            assert(em3 > 0); assert(em2 > 0); assert(em1 > 0); assert(ep1 > 0); assert(ep2 > 0); assert(ep3 > 0);

            Real Gm3, Gm2, Gm1, Gp1, Gp2, Gp3;
            _xfetch_data(texG, ghostL.G, ghostR.G, ix, iy, iz, global_iz, NXP1, NY, Gm3, Gm2, Gm1, Gp1, Gp2, Gp3);
            assert(Gm3 > 0); assert(Gm2 > 0); assert(Gm1 > 0); assert(Gp1 > 0); assert(Gp2 > 0); assert(Gp3 > 0);

            Real Pm3, Pm2, Pm1, Pp1, Pp2, Pp3;
            _xfetch_data(texP, ghostL.P, ghostR.P, ix, iy, iz, global_iz, NXP1, NY, Pm3, Pm2, Pm1, Pp1, Pp2, Pp3);
            assert(Pm3 >= 0); assert(Pm2 >= 0); assert(Pm1 >= 0); assert(Pp1 >= 0); assert(Pp2 >= 0); assert(Pp3 >= 0);

            ///////////////////////////////////////////////////////////////////
            // 2.) Reconstruction of primitive values, using WENO5/3
            ///////////////////////////////////////////////////////////////////
            // Reconstruct primitive value p at face f, using WENO5/3
            // rho
            const Real rp = _weno_pluss_clipped(rm2, rm1, rp1, rp2, rp3);
            const Real rm = _weno_minus_clipped(rm3, rm2, rm1, rp1, rp2);
            assert(!isnan(rp)); assert(!isnan(rm));
            // u (convert primitive variable u = (rho*u) / rho)
            um3 /= rm3; um2 /= rm2; um1 /= rm1; up1 /= rp1; up2 /= rp2; up3 /= rp3;
            const Real up = _weno_pluss_clipped(um2, um1, up1, up2, up3);
            const Real um = _weno_minus_clipped(um3, um2, um1, up1, up2);
            assert(!isnan(up)); assert(!isnan(um));
            // v (convert primitive variable v = (rho*v) / rho)
            vm3 /= rm3; vm2 /= rm2; vm1 /= rm1; vp1 /= rp1; vp2 /= rp2; vp3 /= rp3;
            const Real vp = _weno_pluss_clipped(vm2, vm1, vp1, vp2, vp3);
            const Real vm = _weno_minus_clipped(vm3, vm2, vm1, vp1, vp2);
            assert(!isnan(vp)); assert(!isnan(vm));
            // w (convert primitive variable w = (rho*w) / rho)
            wm3 /= rm3; wm2 /= rm2; wm1 /= rm1; wp1 /= rp1; wp2 /= rp2; wp3 /= rp3;
            const Real wp = _weno_pluss_clipped(wm2, wm1, wp1, wp2, wp3);
            const Real wm = _weno_minus_clipped(wm3, wm2, wm1, wp1, wp2);
            assert(!isnan(wp)); assert(!isnan(wm));
            // p (convert primitive variable p = (e - 0.5*rho*(u*u + v*v + w*w) - P) / G
            const Real pm3 = (em3 - 0.5f*rm3*(um3*um3 + vm3*vm3 + wm3*wm3) - Pm3) / Gm3;
            const Real pm2 = (em2 - 0.5f*rm2*(um2*um2 + vm2*vm2 + wm2*wm2) - Pm2) / Gm2;
            const Real pm1 = (em1 - 0.5f*rm1*(um1*um1 + vm1*vm1 + wm1*wm1) - Pm1) / Gm1;
            const Real pp1 = (ep1 - 0.5f*rp1*(up1*up1 + vp1*vp1 + wp1*wp1) - Pp1) / Gp1;
            const Real pp2 = (ep2 - 0.5f*rp2*(up2*up2 + vp2*vp2 + wp2*wp2) - Pp2) / Gp2;
            const Real pp3 = (ep3 - 0.5f*rp3*(up3*up3 + vp3*vp3 + wp3*wp3) - Pp3) / Gp3;
            const Real pp = _weno_pluss_clipped(pm2, pm1, pp1, pp2, pp3);
            const Real pm = _weno_minus_clipped(pm3, pm2, pm1, pp1, pp2);
            assert(!isnan(pp)); assert(!isnan(pm));
            // G
            const Real Gp = _weno_pluss_clipped(Gm2, Gm1, Gp1, Gp2, Gp3);
            const Real Gm = _weno_minus_clipped(Gm3, Gm2, Gm1, Gp1, Gp2);
            assert(!isnan(Gp)); assert(!isnan(Gm));
            // P
            const Real Pp = _weno_pluss_clipped(Pm2, Pm1, Pp1, Pp2, Pp3);
            const Real Pm = _weno_minus_clipped(Pm3, Pm2, Pm1, Pp1, Pp2);
            assert(!isnan(Pp)); assert(!isnan(Pm));

            ///////////////////////////////////////////////////////////////////
            // 3.) Einfeldt characteristic velocities
            ///////////////////////////////////////////////////////////////////
            Real sm, sp;
            _char_vel_einfeldt(rm, rp, um, up, pm, pp, Gm, Gp, Pm, Pp, sm, sp);
            const Real ss = _char_vel_star(rm, rp, um, up, pm, pp, sm, sp);
            assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

            ///////////////////////////////////////////////////////////////////
            // 4.) Compute HLLC fluxes
            ///////////////////////////////////////////////////////////////////
            const Real fr = _hllc_rho(rm, rp, um, up, sm, sp, ss);
            const Real fu = _hllc_pvel(rm, rp, um, up, pm, pp, sm, sp, ss);
            const Real fv = _hllc_vel(rm, rp, vm, vp, um, up, sm, sp, ss);
            const Real fw = _hllc_vel(rm, rp, wm, wp, um, up, sm, sp, ss);
            const Real fe = _hllc_e(rm, rp, um, up, vm, vp, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss);
            const Real fG = _hllc_rho(Gm, Gp, um, up, sm, sp, ss);
            const Real fP = _hllc_rho(Pm, Pp, um, up, sm, sp, ss);
            assert(!isnan(fr)); assert(!isnan(fu)); assert(!isnan(fv)); assert(!isnan(fw)); assert(!isnan(fe)); assert(!isnan(fG)); assert(!isnan(fP));

            const uint_t idx = ID3(ix, iy, iz-3, NXP1, NY);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            ///////////////////////////////////////////////////////////////////
            // 5.) RHS for advection equations
            ///////////////////////////////////////////////////////////////////
            xtra_vel[idx] = _extraterm_hllc_vel(um, up, Gm, Gp, Pm, Pp, sm, sp, ss);
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
#endif
}


__global__
void _yextraterm_hllc(const uint_t nslices,
        const Real * const Gm, const Real * const Gp,
        const Real * const Pm, const Real * const Pp,
        const Real * const vel,
        Real * const sumG, Real * const sumP, Real * const divU)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx  = ID3(ix, iy,   iz, NX, NY);
            const uint_t idxm = ID3(ix, iy,   iz, NX, NYP1);
            const uint_t idxp = ID3(ix, iy+1, iz, NX, NYP1);
            sumG[idx] += Gp[idxm] + Gm[idxp];
            sumP[idx] += Pp[idxm] + Pm[idxp];
            divU[idx] += vel[idxp] - vel[idxm];
        }
    }
}


__global__
void _yflux(const uint_t nslices, const uint_t global_iz,
        devPtrSet ghostL, devPtrSet ghostR, devPtrSet flux,
        Real * const xtra_vel,
        Real * const xtra_Gm, Real * const xtra_Gp,
        Real * const xtra_Pm, Real * const xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NYP1 = NY + 1
     * 2.) NX = NodeBlock::sizeX
     * 3.) NY = NodeBlock::sizeY
     * 4.) nslices = number of slices for currently processed chunk
     * 5.) global_iz is the iz-coordinate in index space of the NodeBlock for
     *     the first slice of the currently processed chunk.  It is needed if
     *     all of the x-/yghosts are uploaded to the GPU prior to processing
     *     the chunks sequentially.  Currently global_iz = 0, since x-/yghosts
     *     are uploaded per chunk.
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    /* *
     * The following code requires 1D thread blocks with dimension
     * dim3(_NTHREADS_, 1, 1)
     * on a 2D grid with dimension
     * dim3((NX + _NTHREADS_ - 1) / _NTHREADS_, NYP1, 1)
     *
     * The load of ghosts is organized into a switch block with 6 cases.
     *
     * NOTE: To minimize the switch cases to 6 (and simplify code) the
     * following requires that NY >= 5
     * */
    assert(NYP1 > 5);

#if 1
    if (ix < NX && iy < NYP1)
    {
        Stencil r, u, v, w, e, G, P;
        Stencil p; // for reconstruction
        for (uint_t iz = 3; iz < nslices+3; ++iz) // first and last 3 slices are zghosts
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load ghosts from GMEM or tex3D into stencil (do this 7x, for each
             *     quantity)
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // 1.)
#if 1
            // more conditionals
            _read_stencil_Y(r, texR, ghostL.r, ghostR.r, ix, iy, iz, global_iz);
            _read_stencil_Y(u, texU, ghostL.u, ghostR.u, ix, iy, iz, global_iz);
            _read_stencil_Y(v, texV, ghostL.v, ghostR.v, ix, iy, iz, global_iz);
            _read_stencil_Y(w, texW, ghostL.w, ghostR.w, ix, iy, iz, global_iz);
            _read_stencil_Y(e, texE, ghostL.e, ghostR.e, ix, iy, iz, global_iz);
            _read_stencil_Y(G, texG, ghostL.G, ghostR.G, ix, iy, iz, global_iz);
            _read_stencil_Y(P, texP, ghostL.P, ghostR.P, ix, iy, iz, global_iz);
#else
            // less conditionals, additional tex3D fetches
            _load_stencil_tex3D_Y(r, texR, ix, iy, iz);
            _load_stencil_tex3D_Y(u, texU, ix, iy, iz);
            _load_stencil_tex3D_Y(v, texV, ix, iy, iz);
            _load_stencil_tex3D_Y(w, texW, ix, iy, iz);
            _load_stencil_tex3D_Y(e, texE, ix, iy, iz);
            _load_stencil_tex3D_Y(G, texG, ix, iy, iz);
            _load_stencil_tex3D_Y(P, texP, ix, iy, iz);
            switch (iy)
            {
                case 0:
                    _load_3ghosts_Y(r.im3, r.im2, r.im1, ghostL.r, ix, iz-3+global_iz);
                    _load_3ghosts_Y(u.im3, u.im2, u.im1, ghostL.u, ix, iz-3+global_iz);
                    _load_3ghosts_Y(v.im3, v.im2, v.im1, ghostL.v, ix, iz-3+global_iz);
                    _load_3ghosts_Y(w.im3, w.im2, w.im1, ghostL.w, ix, iz-3+global_iz);
                    _load_3ghosts_Y(e.im3, e.im2, e.im1, ghostL.e, ix, iz-3+global_iz);
                    _load_3ghosts_Y(G.im3, G.im2, G.im1, ghostL.G, ix, iz-3+global_iz);
                    _load_3ghosts_Y(P.im3, P.im2, P.im1, ghostL.P, ix, iz-3+global_iz);
                    break;
                case 1:
                    _load_2ghosts_Y(r.im3, r.im2, 1, 2, ghostL.r, ix, iz-3+global_iz);
                    _load_2ghosts_Y(u.im3, u.im2, 1, 2, ghostL.u, ix, iz-3+global_iz);
                    _load_2ghosts_Y(v.im3, v.im2, 1, 2, ghostL.v, ix, iz-3+global_iz);
                    _load_2ghosts_Y(w.im3, w.im2, 1, 2, ghostL.w, ix, iz-3+global_iz);
                    _load_2ghosts_Y(e.im3, e.im2, 1, 2, ghostL.e, ix, iz-3+global_iz);
                    _load_2ghosts_Y(G.im3, G.im2, 1, 2, ghostL.G, ix, iz-3+global_iz);
                    _load_2ghosts_Y(P.im3, P.im2, 1, 2, ghostL.P, ix, iz-3+global_iz);
                    break;
                case 2:
                    _load_1ghost_Y(r.im3, 2, ghostL.r, ix, iz-3+global_iz);
                    _load_1ghost_Y(u.im3, 2, ghostL.u, ix, iz-3+global_iz);
                    _load_1ghost_Y(v.im3, 2, ghostL.v, ix, iz-3+global_iz);
                    _load_1ghost_Y(w.im3, 2, ghostL.w, ix, iz-3+global_iz);
                    _load_1ghost_Y(e.im3, 2, ghostL.e, ix, iz-3+global_iz);
                    _load_1ghost_Y(G.im3, 2, ghostL.G, ix, iz-3+global_iz);
                    _load_1ghost_Y(P.im3, 2, ghostL.P, ix, iz-3+global_iz);
                    break;
                case (NYP1-3):
                    _load_1ghost_Y(r.ip2, 0, ghostR.r, ix, iz-3+global_iz);
                    _load_1ghost_Y(u.ip2, 0, ghostR.u, ix, iz-3+global_iz);
                    _load_1ghost_Y(v.ip2, 0, ghostR.v, ix, iz-3+global_iz);
                    _load_1ghost_Y(w.ip2, 0, ghostR.w, ix, iz-3+global_iz);
                    _load_1ghost_Y(e.ip2, 0, ghostR.e, ix, iz-3+global_iz);
                    _load_1ghost_Y(G.ip2, 0, ghostR.G, ix, iz-3+global_iz);
                    _load_1ghost_Y(P.ip2, 0, ghostR.P, ix, iz-3+global_iz);
                    break;
                case (NYP1-2):
                    _load_2ghosts_Y(r.ip1, r.ip2, 0, 1, ghostR.r, ix, iz-3+global_iz);
                    _load_2ghosts_Y(u.ip1, u.ip2, 0, 1, ghostR.u, ix, iz-3+global_iz);
                    _load_2ghosts_Y(v.ip1, v.ip2, 0, 1, ghostR.v, ix, iz-3+global_iz);
                    _load_2ghosts_Y(w.ip1, w.ip2, 0, 1, ghostR.w, ix, iz-3+global_iz);
                    _load_2ghosts_Y(e.ip1, e.ip2, 0, 1, ghostR.e, ix, iz-3+global_iz);
                    _load_2ghosts_Y(G.ip1, G.ip2, 0, 1, ghostR.G, ix, iz-3+global_iz);
                    _load_2ghosts_Y(P.ip1, P.ip2, 0, 1, ghostR.P, ix, iz-3+global_iz);
                    break;
                case (NYP1-1):
                    _load_3ghosts_Y(r.i, r.ip1, r.ip2, ghostR.r, ix, iz-3+global_iz);
                    _load_3ghosts_Y(u.i, u.ip1, u.ip2, ghostR.u, ix, iz-3+global_iz);
                    _load_3ghosts_Y(v.i, v.ip1, v.ip2, ghostR.v, ix, iz-3+global_iz);
                    _load_3ghosts_Y(w.i, w.ip1, w.ip2, ghostR.w, ix, iz-3+global_iz);
                    _load_3ghosts_Y(e.i, e.ip1, e.ip2, ghostR.e, ix, iz-3+global_iz);
                    _load_3ghosts_Y(G.i, G.ip1, G.ip2, ghostR.G, ix, iz-3+global_iz);
                    _load_3ghosts_Y(P.i, P.ip1, P.ip2, ghostR.P, ix, iz-3+global_iz);
                    break;
            } // end switch
#endif
            assert(r > 0);
            assert(e > 0);
            assert(G > 0);
            assert(P >= 0);

            // 2.)
            // rho
            const Real rp = _weno_pluss_clipped(r.im2, r.im1, r.i, r.ip1, r.ip2);
            const Real rm = _weno_minus_clipped(r.im3, r.im2, r.im1, r.i, r.ip1);
            assert(!isnan(rp)); assert(!isnan(rm));
            // u (convert primitive variable u = (rho*u) / rho)
            u.im3 /= r.im3;
            u.im2 /= r.im2;
            u.im1 /= r.im1;
            u.i   /= r.i;
            u.ip1 /= r.ip1;
            u.ip2 /= r.ip2;
            const Real up = _weno_pluss_clipped(u.im2, u.im1, u.i, u.ip1, u.ip2);
            const Real um = _weno_minus_clipped(u.im3, u.im2, u.im1, u.i, u.ip1);
            assert(!isnan(up)); assert(!isnan(um));
            // v (convert primitive variable v = (rho*v) / rho)
            v.im3 /= r.im3;
            v.im2 /= r.im2;
            v.im1 /= r.im1;
            v.i   /= r.i;
            v.ip1 /= r.ip1;
            v.ip2 /= r.ip2;
            const Real vp = _weno_pluss_clipped(v.im2, v.im1, v.i, v.ip1, v.ip2);
            const Real vm = _weno_minus_clipped(v.im3, v.im2, v.im1, v.i, v.ip1);
            assert(!isnan(vp)); assert(!isnan(vm));
            // w (convert primitive variable w = (rho*w) / rho)
            w.im3 /= r.im3;
            w.im2 /= r.im2;
            w.im1 /= r.im1;
            w.i   /= r.i;
            w.ip1 /= r.ip1;
            w.ip2 /= r.ip2;
            const Real wp = _weno_pluss_clipped(w.im2, w.im1, w.i, w.ip1, w.ip2);
            const Real wm = _weno_minus_clipped(w.im3, w.im2, w.im1, w.i, w.ip1);
            assert(!isnan(wp)); assert(!isnan(wm));
            // p (convert primitive variable p = (e - 0.5*rho*(u*u + v*v + w*w) - P) / G
            p.im3 = (e.im3 - 0.5f*r.im3*(u.im3*u.im3 + v.im3*v.im3 + w.im3*w.im3) - P.im3) / G.im3;
            p.im2 = (e.im2 - 0.5f*r.im2*(u.im2*u.im2 + v.im2*v.im2 + w.im2*w.im2) - P.im2) / G.im2;
            p.im1 = (e.im1 - 0.5f*r.im1*(u.im1*u.im1 + v.im1*v.im1 + w.im1*w.im1) - P.im1) / G.im1;
            p.i   = (e.i   - 0.5f*r.i*(u.i*u.i       + v.i*v.i     + w.i*w.i)     - P.i)   / G.i;
            p.ip1 = (e.ip1 - 0.5f*r.ip1*(u.ip1*u.ip1 + v.ip1*v.ip1 + w.ip1*w.ip1) - P.ip1) / G.ip1;
            p.ip2 = (e.ip2 - 0.5f*r.ip2*(u.ip2*u.ip2 + v.ip2*v.ip2 + w.ip2*w.ip2) - P.ip2) / G.ip2;
            const Real pp = _weno_pluss_clipped(p.im2, p.im1, p.i, p.ip1, p.ip2);
            const Real pm = _weno_minus_clipped(p.im3, p.im2, p.im1, p.i, p.ip1);
            assert(!isnan(pp)); assert(!isnan(pm));
            // G
            const Real Gp = _weno_pluss_clipped(G.im2, G.im1, G.i, G.ip1, G.ip2);
            const Real Gm = _weno_minus_clipped(G.im3, G.im2, G.im1, G.i, G.ip1);
            assert(!isnan(Gp)); assert(!isnan(Gm));
            // P
            const Real Pp = _weno_pluss_clipped(P.im2, P.im1, P.i, P.ip1, P.ip2);
            const Real Pm = _weno_minus_clipped(P.im3, P.im2, P.im1, P.i, P.ip1);
            assert(!isnan(Pp)); assert(!isnan(Pm));

            // 3.)
            Real sm, sp;
            _char_vel_einfeldt(rm, rp, vm, vp, pm, pp, Gm, Gp, Pm, Pp, sm, sp);
            const Real ss = _char_vel_star(rm, rp, vm, vp, pm, pp, sm, sp);
            assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

            // 4.)
            const Real fr = _hllc_rho(rm, rp, vm, vp, sm, sp, ss);
            const Real fu = _hllc_vel(rm, rp, um, up, vm, vp, sm, sp, ss);
            const Real fv = _hllc_pvel(rm, rp, vm, vp, pm, pp, sm, sp, ss);
            const Real fw = _hllc_vel(rm, rp, wm, wp, vm, vp, sm, sp, ss);
            const Real fe = _hllc_e(rm, rp, vm, vp, um, up, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss);
            const Real fG = _hllc_rho(Gm, Gp, vm, vp, sm, sp, ss);
            const Real fP = _hllc_rho(Pm, Pp, vm, vp, sm, sp, ss);
            assert(!isnan(fr)); assert(!isnan(fu)); assert(!isnan(fv)); assert(!isnan(fw)); assert(!isnan(fe)); assert(!isnan(fG)); assert(!isnan(fP));

            const uint_t idx = ID3(ix, iy, iz-3, NX, NYP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = _extraterm_hllc_vel(vm, vp, Gm, Gp, Pm, Pp, sm, sp, ss);
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
#endif

#if 0
    if (ix < NX && iy < NYP1)
    {
        for (uint_t iz = 3; iz < nslices+3; ++iz)
        {
            /* *
             * 1.) Get cell values
             * 2.) Reconstruct face values (in primitive variables)
             * 3.) Compute characteristic velocities
             * 4.) Compute 7 flux contributions
             * 5.) Compute right hand side for the advection equations
             * */

            ///////////////////////////////////////////////////////////////////
            // 1.) Load data
            ///////////////////////////////////////////////////////////////////
            Real rm3, rm2, rm1, rp1, rp2, rp3;
            _yfetch_data(texR, ghostL.r, ghostR.r, ix, iy, iz, global_iz, NX, NYP1, rm3, rm2, rm1, rp1, rp2, rp3);
            assert(rm3 > 0); assert(rm2 > 0); assert(rm1 > 0); assert(rp1 > 0); assert(rp2 > 0); assert(rp3 > 0);

            Real um3, um2, um1, up1, up2, up3;
            _yfetch_data(texU, ghostL.u, ghostR.u, ix, iy, iz, global_iz, NX, NYP1, um3, um2, um1, up1, up2, up3);

            Real vm3, vm2, vm1, vp1, vp2, vp3;
            _yfetch_data(texV, ghostL.v, ghostR.v, ix, iy, iz, global_iz, NX, NYP1, vm3, vm2, vm1, vp1, vp2, vp3);

            Real wm3, wm2, wm1, wp1, wp2, wp3;
            _yfetch_data(texW, ghostL.w, ghostR.w, ix, iy, iz, global_iz, NX, NYP1, wm3, wm2, wm1, wp1, wp2, wp3);

            Real em3, em2, em1, ep1, ep2, ep3;
            _yfetch_data(texE, ghostL.e, ghostR.e, ix, iy, iz, global_iz, NX, NYP1, em3, em2, em1, ep1, ep2, ep3);
            assert(em3 > 0); assert(em2 > 0); assert(em1 > 0); assert(ep1 > 0); assert(ep2 > 0); assert(ep3 > 0);

            Real Gm3, Gm2, Gm1, Gp1, Gp2, Gp3;
            _yfetch_data(texG, ghostL.G, ghostR.G, ix, iy, iz, global_iz, NX, NYP1, Gm3, Gm2, Gm1, Gp1, Gp2, Gp3);
            assert(Gm3 > 0); assert(Gm2 > 0); assert(Gm1 > 0); assert(Gp1 > 0); assert(Gp2 > 0); assert(Gp3 > 0);

            Real Pm3, Pm2, Pm1, Pp1, Pp2, Pp3;
            _yfetch_data(texP, ghostL.P, ghostR.P, ix, iy, iz, global_iz, NX, NYP1, Pm3, Pm2, Pm1, Pp1, Pp2, Pp3);
            assert(Pm3 >= 0); assert(Pm2 >= 0); assert(Pm1 >= 0); assert(Pp1 >= 0); assert(Pp2 >= 0); assert(Pp3 >= 0);

            ///////////////////////////////////////////////////////////////////
            // 2.) Reconstruction of primitive values, using WENO5/3
            ///////////////////////////////////////////////////////////////////
            // rho
            const Real rp = _weno_pluss_clipped(rm2, rm1, rp1, rp2, rp3);
            const Real rm = _weno_minus_clipped(rm3, rm2, rm1, rp1, rp2);
            assert(!isnan(rp)); assert(!isnan(rm));
            // u (convert primitive variable u = (rho*u) / rho)
            um3 /= rm3; um2 /= rm2; um1 /= rm1; up1 /= rp1; up2 /= rp2; up3 /= rp3;
            const Real up = _weno_pluss_clipped(um2, um1, up1, up2, up3);
            const Real um = _weno_minus_clipped(um3, um2, um1, up1, up2);
            assert(!isnan(up)); assert(!isnan(um));
            // v (convert primitive variable v = (rho*v) / rho)
            vm3 /= rm3; vm2 /= rm2; vm1 /= rm1; vp1 /= rp1; vp2 /= rp2; vp3 /= rp3;
            const Real vp = _weno_pluss_clipped(vm2, vm1, vp1, vp2, vp3);
            const Real vm = _weno_minus_clipped(vm3, vm2, vm1, vp1, vp2);
            assert(!isnan(vp)); assert(!isnan(vm));
            // w (convert primitive variable w = (rho*w) / rho)
            wm3 /= rm3; wm2 /= rm2; wm1 /= rm1; wp1 /= rp1; wp2 /= rp2; wp3 /= rp3;
            const Real wp = _weno_pluss_clipped(wm2, wm1, wp1, wp2, wp3);
            const Real wm = _weno_minus_clipped(wm3, wm2, wm1, wp1, wp2);
            assert(!isnan(wp)); assert(!isnan(wm));
            // p (convert primitive variable p = (e - 0.5*rho*(u*u + v*v + w*w) - P) / G
            const Real pm3 = (em3 - 0.5f*rm3*(um3*um3 + vm3*vm3 + wm3*wm3) - Pm3) / Gm3;
            const Real pm2 = (em2 - 0.5f*rm2*(um2*um2 + vm2*vm2 + wm2*wm2) - Pm2) / Gm2;
            const Real pm1 = (em1 - 0.5f*rm1*(um1*um1 + vm1*vm1 + wm1*wm1) - Pm1) / Gm1;
            const Real pp1 = (ep1 - 0.5f*rp1*(up1*up1 + vp1*vp1 + wp1*wp1) - Pp1) / Gp1;
            const Real pp2 = (ep2 - 0.5f*rp2*(up2*up2 + vp2*vp2 + wp2*wp2) - Pp2) / Gp2;
            const Real pp3 = (ep3 - 0.5f*rp3*(up3*up3 + vp3*vp3 + wp3*wp3) - Pp3) / Gp3;
            const Real pp = _weno_pluss_clipped(pm2, pm1, pp1, pp2, pp3);
            const Real pm = _weno_minus_clipped(pm3, pm2, pm1, pp1, pp2);
            assert(!isnan(pp)); assert(!isnan(pm));
            // G
            const Real Gp = _weno_pluss_clipped(Gm2, Gm1, Gp1, Gp2, Gp3);
            const Real Gm = _weno_minus_clipped(Gm3, Gm2, Gm1, Gp1, Gp2);
            assert(!isnan(Gp)); assert(!isnan(Gm));
            // P
            const Real Pp = _weno_pluss_clipped(Pm2, Pm1, Pp1, Pp2, Pp3);
            const Real Pm = _weno_minus_clipped(Pm3, Pm2, Pm1, Pp1, Pp2);
            assert(!isnan(Pp)); assert(!isnan(Pm));

            ///////////////////////////////////////////////////////////////////
            // 3.) Einfeldt characteristic velocities
            ///////////////////////////////////////////////////////////////////
            Real sm, sp;
            _char_vel_einfeldt(rm, rp, vm, vp, pm, pp, Gm, Gp, Pm, Pp, sm, sp);
            const Real ss = _char_vel_star(rm, rp, vm, vp, pm, pp, sm, sp);
            assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

            ///////////////////////////////////////////////////////////////////
            // 4.) Compute HLLC fluxes
            ///////////////////////////////////////////////////////////////////
            const Real fr = _hllc_rho(rm, rp, vm, vp, sm, sp, ss);
            const Real fu = _hllc_vel(rm, rp, um, up, vm, vp, sm, sp, ss);
            const Real fv = _hllc_pvel(rm, rp, vm, vp, pm, pp, sm, sp, ss);
            const Real fw = _hllc_vel(rm, rp, wm, wp, vm, vp, sm, sp, ss);
            const Real fe = _hllc_e(rm, rp, vm, vp, um, up, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss);
            const Real fG = _hllc_rho(Gm, Gp, vm, vp, sm, sp, ss);
            const Real fP = _hllc_rho(Pm, Pp, vm, vp, sm, sp, ss);

            const uint_t idx = ID3(ix, iy, iz-3, NX, NYP1);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            ///////////////////////////////////////////////////////////////////
            // 5.)
            ///////////////////////////////////////////////////////////////////
            xtra_vel[idx] = _extraterm_hllc_vel(vm, vp, Gm, Gp, Pm, Pp, sm, sp, ss);
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
#endif
}


__global__
void _zextraterm_hllc(const uint_t nslices,
        const Real * const Gm, const Real * const Gp,
        const Real * const Pm, const Real * const Pp,
        const Real * const vel,
        Real * const sumG, Real * const sumP, Real * const divU)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx  = ID3(ix, iy, iz,   NX, NY);
            const uint_t idxm = ID3(ix, iy, iz,   NX, NY);
            const uint_t idxp = ID3(ix, iy, iz+1, NX, NY);
            sumG[idx] += Gp[idxm] + Gm[idxp];
            sumP[idx] += Pp[idxm] + Pm[idxp];
            divU[idx] += vel[idxp] - vel[idxm];
        }
    }
}


__global__
void _zflux(const uint_t nslices, devPtrSet flux,
        Real * const xtra_vel,
        Real * const xtra_Gm, Real * const xtra_Gp,
        Real * const xtra_Pm, Real * const xtra_Pp)
{
    /* *
     * Notes:
     * ======
     * 1.) NX = NodeBlock::sizeX
     * 2.) NY = NodeBlock::sizeY
     * 3.) NZ = NodeBlock::sizeZ
     * 4.) nslices = number of slices for currently processed chunk
     * */
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    /* *
     * The following code requires 1D thread blocks with dimension
     * dim3(_NTHREADS_, 1, 1)
     * on a 2D grid with dimension
     * dim3((NX + _NTHREADS_ - 1) / _NTHREADS_, NY, 1)
     *
     * The following requires that NZ > 0
     * */
    assert(NodeBlock::sizeZ > 0);

#if 1
    if (ix < NX && iy < NY)
    {
        Stencil r, u, v, w, e, G, P;
        Stencil p; // for reconstruction
        for (uint_t iz = 3; iz < (nslices+1)+3; ++iz) // first and last 3 slices are zghosts, need to compute nslices+1 fluxes in z-direction
        {
            /* *
             * The general task order is (for each chunk slice along NZ):
             * 1.) Load tex3D into stencil (do this 7x, for each quantity)
             * 2.) Reconstruct primitive values using WENO5/WENO3
             * 3.) Compute characteristic velocities
             * 4.) Compute fluxes
             * 5.) Compute RHS for advection of G and P
             * */

            // 1.)
            // that was easy!
            _read_stencil_Z(r, texR, ix, iy, iz);
            _read_stencil_Z(u, texU, ix, iy, iz);
            _read_stencil_Z(v, texV, ix, iy, iz);
            _read_stencil_Z(w, texW, ix, iy, iz);
            _read_stencil_Z(e, texE, ix, iy, iz);
            _read_stencil_Z(G, texG, ix, iy, iz);
            _read_stencil_Z(P, texP, ix, iy, iz);

            assert(r > 0);
            assert(e > 0);
            assert(G > 0);
            assert(P >= 0);

            // 2.)
            // rho
            const Real rp = _weno_pluss_clipped(r.im2, r.im1, r.i, r.ip1, r.ip2);
            const Real rm = _weno_minus_clipped(r.im3, r.im2, r.im1, r.i, r.ip1);
            assert(!isnan(rp)); assert(!isnan(rm));
            // u (convert primitive variable u = (rho*u) / rho)
            u.im3 /= r.im3;
            u.im2 /= r.im2;
            u.im1 /= r.im1;
            u.i   /= r.i;
            u.ip1 /= r.ip1;
            u.ip2 /= r.ip2;
            const Real up = _weno_pluss_clipped(u.im2, u.im1, u.i, u.ip1, u.ip2);
            const Real um = _weno_minus_clipped(u.im3, u.im2, u.im1, u.i, u.ip1);
            assert(!isnan(up)); assert(!isnan(um));
            // v (convert primitive variable v = (rho*v) / rho)
            v.im3 /= r.im3;
            v.im2 /= r.im2;
            v.im1 /= r.im1;
            v.i   /= r.i;
            v.ip1 /= r.ip1;
            v.ip2 /= r.ip2;
            const Real vp = _weno_pluss_clipped(v.im2, v.im1, v.i, v.ip1, v.ip2);
            const Real vm = _weno_minus_clipped(v.im3, v.im2, v.im1, v.i, v.ip1);
            assert(!isnan(vp)); assert(!isnan(vm));
            // w (convert primitive variable w = (rho*w) / rho)
            w.im3 /= r.im3;
            w.im2 /= r.im2;
            w.im1 /= r.im1;
            w.i   /= r.i;
            w.ip1 /= r.ip1;
            w.ip2 /= r.ip2;
            const Real wp = _weno_pluss_clipped(w.im2, w.im1, w.i, w.ip1, w.ip2);
            const Real wm = _weno_minus_clipped(w.im3, w.im2, w.im1, w.i, w.ip1);
            assert(!isnan(wp)); assert(!isnan(wm));
            // p (convert primitive variable p = (e - 0.5*rho*(u*u + v*v + w*w) - P) / G
            p.im3 = (e.im3 - 0.5f*r.im3*(u.im3*u.im3 + v.im3*v.im3 + w.im3*w.im3) - P.im3) / G.im3;
            p.im2 = (e.im2 - 0.5f*r.im2*(u.im2*u.im2 + v.im2*v.im2 + w.im2*w.im2) - P.im2) / G.im2;
            p.im1 = (e.im1 - 0.5f*r.im1*(u.im1*u.im1 + v.im1*v.im1 + w.im1*w.im1) - P.im1) / G.im1;
            p.i   = (e.i   - 0.5f*r.i*(u.i*u.i       + v.i*v.i     + w.i*w.i)     - P.i)   / G.i;
            p.ip1 = (e.ip1 - 0.5f*r.ip1*(u.ip1*u.ip1 + v.ip1*v.ip1 + w.ip1*w.ip1) - P.ip1) / G.ip1;
            p.ip2 = (e.ip2 - 0.5f*r.ip2*(u.ip2*u.ip2 + v.ip2*v.ip2 + w.ip2*w.ip2) - P.ip2) / G.ip2;
            const Real pp = _weno_pluss_clipped(p.im2, p.im1, p.i, p.ip1, p.ip2);
            const Real pm = _weno_minus_clipped(p.im3, p.im2, p.im1, p.i, p.ip1);
            assert(!isnan(pp)); assert(!isnan(pm));
            // G
            const Real Gp = _weno_pluss_clipped(G.im2, G.im1, G.i, G.ip1, G.ip2);
            const Real Gm = _weno_minus_clipped(G.im3, G.im2, G.im1, G.i, G.ip1);
            assert(!isnan(Gp)); assert(!isnan(Gm));
            // P
            const Real Pp = _weno_pluss_clipped(P.im2, P.im1, P.i, P.ip1, P.ip2);
            const Real Pm = _weno_minus_clipped(P.im3, P.im2, P.im1, P.i, P.ip1);
            assert(!isnan(Pp)); assert(!isnan(Pm));

            // 3.)
            Real sm, sp;
            _char_vel_einfeldt(rm, rp, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp);
            const Real ss = _char_vel_star(rm, rp, wm, wp, pm, pp, sm, sp);
            assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

            // 4.)
            const Real fr = _hllc_rho(rm, rp, wm, wp, sm, sp, ss);
            const Real fu = _hllc_vel(rm, rp, um, up, wm, wp, sm, sp, ss);
            const Real fv = _hllc_vel(rm, rp, vm, vp, wm, wp, sm, sp, ss);
            const Real fw = _hllc_pvel(rm, rp, wm, wp, pm, pp, sm, sp, ss);
            const Real fe = _hllc_e(rm, rp, wm, wp, um, up, vm, vp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss);
            const Real fG = _hllc_rho(Gm, Gp, wm, wp, sm, sp, ss);
            const Real fP = _hllc_rho(Pm, Pp, wm, wp, sm, sp, ss);

            const uint_t idx = ID3(ix, iy, iz-3, NX, NY);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            // 5.)
            xtra_vel[idx] = _extraterm_hllc_vel(wm, wp, Gm, Gp, Pm, Pp, sm, sp, ss);
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
#endif


#if 0
    if (ix < NX && iy < NY)
    {
        // need to compute nslices+1 fluxes in z-direction
        for (uint_t iz = 3; iz < (nslices+1)+3; ++iz)
        {
            /* *
             * 1.) Get cell values
             * 2.) Reconstruct face values (in primitive variables)
             * 3.) Compute characteristic velocities
             * 4.) Compute 7 flux contributions
             * 5.) Compute right hand side for the advection equations
             * */

            ///////////////////////////////////////////////////////////////////
            // 1.) Load data
            ///////////////////////////////////////////////////////////////////
            Real rm3, rm2, rm1, rp1, rp2, rp3;
            _zfetch_data(texR, ix, iy, iz, rm3, rm2, rm1, rp1, rp2, rp3);
            assert(rm3 > 0); assert(rm2 > 0); assert(rm1 > 0); assert(rp1 > 0); assert(rp2 > 0); assert(rp3 > 0);

            Real um3, um2, um1, up1, up2, up3;
            _zfetch_data(texU, ix, iy, iz, um3, um2, um1, up1, up2, up3);

            Real vm3, vm2, vm1, vp1, vp2, vp3;
            _zfetch_data(texV, ix, iy, iz, vm3, vm2, vm1, vp1, vp2, vp3);

            Real wm3, wm2, wm1, wp1, wp2, wp3;
            _zfetch_data(texW, ix, iy, iz, wm3, wm2, wm1, wp1, wp2, wp3);

            Real em3, em2, em1, ep1, ep2, ep3;
            _zfetch_data(texE, ix, iy, iz, em3, em2, em1, ep1, ep2, ep3);
            assert(em3 > 0); assert(em2 > 0); assert(em1 > 0); assert(ep1 > 0); assert(ep2 > 0); assert(ep3 > 0);

            Real Gm3, Gm2, Gm1, Gp1, Gp2, Gp3;
            _zfetch_data(texG, ix, iy, iz, Gm3, Gm2, Gm1, Gp1, Gp2, Gp3);
            assert(Gm3 > 0); assert(Gm2 > 0); assert(Gm1 > 0); assert(Gp1 > 0); assert(Gp2 > 0); assert(Gp3 > 0);

            Real Pm3, Pm2, Pm1, Pp1, Pp2, Pp3;
            _zfetch_data(texP, ix, iy, iz, Pm3, Pm2, Pm1, Pp1, Pp2, Pp3);
            assert(Pm3 >= 0); assert(Pm2 >= 0); assert(Pm1 >= 0); assert(Pp1 >= 0); assert(Pp2 >= 0); assert(Pp3 >= 0);

            ///////////////////////////////////////////////////////////////////
            // 2.) Reconstruction of primitive values, using WENO5/3
            ///////////////////////////////////////////////////////////////////
            // rho
            const Real rp = _weno_pluss_clipped(rm2, rm1, rp1, rp2, rp3);
            const Real rm = _weno_minus_clipped(rm3, rm2, rm1, rp1, rp2);
            assert(!isnan(rp)); assert(!isnan(rm));
            // u (convert primitive variable u = (rho*u) / rho)
            um3 /= rm3; um2 /= rm2; um1 /= rm1; up1 /= rp1; up2 /= rp2; up3 /= rp3;
            const Real up = _weno_pluss_clipped(um2, um1, up1, up2, up3);
            const Real um = _weno_minus_clipped(um3, um2, um1, up1, up2);
            assert(!isnan(up)); assert(!isnan(um));
            // v (convert primitive variable v = (rho*v) / rho)
            vm3 /= rm3; vm2 /= rm2; vm1 /= rm1; vp1 /= rp1; vp2 /= rp2; vp3 /= rp3;
            const Real vp = _weno_pluss_clipped(vm2, vm1, vp1, vp2, vp3);
            const Real vm = _weno_minus_clipped(vm3, vm2, vm1, vp1, vp2);
            assert(!isnan(vp)); assert(!isnan(vm));
            // w (convert primitive variable w = (rho*w) / rho)
            wm3 /= rm3; wm2 /= rm2; wm1 /= rm1; wp1 /= rp1; wp2 /= rp2; wp3 /= rp3;
            const Real wp = _weno_pluss_clipped(wm2, wm1, wp1, wp2, wp3);
            const Real wm = _weno_minus_clipped(wm3, wm2, wm1, wp1, wp2);
            assert(!isnan(wp)); assert(!isnan(wm));
            // p (convert primitive variable p = (e - 0.5*rho*(u*u + v*v + w*w) - P) / G
            const Real pm3 = (em3 - 0.5f*rm3*(um3*um3 + vm3*vm3 + wm3*wm3) - Pm3) / Gm3;
            const Real pm2 = (em2 - 0.5f*rm2*(um2*um2 + vm2*vm2 + wm2*wm2) - Pm2) / Gm2;
            const Real pm1 = (em1 - 0.5f*rm1*(um1*um1 + vm1*vm1 + wm1*wm1) - Pm1) / Gm1;
            const Real pp1 = (ep1 - 0.5f*rp1*(up1*up1 + vp1*vp1 + wp1*wp1) - Pp1) / Gp1;
            const Real pp2 = (ep2 - 0.5f*rp2*(up2*up2 + vp2*vp2 + wp2*wp2) - Pp2) / Gp2;
            const Real pp3 = (ep3 - 0.5f*rp3*(up3*up3 + vp3*vp3 + wp3*wp3) - Pp3) / Gp3;
            const Real pp = _weno_pluss_clipped(pm2, pm1, pp1, pp2, pp3);
            const Real pm = _weno_minus_clipped(pm3, pm2, pm1, pp1, pp2);
            assert(!isnan(pp)); assert(!isnan(pm));
            // G
            const Real Gp = _weno_pluss_clipped(Gm2, Gm1, Gp1, Gp2, Gp3);
            const Real Gm = _weno_minus_clipped(Gm3, Gm2, Gm1, Gp1, Gp2);
            assert(!isnan(Gp)); assert(!isnan(Gm));
            // P
            const Real Pp = _weno_pluss_clipped(Pm2, Pm1, Pp1, Pp2, Pp3);
            const Real Pm = _weno_minus_clipped(Pm3, Pm2, Pm1, Pp1, Pp2);
            assert(!isnan(Pp)); assert(!isnan(Pm));

            ///////////////////////////////////////////////////////////////////
            // 3.) Einfeldt characteristic velocities
            ///////////////////////////////////////////////////////////////////
            Real sm, sp;
            _char_vel_einfeldt(rm, rp, wm, wp, pm, pp, Gm, Gp, Pm, Pp, sm, sp);
            const Real ss = _char_vel_star(rm, rp, wm, wp, pm, pp, sm, sp);
            assert(!isnan(sm)); assert(!isnan(sp)); assert(!isnan(ss));

            ///////////////////////////////////////////////////////////////////
            // 4.) Compute HLLC fluxes
            ///////////////////////////////////////////////////////////////////
            const Real fr = _hllc_rho(rm, rp, wm, wp, sm, sp, ss);
            const Real fu = _hllc_vel(rm, rp, um, up, wm, wp, sm, sp, ss);
            const Real fv = _hllc_vel(rm, rp, vm, vp, wm, wp, sm, sp, ss);
            const Real fw = _hllc_pvel(rm, rp, wm, wp, pm, pp, sm, sp, ss);
            const Real fe = _hllc_e(rm, rp, wm, wp, um, up, vm, vp, pm, pp, Gm, Gp, Pm, Pp, sm, sp, ss);
            const Real fG = _hllc_rho(Gm, Gp, wm, wp, sm, sp, ss);
            const Real fP = _hllc_rho(Pm, Pp, wm, wp, sm, sp, ss);

            const uint_t idx = ID3(ix, iy, iz-3, NX, NY);
            flux.r[idx] = fr;
            flux.u[idx] = fu;
            flux.v[idx] = fv;
            flux.w[idx] = fw;
            flux.e[idx] = fe;
            flux.G[idx] = fG;
            flux.P[idx] = fP;

            ///////////////////////////////////////////////////////////////////
            // 5.)
            ///////////////////////////////////////////////////////////////////
            xtra_vel[idx] = _extraterm_hllc_vel(wm, wp, Gm, Gp, Pm, Pp, sm, sp, ss);
            xtra_Gm[idx]  = Gm;
            xtra_Gp[idx]  = Gp;
            xtra_Pm[idx]  = Pm;
            xtra_Pp[idx]  = Pp;
        }
    }
#endif
}


__global__
void _divergence(const uint_t nslices,
        const devPtrSet xflux, const devPtrSet yflux, const devPtrSet zflux,
        devPtrSet rhs, const Real a, const Real dtinvh, const devPtrSet tmp,
        const Real * const sumG, const Real * const sumP, const Real * const divU)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        Real fxp, fxm, fyp, fym, fzp, fzm;
        const Real factor6 = 1.0f / 6.0f;

        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx = ID3(ix, iy, iz, NX, NY);

            _fetch_flux(ix, iy, iz, xflux.r, yflux.r, zflux.r, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_r = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm);
            rhs.r[idx] = a*tmp.r[idx] - rhs_r;

            _fetch_flux(ix, iy, iz, xflux.u, yflux.u, zflux.u, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_u = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm);
            rhs.u[idx] = a*tmp.u[idx] - rhs_u;

            _fetch_flux(ix, iy, iz, xflux.v, yflux.v, zflux.v, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_v = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm);
            rhs.v[idx] = a*tmp.v[idx] - rhs_v;

            _fetch_flux(ix, iy, iz, xflux.w, yflux.w, zflux.w, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_w = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm);
            rhs.w[idx] = a*tmp.w[idx] - rhs_w;

            _fetch_flux(ix, iy, iz, xflux.e, yflux.e, zflux.e, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_e = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm);
            rhs.e[idx] = a*tmp.e[idx] - rhs_e;

            _fetch_flux(ix, iy, iz, xflux.G, yflux.G, zflux.G, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_G = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm   - divU[idx] * sumG[idx] * factor6);
            rhs.G[idx] = a*tmp.G[idx] - rhs_G;

            _fetch_flux(ix, iy, iz, xflux.P, yflux.P, zflux.P, fxp, fxm, fyp, fym, fzp, fzm);
            const Real rhs_P = dtinvh*(fxp - fxm + fyp - fym + fzp - fzm   - divU[idx] * sumP[idx] * factor6);
            rhs.P[idx] = a*tmp.P[idx] - rhs_P;
        }
    }
}


__global__
void _update(const uint_t nslices, const Real b, devPtrSet tmp, const devPtrSet rhs)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < NX && iy < NY)
    {
        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const uint_t idx = ID3(ix, iy, iz, NX, NY);

            const Real r = tex3D(texR, ix, iy, iz+3);
            const Real u = tex3D(texU, ix, iy, iz+3);
            const Real v = tex3D(texV, ix, iy, iz+3);
            const Real w = tex3D(texW, ix, iy, iz+3);
            const Real e = tex3D(texE, ix, iy, iz+3);
            const Real G = tex3D(texG, ix, iy, iz+3);
            const Real P = tex3D(texP, ix, iy, iz+3);

            // this overwrites the rhs from the previous stage, stored in tmp,
            // with the updated solution.
            tmp.r[idx] = b*rhs.r[idx] + r;
            tmp.u[idx] = b*rhs.u[idx] + u;
            tmp.v[idx] = b*rhs.v[idx] + v;
            tmp.w[idx] = b*rhs.w[idx] + w;
            tmp.e[idx] = b*rhs.e[idx] + e;
            tmp.G[idx] = b*rhs.G[idx] + G;
            tmp.P[idx] = b*rhs.P[idx] + P;
            assert(tmp.r[idx] > 0);
            assert(tmp.e[idx] > 0);
            assert(tmp.G[idx] > 0);
            assert(tmp.P[idx] >= 0);
            /* if (tmp.P[idx] < 0) */
            /*     printf("(%d, %d, %d):\trhs.P = %f, tmp.P = %f, P = %f\n", ix, iy, iz, rhs.P[idx], tmp.P[idx], P); */
        }
    }
}


__global__
void _maxSOS(const uint_t nslices, int* g_maxSOS)
{
    const uint_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    const uint_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    const uint_t loc_idx = blockDim.x * threadIdx.y + threadIdx.x;
    __shared__ Real block_sos[_NTHREADS_];
    block_sos[loc_idx] = 0.0f;

    if (ix < NX && iy < NY)
    {
        Real sos = 0.0f;

        for (uint_t iz = 0; iz < nslices; ++iz)
        {
            const Real r = tex3D(texR, ix, iy, iz);
            const Real u = tex3D(texU, ix, iy, iz);
            const Real v = tex3D(texV, ix, iy, iz);
            const Real w = tex3D(texW, ix, iy, iz);
            const Real e = tex3D(texE, ix, iy, iz);
            const Real G = tex3D(texG, ix, iy, iz);
            const Real P = tex3D(texP, ix, iy, iz);

            const Real p = (e - 0.5f*(u*u + v*v + w*w)/r - P) / G;
            const Real c = sqrtf(((p + P) / G + p) / r);

            sos = fmaxf(sos, c + fmaxf(fmaxf(fabsf(u), fabsf(v)), fabsf(w)) / r);
        }
        block_sos[loc_idx] = sos;
        __syncthreads();

        if (0 == loc_idx)
        {
            for (int i = 1; i < _NTHREADS_; ++i)
                sos = fmaxf(sos, block_sos[i]);
            atomicMax(g_maxSOS, __float_as_int(sos));
        }
    }
}

///////////////////////////////////////////////////////////////////////////////
//                              KERNEL WRAPPERS                              //
///////////////////////////////////////////////////////////////////////////////
extern "C"
{
    void GPU::xflux(const uint_t BSX_GPU, const uint_t BSY_GPU, const uint_t nslices, const uint_t global_iz)
    {
#ifndef _MUTE_GPU_
        devPtrSet xghostL(d_xgl);
        devPtrSet xghostR(d_xgr);
        devPtrSet xflux(d_xflux);

        const dim3 blocks(_NTHREADS_, 1, 1);

        {
            const dim3 grid((NXP1 + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
            tCUDA_START(stream1)
            _xflux<<<grid, blocks, 0, stream1>>>(nslices, global_iz, xghostL, xghostR, xflux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
            tCUDA_STOP(stream1, "[_xflux Kernel]: ")
        }

        {
            const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
            tCUDA_START(stream1)
            _xextraterm_hllc<<<grid, blocks, 0, stream1>>>(nslices, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
            tCUDA_STOP(stream1, "[_xextraterm Kernel]: ")
        }
#endif
    }


    void GPU::yflux(const uint_t BSX_GPU, const uint_t BSY_GPU, const uint_t nslices, const uint_t global_iz)
    {
#ifndef _MUTE_GPU_
        devPtrSet yghostL(d_ygl);
        devPtrSet yghostR(d_ygr);
        devPtrSet yflux(d_yflux);

        const dim3 blocks(_NTHREADS_, 1, 1);

        {
            const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NYP1, 1);
            tCUDA_START(stream1)
            _yflux<<<grid, blocks, 0, stream1>>>(nslices, global_iz, yghostL, yghostR, yflux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
            tCUDA_STOP(stream1, "[_yflux Kernel]: ")
        }

        {
            const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
            tCUDA_START(stream1)
            _yextraterm_hllc<<<grid, blocks, 0, stream1>>>(nslices, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
            tCUDA_STOP(stream1, "[_yextraterm Kernel]: ")
        }
#endif
    }


    void GPU::zflux(const uint_t BSX_GPU, const uint_t BSY_GPU, const uint_t nslices)
    {
#ifndef _MUTE_GPU_
        devPtrSet zflux(d_zflux);

        const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
        const dim3 blocks(_NTHREADS_, 1, 1);

        tCUDA_START(stream1)
        // if only everything would be as easy as this!
        _zflux<<<grid, blocks, 0, stream1>>>(nslices, zflux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
        tCUDA_STOP(stream1, "[_zflux Kernel]: ")

        tCUDA_START(stream1)
        _zextraterm_hllc<<<grid, blocks, 0, stream1>>>(nslices, d_Gm, d_Gp, d_Pm, d_Pp, d_hllc_vel, d_sumG, d_sumP, d_divU);
        tCUDA_STOP(stream1, "[_zextraterm Kernel]: ")
#endif
    }


    void GPU::divergence(const Real a, const Real dtinvh, const uint_t BSX_GPU, const uint_t BSY_GPU, const uint_t nslices)
    {
#ifndef _MUTE_GPU_
        hipStreamWaitEvent(stream1, h2d_tmp_completed, 0);

        devPtrSet xflux(d_xflux);
        devPtrSet yflux(d_yflux);
        devPtrSet zflux(d_zflux);
        devPtrSet rhs(d_rhs);
        devPtrSet tmp(d_tmp);

        const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
        const dim3 blocks(_NTHREADS_, 1, 1);

        tCUDA_START(stream1)
        _divergence<<<grid, blocks, 0, stream1>>>(nslices, xflux, yflux, zflux, rhs, a, dtinvh, tmp, d_sumG, d_sumP, d_divU);
        tCUDA_STOP(stream1, "[_divergence Kernel]: ")

        hipEventRecord(divergence_completed, stream1);
#endif
    }


    void GPU::update(const Real b, const uint_t BSX_GPU, const uint_t BSY_GPU, const uint_t nslices)
    {
#ifndef _MUTE_GPU_
        devPtrSet tmp(d_tmp);
        devPtrSet rhs(d_rhs);

        const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
        const dim3 blocks(_NTHREADS_, 1, 1);

        tCUDA_START(stream1)
        _update<<<grid, blocks, 0, stream1>>>(nslices, b, tmp, rhs);
        tCUDA_STOP(stream1, "[_update Kernel]: ")

        hipEventRecord(update_completed, stream1);
#endif
    }


    void GPU::MaxSpeedOfSound(const uint_t BSX_GPU, const uint_t BSY_GPU, const uint_t nslices)
    {
#ifndef _MUTE_GPU_
        const dim3 grid((NX + _NTHREADS_ -1) / _NTHREADS_, NY, 1);
        const dim3 blocks(_NTHREADS_, 1, 1);

        tCUDA_START(stream1)
        _maxSOS<<<grid, blocks, 0, stream1>>>(nslices, d_maxSOS);
        tCUDA_STOP(stream1, "[_maxSOS Kernel]: ")
#endif
    }

    ///////////////////////////////////////////////////////////////////////////
    // TEST SECTION
    ///////////////////////////////////////////////////////////////////////////
    void GPU::TestKernel()
    {
        devPtrSet xghostL(d_xgl);
        devPtrSet xghostR(d_xgr);
        devPtrSet xflux(d_xflux);

        devPtrSet yghostL(d_ygl);
        devPtrSet yghostR(d_ygr);
        devPtrSet yflux(d_yflux);

        devPtrSet zflux(d_zflux);

        {
            const uint_t nslices = NodeBlock::sizeZ;

            const dim3 blocks(_NTHREADS_, 1, 1);
            const dim3 xgrid((NXP1 + _NTHREADS_ - 1) / _NTHREADS_, NY,   1);
            const dim3 ygrid((NX   + _NTHREADS_ - 1) / _NTHREADS_, NYP1, 1);
            const dim3 zgrid((NX   + _NTHREADS_ - 1) / _NTHREADS_, NY,   1);

            tCUDA_START(0)
            /* _xflux<<<xgrid, blocks>>>(nslices, 0, xghostL, xghostR, xflux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp); */
            /* _yflux<<<ygrid, blocks>>>(nslices, 0, yghostL, yghostR, yflux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp); */
            _zflux<<<zgrid, blocks>>>(nslices, zflux, d_hllc_vel, d_Gm, d_Gp, d_Pm, d_Pp);
            tCUDA_STOP(0, "[Testing Kernel]: ")
        }


    }
}


///////////////////////////////////////////////////////////////////////////////
//                                   UTILS                                   //
///////////////////////////////////////////////////////////////////////////////
static void _bindTexture(texture<float, 3, hipReadModeElementType> * const tex, hipArray_t d_ptr)
{
    hipChannelFormatDesc fmt = hipCreateChannelDesc<Real>();
    tex->addressMode[0]       = hipAddressModeClamp;
    tex->addressMode[1]       = hipAddressModeClamp;
    tex->addressMode[2]       = hipAddressModeClamp;
    tex->channelDesc          = fmt;
    tex->filterMode           = hipFilterModePoint;
    tex->mipmapFilterMode     = hipFilterModePoint;
    tex->normalized           = false;

    hipBindTextureToArray(tex, d_ptr, &fmt);
}


extern "C"
{
    void GPU::bind_textures()
    {
#ifndef _MUTE_GPU_
        _bindTexture(&texR, d_SOAin[0]);
        _bindTexture(&texU, d_SOAin[1]);
        _bindTexture(&texV, d_SOAin[2]);
        _bindTexture(&texW, d_SOAin[3]);
        _bindTexture(&texE, d_SOAin[4]);
        _bindTexture(&texG, d_SOAin[5]);
        _bindTexture(&texP, d_SOAin[6]);
#endif
    }


    void GPU::unbind_textures()
    {
#ifndef _MUTE_GPU_
        hipUnbindTexture(&texR);
        hipUnbindTexture(&texU);
        hipUnbindTexture(&texV);
        hipUnbindTexture(&texW);
        hipUnbindTexture(&texE);
        hipUnbindTexture(&texG);
        hipUnbindTexture(&texP);
#endif
    }
}
